
#include <hip/hip_runtime.h>
#include <stdio.h>
#define CHECK_BANK_CONFLICTS 0
#if CHECK_BANK_CONFLICTS
#define AS(i, j) cutilBankChecker(((float*)&As[0][0]), (BLOCK_SIZE * i + j))
#define BS(i, j) cutilBankChecker(((float*)&Bs[0][0]), (BLOCK_SIZE * i + j))
#else
#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]
#endif

//////This kernel is much faster
template <int BLOCK_SIZE> __global__ void
matrixMul( float* C, float* A, float* B, int wA, int wB)
{
    // Block index

    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
             a <= aEnd;
             a += aStep, b += bStep) {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        AS(ty, tx) = A[a + wA * ty + tx];
        BS(ty, tx) = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k)
            Csub += AS(ty, k) * BS(k, tx);

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}




////This kernel is much slower...
__global__ void matrixMulDS(float *C, float *A, float *B, int wA, int wB)
{
   // Block index
   const int BLOCK_SIZE = 32;
   int bx = blockIdx.x;
   int by = blockIdx.y;

   // Thread index
   int tx = threadIdx.x;
   int ty = threadIdx.y;

   // Index of the first sub-matrix of A processed by the block
   int aBegin = wA * BLOCK_SIZE * by;

   // Index of the last sub-matrix of A processed by the block
   int aEnd   = aBegin + wA - 1;

   // Step size used to iterate through the sub-matrices of A
   int aStep  = BLOCK_SIZE;

   // Index of the first sub-matrix of B processed by the block
   int bBegin = BLOCK_SIZE * bx;

   // Step size used to iterate through the sub-matrices of B
   int bStep  = BLOCK_SIZE * wB;

   // Csub is used to store the element of the block sub-matrix
   // that is computed by the thread
   float Csub = 0;

   // Loop over all the sub-matrices of A and B
   // required to compute the block sub-matrix



   for (int a = aBegin, b = bBegin;a <= aEnd;a += aStep, b += bStep)
   {
       // Declaration of the shared memory array As used to
       // store the sub-matrix of A
 	   __shared__ float As[32][32];
       __shared__ float Bs[32][32];

	   // Load the matrices from device memory
       // to shared memory; each thread loads
       // one element of each matrix
       //As[ty][tx] = A[a + wA * ty + tx];
       //Bs[ty][tx] = B[b + wB * ty + tx];
       AS(ty, tx) = A[a + wA * ty + tx];
       BS(ty, tx) = B[b + wB * ty + tx];

       // Synchronize to make sure the matrices are loaded
       __syncthreads();

       // Multiply the two matrices together;
       // each thread computes one element
       // of the block sub-matrix
#pragma unroll

       for (int k = 0; k < BLOCK_SIZE; ++k)
       {
           //Csub += As[ty][k] * Bs[k][tx];
    	   Csub += AS(ty, k) * BS(k, tx);
       }
       // Synchronize to make sure that the preceding
       // computation is done before loading two new
       // sub-matrices of A and B in the next iteration
       __syncthreads();
   }

   // Write the block sub-matrix to device memory;
   // each thread writes one element
   int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
   C[c + wB * ty + tx] = Csub;
}



