static char *Ptxdata = 
    "	.version 1.4\n"
    "	.target sm_10, map_f64_to_f32\n"
    "	// compiled with /usr/local/cuda-6.0/open64/lib//be\n"
    "	// nvopencc 4.1 built on 2014-03-13\n"
    "\n"
    "	//-----------------------------------------------------------\n"
    "	// Compiling /tmp/tmpxft_000006d2_00000000-9_broadcast.cpp3.i (/tmp/ccBI#.172qxP)\n"
    "	//-----------------------------------------------------------\n"
    "\n"
    "	//-----------------------------------------------------------\n"
    "	// Options:\n"
    "	//-----------------------------------------------------------\n"
    "	//  Target:ptx, ISA:sm_10, Endian:little, Pointer Size:64\n"
    "	//  -O3	(Optimization level)\n"
    "	//  -g0	(Debug level)\n"
    "	//  -m2	(Report advisories)\n"
    "	//-----------------------------------------------------------\n"
    "\n"
    "	.file	1	\"<command-line>\"\n"
    "	.file	2	\"/tmp/tmpxft_000006d2_00000000-8_broadcast.cudafe2.gpu\"\n"
    "	.file	3	\"/usr/lib/gcc/x86_64-linux-gnu/4.6/include/stddef.h\"\n"
    "	.file	4	\"/usr/local/cuda/include/crt/device_runtime.h\"\n"
    "	.file	5	\"/usr/local/cuda/include/host_defines.h\"\n"
    "	.file	6	\"/usr/local/cuda/include/builtin_types.h\"\n"
    "	.file	7	\"/usr/local/cuda/include/device_types.h\"\n"
    "	.file	8	\"/usr/local/cuda/include/hip/driver_types.h\"\n"
    "	.file	9	\"/usr/local/cuda/include/surface_types.h\"\n"
    "	.file	10	\"/usr/local/cuda/include/texture_types.h\"\n"
    "	.file	11	\"/usr/local/cuda/include/hip/hip_vector_types.h\"\n"
    "	.file	12	\"/usr/local/cuda/include/\"\n"
    "	.file	13	\"/usr/local/cuda/include/crt/storage_class.h\"\n"
    "	.file	14	\"/usr/local/cuda/include/common_functions.h\"\n"
    "	.file	15	\"/usr/local/cuda/include/math_functions.h\"\n"
    "	.file	16	\"/usr/local/cuda/include/hip/hip_math_constants.h\"\n"
    "	.file	17	\"/usr/local/cuda/include/hip/device_functions.h\"\n"
    "	.file	18	\"/usr/local/cuda/include/sm_11_atomic_functions.h\"\n"
    "	.file	19	\"/usr/local/cuda/include/sm_12_atomic_functions.h\"\n"
    "	.file	20	\"/usr/local/cuda/include/sm_13_double_functions.h\"\n"
    "	.file	21	\"/usr/local/cuda/include/sm_20_atomic_functions.h\"\n"
    "	.file	22	\"/usr/local/cuda/include/sm_32_atomic_functions.h\"\n"
    "	.file	23	\"/usr/local/cuda/include/sm_35_atomic_functions.h\"\n"
    "	.file	24	\"/usr/local/cuda/include/sm_20_intrinsics.h\"\n"
    "	.file	25	\"/usr/local/cuda/include/sm_30_intrinsics.h\"\n"
    "	.file	26	\"/usr/local/cuda/include/sm_32_intrinsics.h\"\n"
    "	.file	27	\"/usr/local/cuda/include/sm_35_intrinsics.h\"\n"
    "	.file	28	\"/usr/local/cuda/include/surface_functions.h\"\n"
    "	.file	29	\"/usr/local/cuda/include/\"\n"
    "	.file	30	\"/usr/local/cuda/include/texture_indirect_functions.h\"\n"
    "	.file	31	\"/usr/local/cuda/include/surface_indirect_functions.h\"\n"
    "	.file	32	\"/usr/local/cuda/include/math_functions_dbl_ptx1.h\"\n"
    "\n"
    "\n";
#pragma dscuda endofptx
#include "dscuda.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <cutil.h>
#include <cutil_inline.h>
#include <sys/time.h>
#ifdef _OPENMP
#include <omp.h>
#endif // _OPENMP

#define MAXDEV 32
static const double MEGA  = 1e6;

hipError_t cudaMemcpyToAlldev(int ndev, void **dst, const void *src, size_t count, enum hipMemcpyKind kind);

static void
get_cputime(double *splittime, double *laptime)
{
    struct timeval x;

    gettimeofday(&x, NULL);

    *splittime = x.tv_sec + x.tv_usec/1000000.0 - *laptime;
    *laptime = x.tv_sec + x.tv_usec/1000000.0;
}

static void
bcastperf(int argc, char **argv)
{
    int maxsize = 1024 * 1024 * 10.0;
    int i, j;
    size_t size0 = 4096, size;
    double sized;
    double lt = 0.0, st = 0.0;
    double ratio = 2.0;
    double nloop = 2e8;
    char *src = (char *)malloc(sizeof(char) * maxsize);
    char *dst[MAXDEV];
    int ndev0 = 1, ndev, ndevmax;
    static int nthread = 0;

    if (1 < argc) {
        ndev0 = atoi(argv[1]);
    }
    if (2 < argc) {
        size0 = atoi(argv[2]);
    }
    cutilSafeCall(hipGetDeviceCount(&ndevmax));
    printf("# %d device%s found.\n", ndevmax, ndevmax > 1 ? "s" : "");

    for (i = 0; i < ndevmax; i++) {
        hipSetDevice(i);
        cutilSafeCall(hipMalloc((void**) &dst[i], sizeof(char) * maxsize));
    }
    printf("\n#\n# hipMemcpy (HostToDevice)\n");
    printf("# broadcast to %d..%d servers.\n#\n", ndev0, ndevmax);

    for (sized = size0; sized < maxsize; sized *= ratio) {
        //    for ( nloop = 2e8, sized = 4096 * 1; ; ) { // !!!
        size = (size_t)sized;

        for (ndev = ndev0; ndev <= ndevmax; ndev++) { // # of devices broadcast to.
            get_cputime(&lt, &st);
#pragma omp parallel for private(j)
            for (i = 0; i < ndev; i++) {
#ifdef _OPENMP
                if (nthread == 0) {
                    nthread = omp_get_num_threads();
                    fprintf(stderr, "nthread:%d\n", nthread);
                }
#endif // _OPENMP
                for (j = 0; j < nloop/size; j++) { // # of iterations.
                    hipSetDevice(i);
                    hipMemcpy(dst[i], src, size, hipMemcpyHostToDevice);
                } // i
                hipDeviceSynchronize();
            } // j
            get_cputime(&lt, &st);
            printf("%d devices %d byte    %f sec    %f MB/s   %f MB/s\n",
                   ndev, size, lt, nloop/MEGA/lt, nloop/MEGA/lt*ndev);
            fflush(stdout);
	} // ndev
    } // sized
}

int
main(int argc, char **argv)
{
    bcastperf(argc, argv);
    fprintf(stderr, "going to quit...\n");
    sleep(1);
    exit(0);
}
