#include "hip/hip_runtime.h"
static char *Ptxdata = 
    "	.version 1.4\n"
    "	.target sm_10, map_f64_to_f32\n"
    "	// compiled with /usr/local/cuda4.1/cuda/open64/lib//be\n"
    "	// nvopencc 4.1 built on 2012-01-12\n"
    "\n"
    "	//-----------------------------------------------------------\n"
    "	// Compiling /tmp/tmpxft_0000053e_00000000-9_matrixMul.cpp3.i (/tmp/ccBI#.rWsLED)\n"
    "	//-----------------------------------------------------------\n"
    "\n"
    "	//-----------------------------------------------------------\n"
    "	// Options:\n"
    "	//-----------------------------------------------------------\n"
    "	//  Target:ptx, ISA:sm_10, Endian:little, Pointer Size:64\n"
    "	//  -O3	(Optimization level)\n"
    "	//  -g0	(Debug level)\n"
    "	//  -m2	(Report advisories)\n"
    "	//-----------------------------------------------------------\n"
    "\n"
    "	.file	1	\"<command-line>\"\n"
    "	.file	2	\"/tmp/tmpxft_0000053e_00000000-8_matrixMul.cudafe2.gpu\"\n"
    "	.file	3	\"/usr/lib/gcc/x86_64-redhat-linux/4.5.1/include/stddef.h\"\n"
    "	.file	4	\"/usr/local/cuda/include/crt/device_runtime.h\"\n"
    "	.file	5	\"/usr/local/cuda/include/host_defines.h\"\n"
    "	.file	6	\"/usr/local/cuda/include/builtin_types.h\"\n"
    "	.file	7	\"/usr/local/cuda/include/device_types.h\"\n"
    "	.file	8	\"/usr/local/cuda/include/hip/driver_types.h\"\n"
    "	.file	9	\"/usr/local/cuda/include/surface_types.h\"\n"
    "	.file	10	\"/usr/local/cuda/include/texture_types.h\"\n"
    "	.file	11	\"/usr/local/cuda/include/hip/hip_vector_types.h\"\n"
    "	.file	12	\"/usr/local/cuda/include/\"\n"
    "	.file	13	\"/usr/local/cuda/include/crt/storage_class.h\"\n"
    "	.file	14	\"matrixMulkernel.cu\"\n"
    "	.file	15	\"/usr/local/cuda/include/common_functions.h\"\n"
    "	.file	16	\"/usr/local/cuda/include/math_functions.h\"\n"
    "	.file	17	\"/usr/local/cuda/include/hip/hip_math_constants.h\"\n"
    "	.file	18	\"/usr/local/cuda/include/hip/device_functions.h\"\n"
    "	.file	19	\"/usr/local/cuda/include/sm_11_atomic_functions.h\"\n"
    "	.file	20	\"/usr/local/cuda/include/sm_12_atomic_functions.h\"\n"
    "	.file	21	\"/usr/local/cuda/include/sm_13_double_functions.h\"\n"
    "	.file	22	\"/usr/local/cuda/include/sm_20_atomic_functions.h\"\n"
    "	.file	23	\"/usr/local/cuda/include/sm_20_intrinsics.h\"\n"
    "	.file	24	\"/usr/local/cuda/include/surface_functions.h\"\n"
    "	.file	25	\"/usr/local/cuda/include/\"\n"
    "	.file	26	\"/usr/local/cuda/include/math_functions_dbl_ptx1.h\"\n"
    "\n"
    "\n"
    "	.entry _Z11matrixMulDSPfS_S_ii (\n"
    "		.param .u64 __cudaparm__Z11matrixMulDSPfS_S_ii_C,\n"
    "		.param .u64 __cudaparm__Z11matrixMulDSPfS_S_ii_A,\n"
    "		.param .u64 __cudaparm__Z11matrixMulDSPfS_S_ii_B,\n"
    "		.param .s32 __cudaparm__Z11matrixMulDSPfS_S_ii_wA,\n"
    "		.param .s32 __cudaparm__Z11matrixMulDSPfS_S_ii_wB)\n"
    "	{\n"
    "	.reg .u32 %r<34>;\n"
    "	.reg .u64 %rd<29>;\n"
    "	.reg .f32 %f<100>;\n"
    "	.reg .pred %p<4>;\n"
    "	.shared .align 4 .b8 __cuda___cuda_local_var_37736_38_non_const_Bs32[4096];\n"
    "	.shared .align 4 .b8 __cuda___cuda_local_var_37735_36_non_const_As4128[4096];\n"
    "	.loc	14	89	0\n"
    "$LDWbegin__Z11matrixMulDSPfS_S_ii:\n"
    "	.loc	14	124	0\n"
    "	cvt.s32.u16 	%r1, %ctaid.x;\n"
    "	mul24.lo.s32 	%r2, %r1, 32;\n"
    "	cvt.s32.u16 	%r3, %ctaid.y;\n"
    "	ld.param.s32 	%r4, [__cudaparm__Z11matrixMulDSPfS_S_ii_wA];\n"
    "	mul.lo.s32 	%r5, %r3, %r4;\n"
    "	mul.lo.s32 	%r6, %r5, 32;\n"
    "	add.s32 	%r7, %r6, %r4;\n"
    "	sub.s32 	%r8, %r7, 1;\n"
    "	cvt.s32.u16 	%r9, %tid.x;\n"
    "	cvt.s32.u16 	%r10, %tid.y;\n"
    "	ld.param.s32 	%r11, [__cudaparm__Z11matrixMulDSPfS_S_ii_wB];\n"
    "	setp.lt.s32 	%p1, %r8, %r6;\n"
    "	@%p1 bra 	$Lt_0_3330;\n"
    "	mov.u64 	%rd1, __cuda___cuda_local_var_37736_38_non_const_Bs32;\n"
    "	mov.u64 	%rd2, __cuda___cuda_local_var_37735_36_non_const_As4128;\n"
    "	ld.param.s32 	%r4, [__cudaparm__Z11matrixMulDSPfS_S_ii_wA];\n"
    "	add.s32 	%r12, %r4, 31;\n"
    "	shr.s32 	%r13, %r12, 31;\n"
    "	mov.s32 	%r14, 31;\n"
    "	and.b32 	%r15, %r13, %r14;\n"
    "	add.s32 	%r16, %r15, %r12;\n"
    "	shr.s32 	%r17, %r16, 5;\n"
    "	ld.param.s32 	%r11, [__cudaparm__Z11matrixMulDSPfS_S_ii_wB];\n"
    "	mul.lo.s32 	%r18, %r10, %r11;\n"
    "	mul.lo.s32 	%r19, %r10, %r4;\n"
    "	cvt.s64.s32 	%rd3, %r9;\n"
    "	cvt.s64.s32 	%rd4, %r10;\n"
    "	add.s32 	%r20, %r19, %r6;\n"
    "	add.s32 	%r21, %r9, %r20;\n"
    "	mul.wide.s32 	%rd5, %r9, 4;\n"
    "	add.u64 	%rd6, %rd1, %rd5;\n"
    "	mul.wide.s32 	%rd7, %r10, 128;\n"
    "	add.u64 	%rd8, %rd2, %rd7;\n"
    "	mul.wide.s32 	%rd9, %r10, 32;\n"
    "	add.u64 	%rd10, %rd3, %rd9;\n"
    "	mul.lo.u64 	%rd11, %rd10, 4;\n"
    "	add.s32 	%r22, %r19, %r8;\n"
    "	mul.lo.s32 	%r23, %r11, 32;\n"
    "	cvt.s64.s32 	%rd12, %r23;\n"
    "	mul.wide.s32 	%rd13, %r23, 4;\n"
    "	add.u64 	%rd14, %rd11, %rd2;\n"
    "	add.u64 	%rd15, %rd11, %rd1;\n"
    "	add.s32 	%r24, %r22, %r9;\n"
    "	ld.param.u64 	%rd16, [__cudaparm__Z11matrixMulDSPfS_S_ii_B];\n"
    "	add.s32 	%r25, %r18, %r2;\n"
    "	add.s32 	%r26, %r9, %r25;\n"
    "	cvt.s64.s32 	%rd17, %r26;\n"
    "	mul.wide.s32 	%rd18, %r26, 4;\n"
    "	add.u64 	%rd19, %rd16, %rd18;\n"
    "	ld.param.u64 	%rd20, [__cudaparm__Z11matrixMulDSPfS_S_ii_A];\n"
    "	cvt.s64.s32 	%rd21, %r21;\n"
    "	mul.wide.s32 	%rd22, %r21, 4;\n"
    "	add.u64 	%rd23, %rd20, %rd22;\n"
    "	mov.f32 	%f1, 0f00000000;     	// 0\n"
    "	mov.s32 	%r27, %r17;\n"
    "$Lt_0_2818:\n"
    " //<loop> Loop body line 124, nesting depth: 1, estimated iterations: unknown\n"
    "	.loc	14	136	0\n"
    "	ld.global.f32 	%f2, [%rd23+0];\n"
    "	st.shared.f32 	[%rd14+0], %f2;\n"
    "	.loc	14	137	0\n"
    "	ld.global.f32 	%f3, [%rd19+0];\n"
    "	st.shared.f32 	[%rd15+0], %f3;\n"
    "	.loc	14	140	0\n"
    "	bar.sync 	0;\n"
    "	.loc	14	150	0\n"
    "	ld.shared.f32 	%f4, [%rd8+0];\n"
    "	ld.shared.f32 	%f5, [%rd6+0];\n"
    "	mad.f32 	%f6, %f4, %f5, %f1;\n"
    "	ld.shared.f32 	%f7, [%rd8+4];\n"
    "	ld.shared.f32 	%f8, [%rd6+128];\n"
    "	mad.f32 	%f9, %f7, %f8, %f6;\n"
    "	ld.shared.f32 	%f10, [%rd8+8];\n"
    "	ld.shared.f32 	%f11, [%rd6+256];\n"
    "	mad.f32 	%f12, %f10, %f11, %f9;\n"
    "	ld.shared.f32 	%f13, [%rd8+12];\n"
    "	ld.shared.f32 	%f14, [%rd6+384];\n"
    "	mad.f32 	%f15, %f13, %f14, %f12;\n"
    "	ld.shared.f32 	%f16, [%rd8+16];\n"
    "	ld.shared.f32 	%f17, [%rd6+512];\n"
    "	mad.f32 	%f18, %f16, %f17, %f15;\n"
    "	ld.shared.f32 	%f19, [%rd8+20];\n"
    "	ld.shared.f32 	%f20, [%rd6+640];\n"
    "	mad.f32 	%f21, %f19, %f20, %f18;\n"
    "	ld.shared.f32 	%f22, [%rd8+24];\n"
    "	ld.shared.f32 	%f23, [%rd6+768];\n"
    "	mad.f32 	%f24, %f22, %f23, %f21;\n"
    "	ld.shared.f32 	%f25, [%rd8+28];\n"
    "	ld.shared.f32 	%f26, [%rd6+896];\n"
    "	mad.f32 	%f27, %f25, %f26, %f24;\n"
    "	ld.shared.f32 	%f28, [%rd8+32];\n"
    "	ld.shared.f32 	%f29, [%rd6+1024];\n"
    "	mad.f32 	%f30, %f28, %f29, %f27;\n"
    "	ld.shared.f32 	%f31, [%rd8+36];\n"
    "	ld.shared.f32 	%f32, [%rd6+1152];\n"
    "	mad.f32 	%f33, %f31, %f32, %f30;\n"
    "	ld.shared.f32 	%f34, [%rd8+40];\n"
    "	ld.shared.f32 	%f35, [%rd6+1280];\n"
    "	mad.f32 	%f36, %f34, %f35, %f33;\n"
    "	ld.shared.f32 	%f37, [%rd8+44];\n"
    "	ld.shared.f32 	%f38, [%rd6+1408];\n"
    "	mad.f32 	%f39, %f37, %f38, %f36;\n"
    "	ld.shared.f32 	%f40, [%rd8+48];\n"
    "	ld.shared.f32 	%f41, [%rd6+1536];\n"
    "	mad.f32 	%f42, %f40, %f41, %f39;\n"
    "	ld.shared.f32 	%f43, [%rd8+52];\n"
    "	ld.shared.f32 	%f44, [%rd6+1664];\n"
    "	mad.f32 	%f45, %f43, %f44, %f42;\n"
    "	ld.shared.f32 	%f46, [%rd8+56];\n"
    "	ld.shared.f32 	%f47, [%rd6+1792];\n"
    "	mad.f32 	%f48, %f46, %f47, %f45;\n"
    "	ld.shared.f32 	%f49, [%rd8+60];\n"
    "	ld.shared.f32 	%f50, [%rd6+1920];\n"
    "	mad.f32 	%f51, %f49, %f50, %f48;\n"
    "	ld.shared.f32 	%f52, [%rd8+64];\n"
    "	ld.shared.f32 	%f53, [%rd6+2048];\n"
    "	mad.f32 	%f54, %f52, %f53, %f51;\n"
    "	ld.shared.f32 	%f55, [%rd8+68];\n"
    "	ld.shared.f32 	%f56, [%rd6+2176];\n"
    "	mad.f32 	%f57, %f55, %f56, %f54;\n"
    "	ld.shared.f32 	%f58, [%rd8+72];\n"
    "	ld.shared.f32 	%f59, [%rd6+2304];\n"
    "	mad.f32 	%f60, %f58, %f59, %f57;\n"
    "	ld.shared.f32 	%f61, [%rd8+76];\n"
    "	ld.shared.f32 	%f62, [%rd6+2432];\n"
    "	mad.f32 	%f63, %f61, %f62, %f60;\n"
    "	ld.shared.f32 	%f64, [%rd8+80];\n"
    "	ld.shared.f32 	%f65, [%rd6+2560];\n"
    "	mad.f32 	%f66, %f64, %f65, %f63;\n"
    "	ld.shared.f32 	%f67, [%rd8+84];\n"
    "	ld.shared.f32 	%f68, [%rd6+2688];\n"
    "	mad.f32 	%f69, %f67, %f68, %f66;\n"
    "	ld.shared.f32 	%f70, [%rd8+88];\n"
    "	ld.shared.f32 	%f71, [%rd6+2816];\n"
    "	mad.f32 	%f72, %f70, %f71, %f69;\n"
    "	ld.shared.f32 	%f73, [%rd8+92];\n"
    "	ld.shared.f32 	%f74, [%rd6+2944];\n"
    "	mad.f32 	%f75, %f73, %f74, %f72;\n"
    "	ld.shared.f32 	%f76, [%rd8+96];\n"
    "	ld.shared.f32 	%f77, [%rd6+3072];\n"
    "	mad.f32 	%f78, %f76, %f77, %f75;\n"
    "	ld.shared.f32 	%f79, [%rd8+100];\n"
    "	ld.shared.f32 	%f80, [%rd6+3200];\n"
    "	mad.f32 	%f81, %f79, %f80, %f78;\n"
    "	ld.shared.f32 	%f82, [%rd8+104];\n"
    "	ld.shared.f32 	%f83, [%rd6+3328];\n"
    "	mad.f32 	%f84, %f82, %f83, %f81;\n"
    "	ld.shared.f32 	%f85, [%rd8+108];\n"
    "	ld.shared.f32 	%f86, [%rd6+3456];\n"
    "	mad.f32 	%f87, %f85, %f86, %f84;\n"
    "	ld.shared.f32 	%f88, [%rd8+112];\n"
    "	ld.shared.f32 	%f89, [%rd6+3584];\n"
    "	mad.f32 	%f90, %f88, %f89, %f87;\n"
    "	ld.shared.f32 	%f91, [%rd8+116];\n"
    "	ld.shared.f32 	%f92, [%rd6+3712];\n"
    "	mad.f32 	%f93, %f91, %f92, %f90;\n"
    "	ld.shared.f32 	%f94, [%rd8+120];\n"
    "	ld.shared.f32 	%f95, [%rd6+3840];\n"
    "	mad.f32 	%f96, %f94, %f95, %f93;\n"
    "	ld.shared.f32 	%f97, [%rd8+124];\n"
    "	ld.shared.f32 	%f98, [%rd6+3968];\n"
    "	mad.f32 	%f1, %f97, %f98, %f96;\n"
    "	.loc	14	155	0\n"
    "	bar.sync 	0;\n"
    "	.loc	14	124	0\n"
    "	add.u64 	%rd19, %rd13, %rd19;\n"
    "	add.s32 	%r21, %r21, 32;\n"
    "	add.u64 	%rd23, %rd23, 128;\n"
    "	setp.le.s32 	%p2, %r21, %r24;\n"
    "	@%p2 bra 	$Lt_0_2818;\n"
    "	bra.uni 	$Lt_0_2306;\n"
    "$Lt_0_3330:\n"
    "	ld.param.s32 	%r11, [__cudaparm__Z11matrixMulDSPfS_S_ii_wB];\n"
    "	mul.lo.s32 	%r18, %r10, %r11;\n"
    "	mov.f32 	%f1, 0f00000000;     	// 0\n"
    "$Lt_0_2306:\n"
    "	.loc	14	161	0\n"
    "	ld.param.u64 	%rd24, [__cudaparm__Z11matrixMulDSPfS_S_ii_C];\n"
    "	mul.lo.s32 	%r28, %r11, %r3;\n"
    "	add.s32 	%r29, %r1, %r28;\n"
    "	mul.lo.s32 	%r30, %r29, 32;\n"
    "	add.s32 	%r31, %r18, %r30;\n"
    "	add.s32 	%r32, %r9, %r31;\n"
    "	cvt.s64.s32 	%rd25, %r32;\n"
    "	mul.wide.s32 	%rd26, %r32, 4;\n"
    "	add.u64 	%rd27, %rd24, %rd26;\n"
    "	st.global.f32 	[%rd27+0], %f1;\n"
    "	.loc	14	162	0\n"
    "	exit;\n"
    "$LDWend__Z11matrixMulDSPfS_S_ii:\n"
    "	} // _Z11matrixMulDSPfS_S_ii\n"
    "\n";
#pragma dscuda endofptx
#include "dscuda.h"
// Martinez Noriega Edgar Josafat     14/05/2013
// Based on CUDA SDK 4.1 MatrixMul

// C includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

// CUDA Includes
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
#include <cutil.h>
#include <cutil_inline.h>
//DSCUDA Includes
//#include "dscuda.h"
//CUBLAS include
//#include <hipblas.h>
//#include <hip/hip_runtime.h>

// Kernel Includes
#include "matrixMulkernel.cu"

#define WA (4 * block_size) // Matrix A width
#define HA (6 * block_size) // Matrix A height
#define WB (4 * block_size) // Matrix B width
#define HB WA  // Matrix B height
#define WC WB  // Matrix C width 
#define HC HA  // Matrix C height
#define MEGA 1024*1024




///To measure the time...
static void
get_cputime(double *splittime, double *laptime)
{
    struct timeval x;

    gettimeofday(&x, NULL);

    *splittime = x.tv_sec + x.tv_usec/1000000.0 - *laptime;
    *laptime = x.tv_sec + x.tv_usec/1000000.0;
}

////Random numbers generator for matrix
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

////Matrix Multiplication routine in CPU
void KernelCPU(float* C, const float* A, const float* B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    for (unsigned int i = 0; i < hA; ++i)
        for (unsigned int j = 0; j < wB; ++j) {
            double sum = 0;
            for (unsigned int k = 0; k < wA; ++k) {
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }
            C[i * wB + j] = (float)sum;
        }
}

////////////////////////////////Main//////////////////////////////////
//////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
	printf("\n\n[matrixMul starting...]\n");

	char *as=NULL;
	char num ='1';

	int devID=0;
	int iSizeMultiple=5;
	hipDeviceProp_t props;
	double lt=0.0, st=0.0;

	cutilSafeCall(hipGetDeviceCount(&devID));
	printf("\n# %d device%s found.\n", devID, devID > 1 ? "s" : "");
	devID=0;
	cutilSafeCall(hipSetDevice(devID));
	cutilSafeCall(hipGetDeviceProperties(&props, devID));
	int block_size = 32;
	
	printf("\n|||||||||||||||||||||||||||||||||||||||||DS_CUDA mulMatrix....|||||");
	printf("\n\nDevice %d: \"%s\" with Compute %d.%d capability\n", 0, props.name, props.major, props.minor);
	
	// Optional Command-line multiplier for matrix sizes
	unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;

	//Checking for Arguments....
	////////////////////////////////////
	if ( argc < 2 ) /* argc should be 2 for correct execution */
		{
			iSizeMultiple = 5;
		}
		else
		{
			//ts = (int) argv[1];
			as = argv[1];

			for(int i=1;i<11;i++)
				{
					if(num == as[0])
					{
						iSizeMultiple = i;
						break;
					}
					num++;
				}
		}
	/////////////////////////////////////
	
	// For GPUs with fewer # of SM's, we limit the maximum size of the matrix
	if (props.multiProcessorCount <= 4) {
		uiWA = 2 * block_size * iSizeMultiple;
		uiHA = 4 * block_size * iSizeMultiple;
		uiWB = 2 * block_size * iSizeMultiple;
		uiHB = 4 * block_size * iSizeMultiple;
		uiWC = 2 * block_size * iSizeMultiple;
		uiHC = 4 * block_size * iSizeMultiple;
	} else {
		uiWA = WA * iSizeMultiple;
		uiHA = HA * iSizeMultiple;
		uiWB = WB * iSizeMultiple;
		uiHB = HB * iSizeMultiple;
		uiWC = WC * iSizeMultiple;
		uiHC = HC * iSizeMultiple;
	}


	//Variables for GPU kernell
	// setup execution parameters
	printf("\nMatrix Sizes:A(%u x %u), B(%u x %u), C(%u x %u)\n\n",uiWA, uiHA, uiWB, uiHB, uiWC, uiHC);
	dim3 threads(block_size, block_size);
	dim3 grid(uiWC / threads.x, uiHC / threads.y);
	
	printf("\nRunning kernels......\n");
	int nIter = 60;
	printf("\nNumber of iterations for each kernel %i",nIter);
	
	
	// allocate host memory for matrices A,B and C
	unsigned int size_A = uiWA * uiHA;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float* h_A = (float*)malloc(mem_size_A);
	
	unsigned int size_B = uiWB * uiHB;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float* h_B = (float*)malloc(mem_size_B);
	
	unsigned int size_C = uiWC * uiHC;	
	unsigned int mem_size_C = sizeof(float) * size_C;
	float* h_C      = (float*) malloc(mem_size_C);
	
	//Generate Aleatory Data
	srand(2013);
	randomInit(h_A, size_A);
	randomInit(h_B, size_B);

	
	// Allocate device (GPU) memory
	float* d_A, *d_B, *d_C;
	
	cutilSafeCall(hipMalloc((void**) &d_A, mem_size_A));
	cutilSafeCall(hipMalloc((void**) &d_B, mem_size_B));
	cutilSafeCall(hipMalloc((void**) &d_C, mem_size_C));
	
	// Copy data from CPU memory to GPU memory
	cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) );
	cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) );
	printf("\nTotal amount of memory to be sent from CPU to GPU: %d Bytes",mem_size_A+mem_size_B);

	
	
	// Make warmup operation and Synchronize GPU
	matrixMulDS<<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
	//matrixMul<32><<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
	hipDeviceSynchronize();
	
	
	// execute the kernel in GPU
	get_cputime(&lt,&st);
	for (int j = 0;j< nIter;j++){
		matrixMulDS<<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
		//matrixMul<32><<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
	}
	hipDeviceSynchronize();
	get_cputime(&lt,&st);

	
	// Copying memory back from GPU to CPU
	cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost) );
	

	//Calculating of Performance .... Gflops
	printf("\nTotal amount of memory to be sent from GPU to CPU: %d Bytes",mem_size_C);
	double dSeconds = lt/((double)nIter);
	double dNumOps = 2.0 * (double)uiWA * (double)uiHA * (double)uiWB;
	double gflops = 1.0e-9 * dNumOps/dSeconds;
	printf("\n>DSCUDA\t%.4f GFlop/s,\tTime:%.8f s,\tSize:%.0f Ops ",gflops,dSeconds,dNumOps);

#if 1	
	// Variables for measuring the time
	lt = 0.0;
	st = 0.0;
	nIter = 2;
	dSeconds = 0.0;
	dNumOps = 0.0;
	gflops = 0.0;
	bool correct = true;
	float* reference = (float*)malloc(mem_size_C);
	
	// execute kernel in CPU
	get_cputime(&lt,&st);
	
	for(int i=0;i<nIter;i++)
	{
		KernelCPU(reference, h_A, h_B, uiHA, uiWA, uiWB);
		
	}
	get_cputime(&lt,&st);
	
	
	
	dSeconds = lt/((double)nIter);
	dNumOps = 2.0 * (double)uiWA * (double)uiHA * (double)uiWB;
	gflops = 1.0e-9 * dNumOps/dSeconds;
	printf("\n> CPU\t\t%.4f GFlop/s\t, Time:%.5f s, Size:%.0f Ops ",gflops,dSeconds,dNumOps);

	printf("\n\nComparing GPU results with CPU calculation...");
	// Comparing Results between CPU and GPU calculations
	for (int i = 0; i < size_C; i++)
	    {
	        if (fabs(h_C[i] - reference[i]) > 1e-3)
	        {
	            printf("Error! GPUmem[%05d]=%.8f, CPUmem=%.8f error term is %.8f > 1e-3\n", i, h_C[i],reference[i],fabs(h_C[i] - reference[i]));
	            correct = false;
	        }
	    }

	printf("%s\n", correct ? "OK" : "FAIL");
	if (correct){
		printf("\nMatrix");
		for (int i=0;i<10;i++){
			printf("\nGPUmem[%05d]=%.8f, CPUmem=%.8f ---- Difference... %.8f < 1e-3", i, h_C[i],reference[i],fabs(h_C[i] - reference[i]));
		}
	}
	
	//Free memory
	free(reference);
	
#endif

	//Cleaning Memory....
	free(h_A);
	free(h_B);
	free(h_C);
	
	cutilSafeCall(hipFree(d_A));
	cutilSafeCall(hipFree(d_B));
	cutilSafeCall(hipFree(d_C));
	printf("\n\nExit mulMatrix Program...\n\n");

	
    exit(0);
    return 0;
}
