#include "hip/hip_runtime.h"
static char *Ptxdata = 
    "	.version 1.4\n"
    "	.target sm_10, map_f64_to_f32\n"
    "	// compiled with /usr/local/cuda-4.1/open64/lib//be\n"
    "	// nvopencc 4.1 built on 2012-01-12\n"
    "\n"
    "	//-----------------------------------------------------------\n"
    "	// Compiling /tmp/tmpxft_00005f50_00000000-9_matrixMul.cpp3.i (/tmp/ccBI#.tnKWs0)\n"
    "	//-----------------------------------------------------------\n"
    "\n"
    "	//-----------------------------------------------------------\n"
    "	// Options:\n"
    "	//-----------------------------------------------------------\n"
    "	//  Target:ptx, ISA:sm_10, Endian:little, Pointer Size:64\n"
    "	//  -O3	(Optimization level)\n"
    "	//  -g0	(Debug level)\n"
    "	//  -m2	(Report advisories)\n"
    "	//-----------------------------------------------------------\n"
    "\n"
    "	.file	1	\"<command-line>\"\n"
    "	.file	2	\"/tmp/tmpxft_00005f50_00000000-8_matrixMul.cudafe2.gpu\"\n"
    "	.file	3	\"/usr/lib/gcc/x86_64-linux-gnu/4.4.7/include/stddef.h\"\n"
    "	.file	4	\"/usr/local/cuda/include/crt/device_runtime.h\"\n"
    "	.file	5	\"/usr/local/cuda/include/host_defines.h\"\n"
    "	.file	6	\"/usr/local/cuda/include/builtin_types.h\"\n"
    "	.file	7	\"/usr/local/cuda/include/device_types.h\"\n"
    "	.file	8	\"/usr/local/cuda/include/hip/driver_types.h\"\n"
    "	.file	9	\"/usr/local/cuda/include/surface_types.h\"\n"
    "	.file	10	\"/usr/local/cuda/include/texture_types.h\"\n"
    "	.file	11	\"/usr/local/cuda/include/hip/hip_vector_types.h\"\n"
    "	.file	12	\"/usr/local/cuda/include/\"\n"
    "	.file	13	\"/usr/local/cuda/include/crt/storage_class.h\"\n"
    "	.file	14	\"matrixMulkernel.cu\"\n"
    "	.file	15	\"/usr/local/cuda/include/common_functions.h\"\n"
    "	.file	16	\"/usr/local/cuda/include/math_functions.h\"\n"
    "	.file	17	\"/usr/local/cuda/include/hip/hip_math_constants.h\"\n"
    "	.file	18	\"/usr/local/cuda/include/hip/device_functions.h\"\n"
    "	.file	19	\"/usr/local/cuda/include/sm_11_atomic_functions.h\"\n"
    "	.file	20	\"/usr/local/cuda/include/sm_12_atomic_functions.h\"\n"
    "	.file	21	\"/usr/local/cuda/include/sm_13_double_functions.h\"\n"
    "	.file	22	\"/usr/local/cuda/include/sm_20_atomic_functions.h\"\n"
    "	.file	23	\"/usr/local/cuda/include/sm_20_intrinsics.h\"\n"
    "	.file	24	\"/usr/local/cuda/include/surface_functions.h\"\n"
    "	.file	25	\"/usr/local/cuda/include/\"\n"
    "	.file	26	\"/usr/local/cuda/include/math_functions_dbl_ptx1.h\"\n"
    "\n"
    "\n"
    "	.entry _Z11matrixMulDSPfS_S_ii (\n"
    "		.param .u64 __cudaparm__Z11matrixMulDSPfS_S_ii_C,\n"
    "		.param .u64 __cudaparm__Z11matrixMulDSPfS_S_ii_A,\n"
    "		.param .u64 __cudaparm__Z11matrixMulDSPfS_S_ii_B,\n"
    "		.param .s32 __cudaparm__Z11matrixMulDSPfS_S_ii_wA,\n"
    "		.param .s32 __cudaparm__Z11matrixMulDSPfS_S_ii_wB)\n"
    "	{\n"
    "	.reg .u32 %r<34>;\n"
    "	.reg .u64 %rd<29>;\n"
    "	.reg .f32 %f<100>;\n"
    "	.reg .pred %p<4>;\n"
    "	.shared .align 4 .b8 __cuda___cuda_local_var_37569_38_non_const_Bs32[4096];\n"
    "	.shared .align 4 .b8 __cuda___cuda_local_var_37568_36_non_const_As4128[4096];\n"
    "	.loc	14	89	0\n"
    "$LDWbegin__Z11matrixMulDSPfS_S_ii:\n"
    "	.loc	14	124	0\n"
    "	cvt.s32.u16 	%r1, %ctaid.x;\n"
    "	mul24.lo.s32 	%r2, %r1, 32;\n"
    "	cvt.s32.u16 	%r3, %ctaid.y;\n"
    "	ld.param.s32 	%r4, [__cudaparm__Z11matrixMulDSPfS_S_ii_wA];\n"
    "	mul.lo.s32 	%r5, %r3, %r4;\n"
    "	mul.lo.s32 	%r6, %r5, 32;\n"
    "	add.s32 	%r7, %r6, %r4;\n"
    "	sub.s32 	%r8, %r7, 1;\n"
    "	cvt.s32.u16 	%r9, %tid.x;\n"
    "	cvt.s32.u16 	%r10, %tid.y;\n"
    "	ld.param.s32 	%r11, [__cudaparm__Z11matrixMulDSPfS_S_ii_wB];\n"
    "	setp.lt.s32 	%p1, %r8, %r6;\n"
    "	@%p1 bra 	$Lt_0_3330;\n"
    "	mov.u64 	%rd1, __cuda___cuda_local_var_37569_38_non_const_Bs32;\n"
    "	mov.u64 	%rd2, __cuda___cuda_local_var_37568_36_non_const_As4128;\n"
    "	ld.param.s32 	%r4, [__cudaparm__Z11matrixMulDSPfS_S_ii_wA];\n"
    "	add.s32 	%r12, %r4, 31;\n"
    "	shr.s32 	%r13, %r12, 31;\n"
    "	mov.s32 	%r14, 31;\n"
    "	and.b32 	%r15, %r13, %r14;\n"
    "	add.s32 	%r16, %r15, %r12;\n"
    "	shr.s32 	%r17, %r16, 5;\n"
    "	ld.param.s32 	%r11, [__cudaparm__Z11matrixMulDSPfS_S_ii_wB];\n"
    "	mul.lo.s32 	%r18, %r10, %r11;\n"
    "	mul.lo.s32 	%r19, %r10, %r4;\n"
    "	cvt.s64.s32 	%rd3, %r9;\n"
    "	cvt.s64.s32 	%rd4, %r10;\n"
    "	add.s32 	%r20, %r19, %r6;\n"
    "	add.s32 	%r21, %r9, %r20;\n"
    "	mul.wide.s32 	%rd5, %r9, 4;\n"
    "	add.u64 	%rd6, %rd1, %rd5;\n"
    "	mul.wide.s32 	%rd7, %r10, 128;\n"
    "	add.u64 	%rd8, %rd2, %rd7;\n"
    "	mul.wide.s32 	%rd9, %r10, 32;\n"
    "	add.u64 	%rd10, %rd3, %rd9;\n"
    "	mul.lo.u64 	%rd11, %rd10, 4;\n"
    "	add.s32 	%r22, %r19, %r8;\n"
    "	mul.lo.s32 	%r23, %r11, 32;\n"
    "	cvt.s64.s32 	%rd12, %r23;\n"
    "	mul.wide.s32 	%rd13, %r23, 4;\n"
    "	add.u64 	%rd14, %rd11, %rd2;\n"
    "	add.u64 	%rd15, %rd11, %rd1;\n"
    "	add.s32 	%r24, %r22, %r9;\n"
    "	ld.param.u64 	%rd16, [__cudaparm__Z11matrixMulDSPfS_S_ii_B];\n"
    "	add.s32 	%r25, %r18, %r2;\n"
    "	add.s32 	%r26, %r9, %r25;\n"
    "	cvt.s64.s32 	%rd17, %r26;\n"
    "	mul.wide.s32 	%rd18, %r26, 4;\n"
    "	add.u64 	%rd19, %rd16, %rd18;\n"
    "	ld.param.u64 	%rd20, [__cudaparm__Z11matrixMulDSPfS_S_ii_A];\n"
    "	cvt.s64.s32 	%rd21, %r21;\n"
    "	mul.wide.s32 	%rd22, %r21, 4;\n"
    "	add.u64 	%rd23, %rd20, %rd22;\n"
    "	mov.f32 	%f1, 0f00000000;     	// 0\n"
    "	mov.s32 	%r27, %r17;\n"
    "$Lt_0_2818:\n"
    " //<loop> Loop body line 124, nesting depth: 1, estimated iterations: unknown\n"
    "	.loc	14	136	0\n"
    "	ld.global.f32 	%f2, [%rd23+0];\n"
    "	st.shared.f32 	[%rd14+0], %f2;\n"
    "	.loc	14	137	0\n"
    "	ld.global.f32 	%f3, [%rd19+0];\n"
    "	st.shared.f32 	[%rd15+0], %f3;\n"
    "	.loc	14	140	0\n"
    "	bar.sync 	0;\n"
    "	.loc	14	150	0\n"
    "	ld.shared.f32 	%f4, [%rd8+0];\n"
    "	ld.shared.f32 	%f5, [%rd6+0];\n"
    "	mad.f32 	%f6, %f4, %f5, %f1;\n"
    "	ld.shared.f32 	%f7, [%rd8+4];\n"
    "	ld.shared.f32 	%f8, [%rd6+128];\n"
    "	mad.f32 	%f9, %f7, %f8, %f6;\n"
    "	ld.shared.f32 	%f10, [%rd8+8];\n"
    "	ld.shared.f32 	%f11, [%rd6+256];\n"
    "	mad.f32 	%f12, %f10, %f11, %f9;\n"
    "	ld.shared.f32 	%f13, [%rd8+12];\n"
    "	ld.shared.f32 	%f14, [%rd6+384];\n"
    "	mad.f32 	%f15, %f13, %f14, %f12;\n"
    "	ld.shared.f32 	%f16, [%rd8+16];\n"
    "	ld.shared.f32 	%f17, [%rd6+512];\n"
    "	mad.f32 	%f18, %f16, %f17, %f15;\n"
    "	ld.shared.f32 	%f19, [%rd8+20];\n"
    "	ld.shared.f32 	%f20, [%rd6+640];\n"
    "	mad.f32 	%f21, %f19, %f20, %f18;\n"
    "	ld.shared.f32 	%f22, [%rd8+24];\n"
    "	ld.shared.f32 	%f23, [%rd6+768];\n"
    "	mad.f32 	%f24, %f22, %f23, %f21;\n"
    "	ld.shared.f32 	%f25, [%rd8+28];\n"
    "	ld.shared.f32 	%f26, [%rd6+896];\n"
    "	mad.f32 	%f27, %f25, %f26, %f24;\n"
    "	ld.shared.f32 	%f28, [%rd8+32];\n"
    "	ld.shared.f32 	%f29, [%rd6+1024];\n"
    "	mad.f32 	%f30, %f28, %f29, %f27;\n"
    "	ld.shared.f32 	%f31, [%rd8+36];\n"
    "	ld.shared.f32 	%f32, [%rd6+1152];\n"
    "	mad.f32 	%f33, %f31, %f32, %f30;\n"
    "	ld.shared.f32 	%f34, [%rd8+40];\n"
    "	ld.shared.f32 	%f35, [%rd6+1280];\n"
    "	mad.f32 	%f36, %f34, %f35, %f33;\n"
    "	ld.shared.f32 	%f37, [%rd8+44];\n"
    "	ld.shared.f32 	%f38, [%rd6+1408];\n"
    "	mad.f32 	%f39, %f37, %f38, %f36;\n"
    "	ld.shared.f32 	%f40, [%rd8+48];\n"
    "	ld.shared.f32 	%f41, [%rd6+1536];\n"
    "	mad.f32 	%f42, %f40, %f41, %f39;\n"
    "	ld.shared.f32 	%f43, [%rd8+52];\n"
    "	ld.shared.f32 	%f44, [%rd6+1664];\n"
    "	mad.f32 	%f45, %f43, %f44, %f42;\n"
    "	ld.shared.f32 	%f46, [%rd8+56];\n"
    "	ld.shared.f32 	%f47, [%rd6+1792];\n"
    "	mad.f32 	%f48, %f46, %f47, %f45;\n"
    "	ld.shared.f32 	%f49, [%rd8+60];\n"
    "	ld.shared.f32 	%f50, [%rd6+1920];\n"
    "	mad.f32 	%f51, %f49, %f50, %f48;\n"
    "	ld.shared.f32 	%f52, [%rd8+64];\n"
    "	ld.shared.f32 	%f53, [%rd6+2048];\n"
    "	mad.f32 	%f54, %f52, %f53, %f51;\n"
    "	ld.shared.f32 	%f55, [%rd8+68];\n"
    "	ld.shared.f32 	%f56, [%rd6+2176];\n"
    "	mad.f32 	%f57, %f55, %f56, %f54;\n"
    "	ld.shared.f32 	%f58, [%rd8+72];\n"
    "	ld.shared.f32 	%f59, [%rd6+2304];\n"
    "	mad.f32 	%f60, %f58, %f59, %f57;\n"
    "	ld.shared.f32 	%f61, [%rd8+76];\n"
    "	ld.shared.f32 	%f62, [%rd6+2432];\n"
    "	mad.f32 	%f63, %f61, %f62, %f60;\n"
    "	ld.shared.f32 	%f64, [%rd8+80];\n"
    "	ld.shared.f32 	%f65, [%rd6+2560];\n"
    "	mad.f32 	%f66, %f64, %f65, %f63;\n"
    "	ld.shared.f32 	%f67, [%rd8+84];\n"
    "	ld.shared.f32 	%f68, [%rd6+2688];\n"
    "	mad.f32 	%f69, %f67, %f68, %f66;\n"
    "	ld.shared.f32 	%f70, [%rd8+88];\n"
    "	ld.shared.f32 	%f71, [%rd6+2816];\n"
    "	mad.f32 	%f72, %f70, %f71, %f69;\n"
    "	ld.shared.f32 	%f73, [%rd8+92];\n"
    "	ld.shared.f32 	%f74, [%rd6+2944];\n"
    "	mad.f32 	%f75, %f73, %f74, %f72;\n"
    "	ld.shared.f32 	%f76, [%rd8+96];\n"
    "	ld.shared.f32 	%f77, [%rd6+3072];\n"
    "	mad.f32 	%f78, %f76, %f77, %f75;\n"
    "	ld.shared.f32 	%f79, [%rd8+100];\n"
    "	ld.shared.f32 	%f80, [%rd6+3200];\n"
    "	mad.f32 	%f81, %f79, %f80, %f78;\n"
    "	ld.shared.f32 	%f82, [%rd8+104];\n"
    "	ld.shared.f32 	%f83, [%rd6+3328];\n"
    "	mad.f32 	%f84, %f82, %f83, %f81;\n"
    "	ld.shared.f32 	%f85, [%rd8+108];\n"
    "	ld.shared.f32 	%f86, [%rd6+3456];\n"
    "	mad.f32 	%f87, %f85, %f86, %f84;\n"
    "	ld.shared.f32 	%f88, [%rd8+112];\n"
    "	ld.shared.f32 	%f89, [%rd6+3584];\n"
    "	mad.f32 	%f90, %f88, %f89, %f87;\n"
    "	ld.shared.f32 	%f91, [%rd8+116];\n"
    "	ld.shared.f32 	%f92, [%rd6+3712];\n"
    "	mad.f32 	%f93, %f91, %f92, %f90;\n"
    "	ld.shared.f32 	%f94, [%rd8+120];\n"
    "	ld.shared.f32 	%f95, [%rd6+3840];\n"
    "	mad.f32 	%f96, %f94, %f95, %f93;\n"
    "	ld.shared.f32 	%f97, [%rd8+124];\n"
    "	ld.shared.f32 	%f98, [%rd6+3968];\n"
    "	mad.f32 	%f1, %f97, %f98, %f96;\n"
    "	.loc	14	155	0\n"
    "	bar.sync 	0;\n"
    "	.loc	14	124	0\n"
    "	add.u64 	%rd19, %rd13, %rd19;\n"
    "	add.s32 	%r21, %r21, 32;\n"
    "	add.u64 	%rd23, %rd23, 128;\n"
    "	setp.le.s32 	%p2, %r21, %r24;\n"
    "	@%p2 bra 	$Lt_0_2818;\n"
    "	bra.uni 	$Lt_0_2306;\n"
    "$Lt_0_3330:\n"
    "	ld.param.s32 	%r11, [__cudaparm__Z11matrixMulDSPfS_S_ii_wB];\n"
    "	mul.lo.s32 	%r18, %r10, %r11;\n"
    "	mov.f32 	%f1, 0f00000000;     	// 0\n"
    "$Lt_0_2306:\n"
    "	.loc	14	161	0\n"
    "	ld.param.u64 	%rd24, [__cudaparm__Z11matrixMulDSPfS_S_ii_C];\n"
    "	mul.lo.s32 	%r28, %r11, %r3;\n"
    "	add.s32 	%r29, %r1, %r28;\n"
    "	mul.lo.s32 	%r30, %r29, 32;\n"
    "	add.s32 	%r31, %r18, %r30;\n"
    "	add.s32 	%r32, %r9, %r31;\n"
    "	cvt.s64.s32 	%rd25, %r32;\n"
    "	mul.wide.s32 	%rd26, %r32, 4;\n"
    "	add.u64 	%rd27, %rd24, %rd26;\n"
    "	st.global.f32 	[%rd27+0], %f1;\n"
    "	.loc	14	162	0\n"
    "	exit;\n"
    "$LDWend__Z11matrixMulDSPfS_S_ii:\n"
    "	} // _Z11matrixMulDSPfS_S_ii\n"
    "\n";
#pragma dscuda endofptx
#pragma begin dscuda.h
#ifndef _DSCUDA_H
#define _DSCUDA_H

#include <hip/hip_runtime_api.h>
#include <cutil.h>
#include <builtin_types.h>
#include <hip/driver_types.h>
#include <hip/hip_texture_types.h>
#include <texture_types.h>
#pragma begin dscudarpc.h


#ifndef _DSCUDARPC_H_RPCGEN
#define _DSCUDARPC_H_RPCGEN

#include <rpc/rpc.h>


#ifdef __cplusplus
extern "C" {
#endif


typedef u_quad_t RCadr;

typedef u_quad_t RCstream;

typedef u_quad_t RCevent;

typedef u_quad_t RCipaddr;

typedef u_int RCsize;

typedef u_int RCerror;

typedef struct {
	u_int RCbuf_len;
	char *RCbuf_val;
} RCbuf;

typedef u_int RCchannelformat;

typedef u_long RCpid;

struct RCchanneldesc_t {
	RCchannelformat f;
	int w;
	int x;
	int y;
	int z;
};
typedef struct RCchanneldesc_t RCchanneldesc_t;

typedef RCchanneldesc_t RCchanneldesc;

struct RCtexture_t {
	int normalized;
	int filterMode;
	int addressMode[3];
	RCchannelformat f;
	int w;
	int x;
	int y;
	int z;
};
typedef struct RCtexture_t RCtexture_t;

typedef RCtexture_t RCtexture;

struct RCfuncattr_t {
	int binaryVersion;
	RCsize constSizeBytes;
	RCsize localSizeBytes;
	int maxThreadsPerBlock;
	int numRegs;
	int ptxVersion;
	RCsize sharedSizeBytes;
};
typedef struct RCfuncattr_t RCfuncattr_t;

typedef RCfuncattr_t RCfuncattr;

enum RCargType {
	dscudaArgTypeP = 0,
	dscudaArgTypeI = 1,
	dscudaArgTypeF = 2,
	dscudaArgTypeV = 3,
};
typedef enum RCargType RCargType;

struct RCargVal {
	RCargType type;
	union {
		RCadr address;
		u_int valuei;
		float valuef;
		char valuev[64];
	} RCargVal_u;
};
typedef struct RCargVal RCargVal;

struct RCarg {
	RCargVal val;
	u_int offset;
	u_int size;
};
typedef struct RCarg RCarg;

typedef struct {
	u_int RCargs_len;
	RCarg *RCargs_val;
} RCargs;

struct dscudaResult {
	RCerror err;
};
typedef struct dscudaResult dscudaResult;

struct dscudaThreadGetLimitResult {
	RCerror err;
	RCsize value;
};
typedef struct dscudaThreadGetLimitResult dscudaThreadGetLimitResult;

struct dscudaThreadGetCacheConfigResult {
	RCerror err;
	int cacheConfig;
};
typedef struct dscudaThreadGetCacheConfigResult dscudaThreadGetCacheConfigResult;

struct dscudaMallocResult {
	RCerror err;
	RCadr devAdr;
};
typedef struct dscudaMallocResult dscudaMallocResult;

struct dscudaHostAllocResult {
	RCerror err;
	RCadr pHost;
};
typedef struct dscudaHostAllocResult dscudaHostAllocResult;

struct dscudaMallocHostResult {
	RCerror err;
	RCadr ptr;
};
typedef struct dscudaMallocHostResult dscudaMallocHostResult;

struct dscudaMallocArrayResult {
	RCerror err;
	RCadr array;
};
typedef struct dscudaMallocArrayResult dscudaMallocArrayResult;

struct dscudaMallocPitchResult {
	RCerror err;
	RCadr devPtr;
	RCsize pitch;
};
typedef struct dscudaMallocPitchResult dscudaMallocPitchResult;

struct dscudaMemcpyD2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyD2HResult dscudaMemcpyD2HResult;

struct dscudaMemcpyH2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyH2HResult dscudaMemcpyH2HResult;

struct dscudaMemcpyToArrayD2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyToArrayD2HResult dscudaMemcpyToArrayD2HResult;

struct dscudaMemcpyToArrayH2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyToArrayH2HResult dscudaMemcpyToArrayH2HResult;

struct dscudaMemcpy2DToArrayD2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpy2DToArrayD2HResult dscudaMemcpy2DToArrayD2HResult;

struct dscudaMemcpy2DToArrayH2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpy2DToArrayH2HResult dscudaMemcpy2DToArrayH2HResult;

struct dscudaMemcpy2DD2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpy2DD2HResult dscudaMemcpy2DD2HResult;

struct dscudaMemcpy2DH2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpy2DH2HResult dscudaMemcpy2DH2HResult;

struct dscudaGetDeviceResult {
	RCerror err;
	int device;
};
typedef struct dscudaGetDeviceResult dscudaGetDeviceResult;

struct dscudaGetDeviceCountResult {
	RCerror err;
	int count;
};
typedef struct dscudaGetDeviceCountResult dscudaGetDeviceCountResult;

struct dscudaGetDevicePropertiesResult {
	RCerror err;
	RCbuf prop;
};
typedef struct dscudaGetDevicePropertiesResult dscudaGetDevicePropertiesResult;

struct dscudaDriverGetVersionResult {
	RCerror err;
	int ver;
};
typedef struct dscudaDriverGetVersionResult dscudaDriverGetVersionResult;

struct dscudaRuntimeGetVersionResult {
	RCerror err;
	int ver;
};
typedef struct dscudaRuntimeGetVersionResult dscudaRuntimeGetVersionResult;

struct dscudaGetErrorStringResult {
	char *errmsg;
};
typedef struct dscudaGetErrorStringResult dscudaGetErrorStringResult;

struct dscudaCreateChannelDescResult {
	int x;
	int y;
	int z;
	int w;
	RCchannelformat f;
};
typedef struct dscudaCreateChannelDescResult dscudaCreateChannelDescResult;

struct dscudaGetChannelDescResult {
	RCerror err;
	int x;
	int y;
	int z;
	int w;
	RCchannelformat f;
};
typedef struct dscudaGetChannelDescResult dscudaGetChannelDescResult;

struct dscudaChooseDeviceResult {
	RCerror err;
	int device;
};
typedef struct dscudaChooseDeviceResult dscudaChooseDeviceResult;

struct dscudaMemcpyAsyncD2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyAsyncD2HResult dscudaMemcpyAsyncD2HResult;

struct dscudaMemcpyAsyncH2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyAsyncH2HResult dscudaMemcpyAsyncH2HResult;

struct dscudaMemcpyFromSymbolD2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyFromSymbolD2HResult dscudaMemcpyFromSymbolD2HResult;

struct dscudaMemcpyFromSymbolAsyncD2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyFromSymbolAsyncD2HResult dscudaMemcpyFromSymbolAsyncD2HResult;

struct dscudaStreamCreateResult {
	RCerror err;
	RCadr stream;
};
typedef struct dscudaStreamCreateResult dscudaStreamCreateResult;

struct dscudaEventCreateResult {
	RCerror err;
	RCadr event;
};
typedef struct dscudaEventCreateResult dscudaEventCreateResult;

struct dscudaEventElapsedTimeResult {
	RCerror err;
	float ms;
};
typedef struct dscudaEventElapsedTimeResult dscudaEventElapsedTimeResult;

struct dscudaHostGetDevicePointerResult {
	RCerror err;
	RCadr pDevice;
};
typedef struct dscudaHostGetDevicePointerResult dscudaHostGetDevicePointerResult;

struct dscudaHostGetFlagsResult {
	RCerror err;
	u_int flags;
};
typedef struct dscudaHostGetFlagsResult dscudaHostGetFlagsResult;

struct dscudaLoadModuleResult {
	u_int id;
};
typedef struct dscudaLoadModuleResult dscudaLoadModuleResult;

struct dscudaFuncGetAttributesResult {
	RCerror err;
	RCfuncattr attr;
};
typedef struct dscudaFuncGetAttributesResult dscudaFuncGetAttributesResult;

struct dscudaBindTextureResult {
	RCerror err;
	RCsize offset;
};
typedef struct dscudaBindTextureResult dscudaBindTextureResult;

struct dscudaBindTexture2DResult {
	RCerror err;
	RCsize offset;
};
typedef struct dscudaBindTexture2DResult dscudaBindTexture2DResult;

struct dscufftResult {
	RCerror err;
};
typedef struct dscufftResult dscufftResult;

struct dscufftPlanResult {
	RCerror err;
	u_int plan;
};
typedef struct dscufftPlanResult dscufftPlanResult;

struct dscublasResult {
	RCerror err;
	u_int stat;
};
typedef struct dscublasResult dscublasResult;

struct dscublasCreateResult {
	RCerror err;
	u_int stat;
	RCadr handle;
};
typedef struct dscublasCreateResult dscublasCreateResult;

struct dscublasGetVectorResult {
	RCerror err;
	u_int stat;
	RCbuf y;
};
typedef struct dscublasGetVectorResult dscublasGetVectorResult;

struct RCdim3 {
	u_int x;
	u_int y;
	u_int z;
};
typedef struct RCdim3 RCdim3;

struct dscudathreadsetlimitid_1_argument {
	int limit;
	RCsize value;
};
typedef struct dscudathreadsetlimitid_1_argument dscudathreadsetlimitid_1_argument;

struct dscudastreamwaiteventid_1_argument {
	RCstream stream;
	RCevent event;
	u_int flags;
};
typedef struct dscudastreamwaiteventid_1_argument dscudastreamwaiteventid_1_argument;

struct dscudaeventelapsedtimeid_1_argument {
	RCevent start;
	RCevent end;
};
typedef struct dscudaeventelapsedtimeid_1_argument dscudaeventelapsedtimeid_1_argument;

struct dscudaeventrecordid_1_argument {
	RCevent event;
	RCstream stream;
};
typedef struct dscudaeventrecordid_1_argument dscudaeventrecordid_1_argument;

struct dscudalaunchkernelid_1_argument {
	int moduleid;
	int kid;
	char *kname;
	RCdim3 gdim;
	RCdim3 bdim;
	RCsize smemsize;
	RCstream stream;
	RCargs args;
};
typedef struct dscudalaunchkernelid_1_argument dscudalaunchkernelid_1_argument;

struct dscudaloadmoduleid_1_argument {
	RCipaddr ipaddr;
	RCpid pid;
	char *mname;
	char *image;
};
typedef struct dscudaloadmoduleid_1_argument dscudaloadmoduleid_1_argument;

struct dscudafuncgetattributesid_1_argument {
	int moduleid;
	char *kname;
};
typedef struct dscudafuncgetattributesid_1_argument dscudafuncgetattributesid_1_argument;

struct dscudamemcpyh2hid_1_argument {
	RCadr dst;
	RCbuf src;
	RCsize count;
};
typedef struct dscudamemcpyh2hid_1_argument dscudamemcpyh2hid_1_argument;

struct dscudamemcpyh2did_1_argument {
	RCadr dst;
	RCbuf src;
	RCsize count;
};
typedef struct dscudamemcpyh2did_1_argument dscudamemcpyh2did_1_argument;

struct dscudamemcpyd2hid_1_argument {
	RCadr src;
	RCsize count;
};
typedef struct dscudamemcpyd2hid_1_argument dscudamemcpyd2hid_1_argument;

struct dscudamemcpyd2did_1_argument {
	RCadr dst;
	RCadr src;
	RCsize count;
};
typedef struct dscudamemcpyd2did_1_argument dscudamemcpyd2did_1_argument;

struct dscudamemcpyasynch2hid_1_argument {
	RCadr dst;
	RCbuf src;
	RCsize count;
	RCstream stream;
};
typedef struct dscudamemcpyasynch2hid_1_argument dscudamemcpyasynch2hid_1_argument;

struct dscudamemcpyasynch2did_1_argument {
	RCadr dst;
	RCbuf src;
	RCsize count;
	RCstream stream;
};
typedef struct dscudamemcpyasynch2did_1_argument dscudamemcpyasynch2did_1_argument;

struct dscudamemcpyasyncd2hid_1_argument {
	RCadr src;
	RCsize count;
	RCstream stream;
};
typedef struct dscudamemcpyasyncd2hid_1_argument dscudamemcpyasyncd2hid_1_argument;

struct dscudamemcpyasyncd2did_1_argument {
	RCadr dst;
	RCadr src;
	RCsize count;
	RCstream stream;
};
typedef struct dscudamemcpyasyncd2did_1_argument dscudamemcpyasyncd2did_1_argument;

struct dscudamemcpytosymbolh2did_1_argument {
	int moduleid;
	char *symbol;
	RCbuf src;
	RCsize count;
	RCsize offset;
};
typedef struct dscudamemcpytosymbolh2did_1_argument dscudamemcpytosymbolh2did_1_argument;

struct dscudamemcpytosymbold2did_1_argument {
	int moduleid;
	char *symbol;
	RCadr src;
	RCsize count;
	RCsize offset;
};
typedef struct dscudamemcpytosymbold2did_1_argument dscudamemcpytosymbold2did_1_argument;

struct dscudamemcpyfromsymbold2hid_1_argument {
	int moduleid;
	char *symbol;
	RCsize count;
	RCsize offset;
};
typedef struct dscudamemcpyfromsymbold2hid_1_argument dscudamemcpyfromsymbold2hid_1_argument;

struct dscudamemcpyfromsymbold2did_1_argument {
	int moduleid;
	RCadr dst;
	char *symbol;
	RCsize count;
	RCsize offset;
};
typedef struct dscudamemcpyfromsymbold2did_1_argument dscudamemcpyfromsymbold2did_1_argument;

struct dscudamemsetid_1_argument {
	RCadr dst;
	int value;
	RCsize count;
};
typedef struct dscudamemsetid_1_argument dscudamemsetid_1_argument;

struct dscudahostallocid_1_argument {
	RCsize size;
	u_int flags;
};
typedef struct dscudahostallocid_1_argument dscudahostallocid_1_argument;

struct dscudahostgetdevicepointerid_1_argument {
	RCadr pHost;
	u_int flags;
};
typedef struct dscudahostgetdevicepointerid_1_argument dscudahostgetdevicepointerid_1_argument;

struct dscudamallocarrayid_1_argument {
	RCchanneldesc desc;
	RCsize width;
	RCsize height;
	u_int flags;
};
typedef struct dscudamallocarrayid_1_argument dscudamallocarrayid_1_argument;

struct dscudamemcpytoarrayh2hid_1_argument {
	RCadr dst;
	RCsize wOffset;
	RCsize hOffset;
	RCbuf src;
	RCsize count;
};
typedef struct dscudamemcpytoarrayh2hid_1_argument dscudamemcpytoarrayh2hid_1_argument;

struct dscudamemcpytoarrayh2did_1_argument {
	RCadr dst;
	RCsize wOffset;
	RCsize hOffset;
	RCbuf src;
	RCsize count;
};
typedef struct dscudamemcpytoarrayh2did_1_argument dscudamemcpytoarrayh2did_1_argument;

struct dscudamemcpytoarrayd2hid_1_argument {
	RCsize wOffset;
	RCsize hOffset;
	RCadr src;
	RCsize count;
};
typedef struct dscudamemcpytoarrayd2hid_1_argument dscudamemcpytoarrayd2hid_1_argument;

struct dscudamemcpytoarrayd2did_1_argument {
	RCadr dst;
	RCsize wOffset;
	RCsize hOffset;
	RCadr src;
	RCsize count;
};
typedef struct dscudamemcpytoarrayd2did_1_argument dscudamemcpytoarrayd2did_1_argument;

struct dscudamallocpitchid_1_argument {
	RCsize width;
	RCsize height;
};
typedef struct dscudamallocpitchid_1_argument dscudamallocpitchid_1_argument;

struct dscudamemcpy2dtoarrayh2hid_1_argument {
	RCadr dst;
	RCsize wOffset;
	RCsize hOffset;
	RCbuf src;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dtoarrayh2hid_1_argument dscudamemcpy2dtoarrayh2hid_1_argument;

struct dscudamemcpy2dtoarrayh2did_1_argument {
	RCadr dst;
	RCsize wOffset;
	RCsize hOffset;
	RCbuf srcbuf;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dtoarrayh2did_1_argument dscudamemcpy2dtoarrayh2did_1_argument;

struct dscudamemcpy2dtoarrayd2hid_1_argument {
	RCsize wOffset;
	RCsize hOffset;
	RCadr src;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dtoarrayd2hid_1_argument dscudamemcpy2dtoarrayd2hid_1_argument;

struct dscudamemcpy2dtoarrayd2did_1_argument {
	RCadr dst;
	RCsize wOffset;
	RCsize hOffset;
	RCadr src;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dtoarrayd2did_1_argument dscudamemcpy2dtoarrayd2did_1_argument;

struct dscudamemcpy2dh2hid_1_argument {
	RCadr dst;
	RCsize dpitch;
	RCbuf src;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dh2hid_1_argument dscudamemcpy2dh2hid_1_argument;

struct dscudamemcpy2dh2did_1_argument {
	RCadr dst;
	RCsize dpitch;
	RCbuf src;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dh2did_1_argument dscudamemcpy2dh2did_1_argument;

struct dscudamemcpy2dd2hid_1_argument {
	RCsize dpitch;
	RCadr src;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dd2hid_1_argument dscudamemcpy2dd2hid_1_argument;

struct dscudamemcpy2dd2did_1_argument {
	RCadr dst;
	RCsize dpitch;
	RCadr src;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dd2did_1_argument dscudamemcpy2dd2did_1_argument;

struct dscudamemset2did_1_argument {
	RCadr dst;
	RCsize pitch;
	int value;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemset2did_1_argument dscudamemset2did_1_argument;

struct dscudamemcpytosymbolasynch2did_1_argument {
	int moduleid;
	char *symbol;
	RCbuf src;
	RCsize count;
	RCsize offset;
	RCstream stream;
};
typedef struct dscudamemcpytosymbolasynch2did_1_argument dscudamemcpytosymbolasynch2did_1_argument;

struct dscudamemcpytosymbolasyncd2did_1_argument {
	int moduleid;
	char *symbol;
	RCadr src;
	RCsize count;
	RCsize offset;
	RCstream stream;
};
typedef struct dscudamemcpytosymbolasyncd2did_1_argument dscudamemcpytosymbolasyncd2did_1_argument;

struct dscudamemcpyfromsymbolasyncd2hid_1_argument {
	int moduleid;
	char *symbol;
	RCsize count;
	RCsize offset;
	RCstream stream;
};
typedef struct dscudamemcpyfromsymbolasyncd2hid_1_argument dscudamemcpyfromsymbolasyncd2hid_1_argument;

struct dscudamemcpyfromsymbolasyncd2did_1_argument {
	int moduleid;
	RCadr dst;
	char *symbol;
	RCsize count;
	RCsize offset;
	RCstream stream;
};
typedef struct dscudamemcpyfromsymbolasyncd2did_1_argument dscudamemcpyfromsymbolasyncd2did_1_argument;

struct dscudacreatechanneldescid_1_argument {
	int x;
	int y;
	int z;
	int w;
	RCchannelformat f;
};
typedef struct dscudacreatechanneldescid_1_argument dscudacreatechanneldescid_1_argument;

struct dscudabindtextureid_1_argument {
	int moduleid;
	char *texname;
	RCadr devPtr;
	RCsize size;
	RCtexture texbuf;
};
typedef struct dscudabindtextureid_1_argument dscudabindtextureid_1_argument;

struct dscudabindtexture2did_1_argument {
	int moduleid;
	char *texname;
	RCadr devPtr;
	RCsize width;
	RCsize height;
	RCsize pitch;
	RCtexture texbuf;
};
typedef struct dscudabindtexture2did_1_argument dscudabindtexture2did_1_argument;

struct dscudabindtexturetoarrayid_1_argument {
	int moduleid;
	char *texname;
	RCadr array;
	RCtexture texbuf;
};
typedef struct dscudabindtexturetoarrayid_1_argument dscudabindtexturetoarrayid_1_argument;

struct dscufftplan3did_1_argument {
	int nx;
	int ny;
	int nz;
	u_int type;
};
typedef struct dscufftplan3did_1_argument dscufftplan3did_1_argument;

struct dscufftexecc2cid_1_argument {
	u_int plan;
	RCadr idata;
	RCadr odata;
	int direction;
};
typedef struct dscufftexecc2cid_1_argument dscufftexecc2cid_1_argument;

#define DSCUDA_PROG 60000
#define DSCUDA_VER 1

#if defined(__STDC__) || defined(__cplusplus)
#define dscudaThreadExitId 100
extern  dscudaResult * dscudathreadexitid_1(CLIENT *);
extern  dscudaResult * dscudathreadexitid_1_svc(struct svc_req *);
#define dscudaThreadSynchronizeId 101
extern  dscudaResult * dscudathreadsynchronizeid_1(CLIENT *);
extern  dscudaResult * dscudathreadsynchronizeid_1_svc(struct svc_req *);
#define dscudaThreadSetLimitId 102
extern  dscudaResult * dscudathreadsetlimitid_1(int , RCsize , CLIENT *);
extern  dscudaResult * dscudathreadsetlimitid_1_svc(int , RCsize , struct svc_req *);
#define dscudaThreadGetLimitId 103
extern  dscudaThreadGetLimitResult * dscudathreadgetlimitid_1(int , CLIENT *);
extern  dscudaThreadGetLimitResult * dscudathreadgetlimitid_1_svc(int , struct svc_req *);
#define dscudaThreadSetCacheConfigId 104
extern  dscudaResult * dscudathreadsetcacheconfigid_1(int , CLIENT *);
extern  dscudaResult * dscudathreadsetcacheconfigid_1_svc(int , struct svc_req *);
#define dscudaThreadGetCacheConfigId 105
extern  dscudaThreadGetCacheConfigResult * dscudathreadgetcacheconfigid_1(CLIENT *);
extern  dscudaThreadGetCacheConfigResult * dscudathreadgetcacheconfigid_1_svc(struct svc_req *);
#define dscudaGetLastErrorId 200
extern  dscudaResult * dscudagetlasterrorid_1(CLIENT *);
extern  dscudaResult * dscudagetlasterrorid_1_svc(struct svc_req *);
#define dscudaPeekAtLastErrorId 201
extern  dscudaResult * dscudapeekatlasterrorid_1(CLIENT *);
extern  dscudaResult * dscudapeekatlasterrorid_1_svc(struct svc_req *);
#define dscudaGetErrorStringId 202
extern  dscudaGetErrorStringResult * dscudageterrorstringid_1(int , CLIENT *);
extern  dscudaGetErrorStringResult * dscudageterrorstringid_1_svc(int , struct svc_req *);
#define dscudaGetDeviceId 300
extern  dscudaGetDeviceResult * dscudagetdeviceid_1(CLIENT *);
extern  dscudaGetDeviceResult * dscudagetdeviceid_1_svc(struct svc_req *);
#define dscudaGetDeviceCountId 301
extern  dscudaGetDeviceCountResult * dscudagetdevicecountid_1(CLIENT *);
extern  dscudaGetDeviceCountResult * dscudagetdevicecountid_1_svc(struct svc_req *);
#define dscudaGetDevicePropertiesId 302
extern  dscudaGetDevicePropertiesResult * dscudagetdevicepropertiesid_1(int , CLIENT *);
extern  dscudaGetDevicePropertiesResult * dscudagetdevicepropertiesid_1_svc(int , struct svc_req *);
#define dscudaDriverGetVersionId 303
extern  dscudaDriverGetVersionResult * dscudadrivergetversionid_1(CLIENT *);
extern  dscudaDriverGetVersionResult * dscudadrivergetversionid_1_svc(struct svc_req *);
#define dscudaRuntimeGetVersionId 304
extern  dscudaRuntimeGetVersionResult * dscudaruntimegetversionid_1(CLIENT *);
extern  dscudaRuntimeGetVersionResult * dscudaruntimegetversionid_1_svc(struct svc_req *);
#define dscudaSetDeviceId 305
extern  dscudaResult * dscudasetdeviceid_1(int , CLIENT *);
extern  dscudaResult * dscudasetdeviceid_1_svc(int , struct svc_req *);
#define dscudaSetDeviceFlagsId 306
extern  dscudaResult * dscudasetdeviceflagsid_1(u_int , CLIENT *);
extern  dscudaResult * dscudasetdeviceflagsid_1_svc(u_int , struct svc_req *);
#define dscudaChooseDeviceId 307
extern  dscudaChooseDeviceResult * dscudachoosedeviceid_1(RCbuf , CLIENT *);
extern  dscudaChooseDeviceResult * dscudachoosedeviceid_1_svc(RCbuf , struct svc_req *);
#define dscudaDeviceSynchronize 308
extern  dscudaResult * dscudadevicesynchronize_1(CLIENT *);
extern  dscudaResult * dscudadevicesynchronize_1_svc(struct svc_req *);
#define dscudaDeviceReset 309
extern  dscudaResult * dscudadevicereset_1(CLIENT *);
extern  dscudaResult * dscudadevicereset_1_svc(struct svc_req *);
#define dscudaStreamCreateId 400
extern  dscudaStreamCreateResult * dscudastreamcreateid_1(CLIENT *);
extern  dscudaStreamCreateResult * dscudastreamcreateid_1_svc(struct svc_req *);
#define dscudaStreamDestroyId 401
extern  dscudaResult * dscudastreamdestroyid_1(RCstream , CLIENT *);
extern  dscudaResult * dscudastreamdestroyid_1_svc(RCstream , struct svc_req *);
#define dscudaStreamSynchronizeId 402
extern  dscudaResult * dscudastreamsynchronizeid_1(RCstream , CLIENT *);
extern  dscudaResult * dscudastreamsynchronizeid_1_svc(RCstream , struct svc_req *);
#define dscudaStreamQueryId 403
extern  dscudaResult * dscudastreamqueryid_1(RCstream , CLIENT *);
extern  dscudaResult * dscudastreamqueryid_1_svc(RCstream , struct svc_req *);
#define dscudaStreamWaitEventId 404
extern  dscudaResult * dscudastreamwaiteventid_1(RCstream , RCevent , u_int , CLIENT *);
extern  dscudaResult * dscudastreamwaiteventid_1_svc(RCstream , RCevent , u_int , struct svc_req *);
#define dscudaEventCreateId 500
extern  dscudaEventCreateResult * dscudaeventcreateid_1(CLIENT *);
extern  dscudaEventCreateResult * dscudaeventcreateid_1_svc(struct svc_req *);
#define dscudaEventCreateWithFlagsId 501
extern  dscudaEventCreateResult * dscudaeventcreatewithflagsid_1(u_int , CLIENT *);
extern  dscudaEventCreateResult * dscudaeventcreatewithflagsid_1_svc(u_int , struct svc_req *);
#define dscudaEventDestroyId 502
extern  dscudaResult * dscudaeventdestroyid_1(RCevent , CLIENT *);
extern  dscudaResult * dscudaeventdestroyid_1_svc(RCevent , struct svc_req *);
#define dscudaEventElapsedTimeId 503
extern  dscudaEventElapsedTimeResult * dscudaeventelapsedtimeid_1(RCevent , RCevent , CLIENT *);
extern  dscudaEventElapsedTimeResult * dscudaeventelapsedtimeid_1_svc(RCevent , RCevent , struct svc_req *);
#define dscudaEventRecordId 504
extern  dscudaResult * dscudaeventrecordid_1(RCevent , RCstream , CLIENT *);
extern  dscudaResult * dscudaeventrecordid_1_svc(RCevent , RCstream , struct svc_req *);
#define dscudaEventSynchronizeId 505
extern  dscudaResult * dscudaeventsynchronizeid_1(RCevent , CLIENT *);
extern  dscudaResult * dscudaeventsynchronizeid_1_svc(RCevent , struct svc_req *);
#define dscudaEventQueryId 506
extern  dscudaResult * dscudaeventqueryid_1(RCevent , CLIENT *);
extern  dscudaResult * dscudaeventqueryid_1_svc(RCevent , struct svc_req *);
#define dscudaLaunchKernelId 600
extern  void * dscudalaunchkernelid_1(int , int , char *, RCdim3 , RCdim3 , RCsize , RCstream , RCargs , CLIENT *);
extern  void * dscudalaunchkernelid_1_svc(int , int , char *, RCdim3 , RCdim3 , RCsize , RCstream , RCargs , struct svc_req *);
#define dscudaLoadModuleId 601
extern  dscudaLoadModuleResult * dscudaloadmoduleid_1(RCipaddr , RCpid , char *, char *, CLIENT *);
extern  dscudaLoadModuleResult * dscudaloadmoduleid_1_svc(RCipaddr , RCpid , char *, char *, struct svc_req *);
#define dscudaFuncGetAttributesId 602
extern  dscudaFuncGetAttributesResult * dscudafuncgetattributesid_1(int , char *, CLIENT *);
extern  dscudaFuncGetAttributesResult * dscudafuncgetattributesid_1_svc(int , char *, struct svc_req *);
#define dscudaMallocId 700
extern  dscudaMallocResult * dscudamallocid_1(RCsize , CLIENT *);
extern  dscudaMallocResult * dscudamallocid_1_svc(RCsize , struct svc_req *);
#define dscudaFreeId 701
extern  dscudaResult * dscudafreeid_1(RCadr , CLIENT *);
extern  dscudaResult * dscudafreeid_1_svc(RCadr , struct svc_req *);
#define dscudaMemcpyH2HId 702
extern  dscudaMemcpyH2HResult * dscudamemcpyh2hid_1(RCadr , RCbuf , RCsize , CLIENT *);
extern  dscudaMemcpyH2HResult * dscudamemcpyh2hid_1_svc(RCadr , RCbuf , RCsize , struct svc_req *);
#define dscudaMemcpyH2DId 703
extern  dscudaResult * dscudamemcpyh2did_1(RCadr , RCbuf , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpyh2did_1_svc(RCadr , RCbuf , RCsize , struct svc_req *);
#define dscudaMemcpyD2HId 704
extern  dscudaMemcpyD2HResult * dscudamemcpyd2hid_1(RCadr , RCsize , CLIENT *);
extern  dscudaMemcpyD2HResult * dscudamemcpyd2hid_1_svc(RCadr , RCsize , struct svc_req *);
#define dscudaMemcpyD2DId 705
extern  dscudaResult * dscudamemcpyd2did_1(RCadr , RCadr , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpyd2did_1_svc(RCadr , RCadr , RCsize , struct svc_req *);
#define dscudaMemcpyAsyncH2HId 706
extern  dscudaMemcpyAsyncH2HResult * dscudamemcpyasynch2hid_1(RCadr , RCbuf , RCsize , RCstream , CLIENT *);
extern  dscudaMemcpyAsyncH2HResult * dscudamemcpyasynch2hid_1_svc(RCadr , RCbuf , RCsize , RCstream , struct svc_req *);
#define dscudaMemcpyAsyncH2DId 707
extern  dscudaResult * dscudamemcpyasynch2did_1(RCadr , RCbuf , RCsize , RCstream , CLIENT *);
extern  dscudaResult * dscudamemcpyasynch2did_1_svc(RCadr , RCbuf , RCsize , RCstream , struct svc_req *);
#define dscudaMemcpyAsyncD2HId 708
extern  dscudaMemcpyAsyncD2HResult * dscudamemcpyasyncd2hid_1(RCadr , RCsize , RCstream , CLIENT *);
extern  dscudaMemcpyAsyncD2HResult * dscudamemcpyasyncd2hid_1_svc(RCadr , RCsize , RCstream , struct svc_req *);
#define dscudaMemcpyAsyncD2DId 709
extern  dscudaResult * dscudamemcpyasyncd2did_1(RCadr , RCadr , RCsize , RCstream , CLIENT *);
extern  dscudaResult * dscudamemcpyasyncd2did_1_svc(RCadr , RCadr , RCsize , RCstream , struct svc_req *);
#define dscudaMemcpyToSymbolH2DId 710
extern  dscudaResult * dscudamemcpytosymbolh2did_1(int , char *, RCbuf , RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpytosymbolh2did_1_svc(int , char *, RCbuf , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpyToSymbolD2DId 711
extern  dscudaResult * dscudamemcpytosymbold2did_1(int , char *, RCadr , RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpytosymbold2did_1_svc(int , char *, RCadr , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpyFromSymbolD2HId 712
extern  dscudaMemcpyFromSymbolD2HResult * dscudamemcpyfromsymbold2hid_1(int , char *, RCsize , RCsize , CLIENT *);
extern  dscudaMemcpyFromSymbolD2HResult * dscudamemcpyfromsymbold2hid_1_svc(int , char *, RCsize , RCsize , struct svc_req *);
#define dscudaMemcpyFromSymbolD2DId 713
extern  dscudaResult * dscudamemcpyfromsymbold2did_1(int , RCadr , char *, RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpyfromsymbold2did_1_svc(int , RCadr , char *, RCsize , RCsize , struct svc_req *);
#define dscudaMemsetId 714
extern  dscudaResult * dscudamemsetid_1(RCadr , int , RCsize , CLIENT *);
extern  dscudaResult * dscudamemsetid_1_svc(RCadr , int , RCsize , struct svc_req *);
#define dscudaHostAllocId 715
extern  dscudaHostAllocResult * dscudahostallocid_1(RCsize , u_int , CLIENT *);
extern  dscudaHostAllocResult * dscudahostallocid_1_svc(RCsize , u_int , struct svc_req *);
#define dscudaMallocHostId 716
extern  dscudaMallocHostResult * dscudamallochostid_1(RCsize , CLIENT *);
extern  dscudaMallocHostResult * dscudamallochostid_1_svc(RCsize , struct svc_req *);
#define dscudaFreeHostId 717
extern  dscudaResult * dscudafreehostid_1(RCadr , CLIENT *);
extern  dscudaResult * dscudafreehostid_1_svc(RCadr , struct svc_req *);
#define dscudaHostGetDevicePointerId 718
extern  dscudaHostGetDevicePointerResult * dscudahostgetdevicepointerid_1(RCadr , u_int , CLIENT *);
extern  dscudaHostGetDevicePointerResult * dscudahostgetdevicepointerid_1_svc(RCadr , u_int , struct svc_req *);
#define dscudaHostGetFlagsID 719
extern  dscudaHostGetFlagsResult * dscudahostgetflagsid_1(RCadr , CLIENT *);
extern  dscudaHostGetFlagsResult * dscudahostgetflagsid_1_svc(RCadr , struct svc_req *);
#define dscudaMallocArrayId 720
extern  dscudaMallocArrayResult * dscudamallocarrayid_1(RCchanneldesc , RCsize , RCsize , u_int , CLIENT *);
extern  dscudaMallocArrayResult * dscudamallocarrayid_1_svc(RCchanneldesc , RCsize , RCsize , u_int , struct svc_req *);
#define dscudaFreeArrayId 721
extern  dscudaResult * dscudafreearrayid_1(RCadr , CLIENT *);
extern  dscudaResult * dscudafreearrayid_1_svc(RCadr , struct svc_req *);
#define dscudaMemcpyToArrayH2HId 722
extern  dscudaMemcpyToArrayH2HResult * dscudamemcpytoarrayh2hid_1(RCadr , RCsize , RCsize , RCbuf , RCsize , CLIENT *);
extern  dscudaMemcpyToArrayH2HResult * dscudamemcpytoarrayh2hid_1_svc(RCadr , RCsize , RCsize , RCbuf , RCsize , struct svc_req *);
#define dscudaMemcpyToArrayH2DId 723
extern  dscudaResult * dscudamemcpytoarrayh2did_1(RCadr , RCsize , RCsize , RCbuf , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpytoarrayh2did_1_svc(RCadr , RCsize , RCsize , RCbuf , RCsize , struct svc_req *);
#define dscudaMemcpyToArrayD2HId 724
extern  dscudaMemcpyToArrayD2HResult * dscudamemcpytoarrayd2hid_1(RCsize , RCsize , RCadr , RCsize , CLIENT *);
extern  dscudaMemcpyToArrayD2HResult * dscudamemcpytoarrayd2hid_1_svc(RCsize , RCsize , RCadr , RCsize , struct svc_req *);
#define dscudaMemcpyToArrayD2DId 725
extern  dscudaResult * dscudamemcpytoarrayd2did_1(RCadr , RCsize , RCsize , RCadr , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpytoarrayd2did_1_svc(RCadr , RCsize , RCsize , RCadr , RCsize , struct svc_req *);
#define dscudaMallocPitchId 726
extern  dscudaMallocPitchResult * dscudamallocpitchid_1(RCsize , RCsize , CLIENT *);
extern  dscudaMallocPitchResult * dscudamallocpitchid_1_svc(RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DToArrayH2HId 727
extern  dscudaMemcpy2DToArrayH2HResult * dscudamemcpy2dtoarrayh2hid_1(RCadr , RCsize , RCsize , RCbuf , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaMemcpy2DToArrayH2HResult * dscudamemcpy2dtoarrayh2hid_1_svc(RCadr , RCsize , RCsize , RCbuf , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DToArrayH2DId 728
extern  dscudaResult * dscudamemcpy2dtoarrayh2did_1(RCadr , RCsize , RCsize , RCbuf , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpy2dtoarrayh2did_1_svc(RCadr , RCsize , RCsize , RCbuf , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DToArrayD2HId 729
extern  dscudaMemcpy2DToArrayD2HResult * dscudamemcpy2dtoarrayd2hid_1(RCsize , RCsize , RCadr , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaMemcpy2DToArrayD2HResult * dscudamemcpy2dtoarrayd2hid_1_svc(RCsize , RCsize , RCadr , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DToArrayD2DId 730
extern  dscudaResult * dscudamemcpy2dtoarrayd2did_1(RCadr , RCsize , RCsize , RCadr , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpy2dtoarrayd2did_1_svc(RCadr , RCsize , RCsize , RCadr , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DH2HId 731
extern  dscudaMemcpy2DH2HResult * dscudamemcpy2dh2hid_1(RCadr , RCsize , RCbuf , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaMemcpy2DH2HResult * dscudamemcpy2dh2hid_1_svc(RCadr , RCsize , RCbuf , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DH2DId 732
extern  dscudaResult * dscudamemcpy2dh2did_1(RCadr , RCsize , RCbuf , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpy2dh2did_1_svc(RCadr , RCsize , RCbuf , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DD2HId 733
extern  dscudaMemcpy2DD2HResult * dscudamemcpy2dd2hid_1(RCsize , RCadr , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaMemcpy2DD2HResult * dscudamemcpy2dd2hid_1_svc(RCsize , RCadr , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DD2DId 734
extern  dscudaResult * dscudamemcpy2dd2did_1(RCadr , RCsize , RCadr , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpy2dd2did_1_svc(RCadr , RCsize , RCadr , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemset2DId 735
extern  dscudaResult * dscudamemset2did_1(RCadr , RCsize , int , RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemset2did_1_svc(RCadr , RCsize , int , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpyToSymbolAsyncH2DId 736
extern  dscudaResult * dscudamemcpytosymbolasynch2did_1(int , char *, RCbuf , RCsize , RCsize , RCstream , CLIENT *);
extern  dscudaResult * dscudamemcpytosymbolasynch2did_1_svc(int , char *, RCbuf , RCsize , RCsize , RCstream , struct svc_req *);
#define dscudaMemcpyToSymbolAsyncD2DId 737
extern  dscudaResult * dscudamemcpytosymbolasyncd2did_1(int , char *, RCadr , RCsize , RCsize , RCstream , CLIENT *);
extern  dscudaResult * dscudamemcpytosymbolasyncd2did_1_svc(int , char *, RCadr , RCsize , RCsize , RCstream , struct svc_req *);
#define dscudaMemcpyFromSymbolAsyncD2HId 738
extern  dscudaMemcpyFromSymbolAsyncD2HResult * dscudamemcpyfromsymbolasyncd2hid_1(int , char *, RCsize , RCsize , RCstream , CLIENT *);
extern  dscudaMemcpyFromSymbolAsyncD2HResult * dscudamemcpyfromsymbolasyncd2hid_1_svc(int , char *, RCsize , RCsize , RCstream , struct svc_req *);
#define dscudaMemcpyFromSymbolAsyncD2DId 739
extern  dscudaResult * dscudamemcpyfromsymbolasyncd2did_1(int , RCadr , char *, RCsize , RCsize , RCstream , CLIENT *);
extern  dscudaResult * dscudamemcpyfromsymbolasyncd2did_1_svc(int , RCadr , char *, RCsize , RCsize , RCstream , struct svc_req *);
#define dscudaCreateChannelDescId 1400
extern  dscudaCreateChannelDescResult * dscudacreatechanneldescid_1(int , int , int , int , RCchannelformat , CLIENT *);
extern  dscudaCreateChannelDescResult * dscudacreatechanneldescid_1_svc(int , int , int , int , RCchannelformat , struct svc_req *);
#define dscudaGetChannelDescId 1401
extern  dscudaGetChannelDescResult * dscudagetchanneldescid_1(RCadr , CLIENT *);
extern  dscudaGetChannelDescResult * dscudagetchanneldescid_1_svc(RCadr , struct svc_req *);
#define dscudaBindTextureId 1402
extern  dscudaBindTextureResult * dscudabindtextureid_1(int , char *, RCadr , RCsize , RCtexture , CLIENT *);
extern  dscudaBindTextureResult * dscudabindtextureid_1_svc(int , char *, RCadr , RCsize , RCtexture , struct svc_req *);
#define dscudaBindTexture2DId 1403
extern  dscudaBindTexture2DResult * dscudabindtexture2did_1(int , char *, RCadr , RCsize , RCsize , RCsize , RCtexture , CLIENT *);
extern  dscudaBindTexture2DResult * dscudabindtexture2did_1_svc(int , char *, RCadr , RCsize , RCsize , RCsize , RCtexture , struct svc_req *);
#define dscudaBindTextureToArrayId 1404
extern  dscudaResult * dscudabindtexturetoarrayid_1(int , char *, RCadr , RCtexture , CLIENT *);
extern  dscudaResult * dscudabindtexturetoarrayid_1_svc(int , char *, RCadr , RCtexture , struct svc_req *);
#define dscudaUnbindTextureId 1405
extern  dscudaResult * dscudaunbindtextureid_1(RCtexture , CLIENT *);
extern  dscudaResult * dscudaunbindtextureid_1_svc(RCtexture , struct svc_req *);
#define dscufftPlan3dId 2002
extern  dscufftPlanResult * dscufftplan3did_1(int , int , int , u_int , CLIENT *);
extern  dscufftPlanResult * dscufftplan3did_1_svc(int , int , int , u_int , struct svc_req *);
#define dscufftDestroyId 2004
extern  dscufftResult * dscufftdestroyid_1(u_int , CLIENT *);
extern  dscufftResult * dscufftdestroyid_1_svc(u_int , struct svc_req *);
#define dscufftExecC2CId 2005
extern  dscufftResult * dscufftexecc2cid_1(u_int , RCadr , RCadr , int , CLIENT *);
extern  dscufftResult * dscufftexecc2cid_1_svc(u_int , RCadr , RCadr , int , struct svc_req *);
extern int dscuda_prog_1_freeresult (SVCXPRT *, xdrproc_t, caddr_t);

#else 
#define dscudaThreadExitId 100
extern  dscudaResult * dscudathreadexitid_1();
extern  dscudaResult * dscudathreadexitid_1_svc();
#define dscudaThreadSynchronizeId 101
extern  dscudaResult * dscudathreadsynchronizeid_1();
extern  dscudaResult * dscudathreadsynchronizeid_1_svc();
#define dscudaThreadSetLimitId 102
extern  dscudaResult * dscudathreadsetlimitid_1();
extern  dscudaResult * dscudathreadsetlimitid_1_svc();
#define dscudaThreadGetLimitId 103
extern  dscudaThreadGetLimitResult * dscudathreadgetlimitid_1();
extern  dscudaThreadGetLimitResult * dscudathreadgetlimitid_1_svc();
#define dscudaThreadSetCacheConfigId 104
extern  dscudaResult * dscudathreadsetcacheconfigid_1();
extern  dscudaResult * dscudathreadsetcacheconfigid_1_svc();
#define dscudaThreadGetCacheConfigId 105
extern  dscudaThreadGetCacheConfigResult * dscudathreadgetcacheconfigid_1();
extern  dscudaThreadGetCacheConfigResult * dscudathreadgetcacheconfigid_1_svc();
#define dscudaGetLastErrorId 200
extern  dscudaResult * dscudagetlasterrorid_1();
extern  dscudaResult * dscudagetlasterrorid_1_svc();
#define dscudaPeekAtLastErrorId 201
extern  dscudaResult * dscudapeekatlasterrorid_1();
extern  dscudaResult * dscudapeekatlasterrorid_1_svc();
#define dscudaGetErrorStringId 202
extern  dscudaGetErrorStringResult * dscudageterrorstringid_1();
extern  dscudaGetErrorStringResult * dscudageterrorstringid_1_svc();
#define dscudaGetDeviceId 300
extern  dscudaGetDeviceResult * dscudagetdeviceid_1();
extern  dscudaGetDeviceResult * dscudagetdeviceid_1_svc();
#define dscudaGetDeviceCountId 301
extern  dscudaGetDeviceCountResult * dscudagetdevicecountid_1();
extern  dscudaGetDeviceCountResult * dscudagetdevicecountid_1_svc();
#define dscudaGetDevicePropertiesId 302
extern  dscudaGetDevicePropertiesResult * dscudagetdevicepropertiesid_1();
extern  dscudaGetDevicePropertiesResult * dscudagetdevicepropertiesid_1_svc();
#define dscudaDriverGetVersionId 303
extern  dscudaDriverGetVersionResult * dscudadrivergetversionid_1();
extern  dscudaDriverGetVersionResult * dscudadrivergetversionid_1_svc();
#define dscudaRuntimeGetVersionId 304
extern  dscudaRuntimeGetVersionResult * dscudaruntimegetversionid_1();
extern  dscudaRuntimeGetVersionResult * dscudaruntimegetversionid_1_svc();
#define dscudaSetDeviceId 305
extern  dscudaResult * dscudasetdeviceid_1();
extern  dscudaResult * dscudasetdeviceid_1_svc();
#define dscudaSetDeviceFlagsId 306
extern  dscudaResult * dscudasetdeviceflagsid_1();
extern  dscudaResult * dscudasetdeviceflagsid_1_svc();
#define dscudaChooseDeviceId 307
extern  dscudaChooseDeviceResult * dscudachoosedeviceid_1();
extern  dscudaChooseDeviceResult * dscudachoosedeviceid_1_svc();
#define dscudaDeviceSynchronize 308
extern  dscudaResult * dscudadevicesynchronize_1();
extern  dscudaResult * dscudadevicesynchronize_1_svc();
#define dscudaDeviceReset 309
extern  dscudaResult * dscudadevicereset_1();
extern  dscudaResult * dscudadevicereset_1_svc();
#define dscudaStreamCreateId 400
extern  dscudaStreamCreateResult * dscudastreamcreateid_1();
extern  dscudaStreamCreateResult * dscudastreamcreateid_1_svc();
#define dscudaStreamDestroyId 401
extern  dscudaResult * dscudastreamdestroyid_1();
extern  dscudaResult * dscudastreamdestroyid_1_svc();
#define dscudaStreamSynchronizeId 402
extern  dscudaResult * dscudastreamsynchronizeid_1();
extern  dscudaResult * dscudastreamsynchronizeid_1_svc();
#define dscudaStreamQueryId 403
extern  dscudaResult * dscudastreamqueryid_1();
extern  dscudaResult * dscudastreamqueryid_1_svc();
#define dscudaStreamWaitEventId 404
extern  dscudaResult * dscudastreamwaiteventid_1();
extern  dscudaResult * dscudastreamwaiteventid_1_svc();
#define dscudaEventCreateId 500
extern  dscudaEventCreateResult * dscudaeventcreateid_1();
extern  dscudaEventCreateResult * dscudaeventcreateid_1_svc();
#define dscudaEventCreateWithFlagsId 501
extern  dscudaEventCreateResult * dscudaeventcreatewithflagsid_1();
extern  dscudaEventCreateResult * dscudaeventcreatewithflagsid_1_svc();
#define dscudaEventDestroyId 502
extern  dscudaResult * dscudaeventdestroyid_1();
extern  dscudaResult * dscudaeventdestroyid_1_svc();
#define dscudaEventElapsedTimeId 503
extern  dscudaEventElapsedTimeResult * dscudaeventelapsedtimeid_1();
extern  dscudaEventElapsedTimeResult * dscudaeventelapsedtimeid_1_svc();
#define dscudaEventRecordId 504
extern  dscudaResult * dscudaeventrecordid_1();
extern  dscudaResult * dscudaeventrecordid_1_svc();
#define dscudaEventSynchronizeId 505
extern  dscudaResult * dscudaeventsynchronizeid_1();
extern  dscudaResult * dscudaeventsynchronizeid_1_svc();
#define dscudaEventQueryId 506
extern  dscudaResult * dscudaeventqueryid_1();
extern  dscudaResult * dscudaeventqueryid_1_svc();
#define dscudaLaunchKernelId 600
extern  void * dscudalaunchkernelid_1();
extern  void * dscudalaunchkernelid_1_svc();
#define dscudaLoadModuleId 601
extern  dscudaLoadModuleResult * dscudaloadmoduleid_1();
extern  dscudaLoadModuleResult * dscudaloadmoduleid_1_svc();
#define dscudaFuncGetAttributesId 602
extern  dscudaFuncGetAttributesResult * dscudafuncgetattributesid_1();
extern  dscudaFuncGetAttributesResult * dscudafuncgetattributesid_1_svc();
#define dscudaMallocId 700
extern  dscudaMallocResult * dscudamallocid_1();
extern  dscudaMallocResult * dscudamallocid_1_svc();
#define dscudaFreeId 701
extern  dscudaResult * dscudafreeid_1();
extern  dscudaResult * dscudafreeid_1_svc();
#define dscudaMemcpyH2HId 702
extern  dscudaMemcpyH2HResult * dscudamemcpyh2hid_1();
extern  dscudaMemcpyH2HResult * dscudamemcpyh2hid_1_svc();
#define dscudaMemcpyH2DId 703
extern  dscudaResult * dscudamemcpyh2did_1();
extern  dscudaResult * dscudamemcpyh2did_1_svc();
#define dscudaMemcpyD2HId 704
extern  dscudaMemcpyD2HResult * dscudamemcpyd2hid_1();
extern  dscudaMemcpyD2HResult * dscudamemcpyd2hid_1_svc();
#define dscudaMemcpyD2DId 705
extern  dscudaResult * dscudamemcpyd2did_1();
extern  dscudaResult * dscudamemcpyd2did_1_svc();
#define dscudaMemcpyAsyncH2HId 706
extern  dscudaMemcpyAsyncH2HResult * dscudamemcpyasynch2hid_1();
extern  dscudaMemcpyAsyncH2HResult * dscudamemcpyasynch2hid_1_svc();
#define dscudaMemcpyAsyncH2DId 707
extern  dscudaResult * dscudamemcpyasynch2did_1();
extern  dscudaResult * dscudamemcpyasynch2did_1_svc();
#define dscudaMemcpyAsyncD2HId 708
extern  dscudaMemcpyAsyncD2HResult * dscudamemcpyasyncd2hid_1();
extern  dscudaMemcpyAsyncD2HResult * dscudamemcpyasyncd2hid_1_svc();
#define dscudaMemcpyAsyncD2DId 709
extern  dscudaResult * dscudamemcpyasyncd2did_1();
extern  dscudaResult * dscudamemcpyasyncd2did_1_svc();
#define dscudaMemcpyToSymbolH2DId 710
extern  dscudaResult * dscudamemcpytosymbolh2did_1();
extern  dscudaResult * dscudamemcpytosymbolh2did_1_svc();
#define dscudaMemcpyToSymbolD2DId 711
extern  dscudaResult * dscudamemcpytosymbold2did_1();
extern  dscudaResult * dscudamemcpytosymbold2did_1_svc();
#define dscudaMemcpyFromSymbolD2HId 712
extern  dscudaMemcpyFromSymbolD2HResult * dscudamemcpyfromsymbold2hid_1();
extern  dscudaMemcpyFromSymbolD2HResult * dscudamemcpyfromsymbold2hid_1_svc();
#define dscudaMemcpyFromSymbolD2DId 713
extern  dscudaResult * dscudamemcpyfromsymbold2did_1();
extern  dscudaResult * dscudamemcpyfromsymbold2did_1_svc();
#define dscudaMemsetId 714
extern  dscudaResult * dscudamemsetid_1();
extern  dscudaResult * dscudamemsetid_1_svc();
#define dscudaHostAllocId 715
extern  dscudaHostAllocResult * dscudahostallocid_1();
extern  dscudaHostAllocResult * dscudahostallocid_1_svc();
#define dscudaMallocHostId 716
extern  dscudaMallocHostResult * dscudamallochostid_1();
extern  dscudaMallocHostResult * dscudamallochostid_1_svc();
#define dscudaFreeHostId 717
extern  dscudaResult * dscudafreehostid_1();
extern  dscudaResult * dscudafreehostid_1_svc();
#define dscudaHostGetDevicePointerId 718
extern  dscudaHostGetDevicePointerResult * dscudahostgetdevicepointerid_1();
extern  dscudaHostGetDevicePointerResult * dscudahostgetdevicepointerid_1_svc();
#define dscudaHostGetFlagsID 719
extern  dscudaHostGetFlagsResult * dscudahostgetflagsid_1();
extern  dscudaHostGetFlagsResult * dscudahostgetflagsid_1_svc();
#define dscudaMallocArrayId 720
extern  dscudaMallocArrayResult * dscudamallocarrayid_1();
extern  dscudaMallocArrayResult * dscudamallocarrayid_1_svc();
#define dscudaFreeArrayId 721
extern  dscudaResult * dscudafreearrayid_1();
extern  dscudaResult * dscudafreearrayid_1_svc();
#define dscudaMemcpyToArrayH2HId 722
extern  dscudaMemcpyToArrayH2HResult * dscudamemcpytoarrayh2hid_1();
extern  dscudaMemcpyToArrayH2HResult * dscudamemcpytoarrayh2hid_1_svc();
#define dscudaMemcpyToArrayH2DId 723
extern  dscudaResult * dscudamemcpytoarrayh2did_1();
extern  dscudaResult * dscudamemcpytoarrayh2did_1_svc();
#define dscudaMemcpyToArrayD2HId 724
extern  dscudaMemcpyToArrayD2HResult * dscudamemcpytoarrayd2hid_1();
extern  dscudaMemcpyToArrayD2HResult * dscudamemcpytoarrayd2hid_1_svc();
#define dscudaMemcpyToArrayD2DId 725
extern  dscudaResult * dscudamemcpytoarrayd2did_1();
extern  dscudaResult * dscudamemcpytoarrayd2did_1_svc();
#define dscudaMallocPitchId 726
extern  dscudaMallocPitchResult * dscudamallocpitchid_1();
extern  dscudaMallocPitchResult * dscudamallocpitchid_1_svc();
#define dscudaMemcpy2DToArrayH2HId 727
extern  dscudaMemcpy2DToArrayH2HResult * dscudamemcpy2dtoarrayh2hid_1();
extern  dscudaMemcpy2DToArrayH2HResult * dscudamemcpy2dtoarrayh2hid_1_svc();
#define dscudaMemcpy2DToArrayH2DId 728
extern  dscudaResult * dscudamemcpy2dtoarrayh2did_1();
extern  dscudaResult * dscudamemcpy2dtoarrayh2did_1_svc();
#define dscudaMemcpy2DToArrayD2HId 729
extern  dscudaMemcpy2DToArrayD2HResult * dscudamemcpy2dtoarrayd2hid_1();
extern  dscudaMemcpy2DToArrayD2HResult * dscudamemcpy2dtoarrayd2hid_1_svc();
#define dscudaMemcpy2DToArrayD2DId 730
extern  dscudaResult * dscudamemcpy2dtoarrayd2did_1();
extern  dscudaResult * dscudamemcpy2dtoarrayd2did_1_svc();
#define dscudaMemcpy2DH2HId 731
extern  dscudaMemcpy2DH2HResult * dscudamemcpy2dh2hid_1();
extern  dscudaMemcpy2DH2HResult * dscudamemcpy2dh2hid_1_svc();
#define dscudaMemcpy2DH2DId 732
extern  dscudaResult * dscudamemcpy2dh2did_1();
extern  dscudaResult * dscudamemcpy2dh2did_1_svc();
#define dscudaMemcpy2DD2HId 733
extern  dscudaMemcpy2DD2HResult * dscudamemcpy2dd2hid_1();
extern  dscudaMemcpy2DD2HResult * dscudamemcpy2dd2hid_1_svc();
#define dscudaMemcpy2DD2DId 734
extern  dscudaResult * dscudamemcpy2dd2did_1();
extern  dscudaResult * dscudamemcpy2dd2did_1_svc();
#define dscudaMemset2DId 735
extern  dscudaResult * dscudamemset2did_1();
extern  dscudaResult * dscudamemset2did_1_svc();
#define dscudaMemcpyToSymbolAsyncH2DId 736
extern  dscudaResult * dscudamemcpytosymbolasynch2did_1();
extern  dscudaResult * dscudamemcpytosymbolasynch2did_1_svc();
#define dscudaMemcpyToSymbolAsyncD2DId 737
extern  dscudaResult * dscudamemcpytosymbolasyncd2did_1();
extern  dscudaResult * dscudamemcpytosymbolasyncd2did_1_svc();
#define dscudaMemcpyFromSymbolAsyncD2HId 738
extern  dscudaMemcpyFromSymbolAsyncD2HResult * dscudamemcpyfromsymbolasyncd2hid_1();
extern  dscudaMemcpyFromSymbolAsyncD2HResult * dscudamemcpyfromsymbolasyncd2hid_1_svc();
#define dscudaMemcpyFromSymbolAsyncD2DId 739
extern  dscudaResult * dscudamemcpyfromsymbolasyncd2did_1();
extern  dscudaResult * dscudamemcpyfromsymbolasyncd2did_1_svc();
#define dscudaCreateChannelDescId 1400
extern  dscudaCreateChannelDescResult * dscudacreatechanneldescid_1();
extern  dscudaCreateChannelDescResult * dscudacreatechanneldescid_1_svc();
#define dscudaGetChannelDescId 1401
extern  dscudaGetChannelDescResult * dscudagetchanneldescid_1();
extern  dscudaGetChannelDescResult * dscudagetchanneldescid_1_svc();
#define dscudaBindTextureId 1402
extern  dscudaBindTextureResult * dscudabindtextureid_1();
extern  dscudaBindTextureResult * dscudabindtextureid_1_svc();
#define dscudaBindTexture2DId 1403
extern  dscudaBindTexture2DResult * dscudabindtexture2did_1();
extern  dscudaBindTexture2DResult * dscudabindtexture2did_1_svc();
#define dscudaBindTextureToArrayId 1404
extern  dscudaResult * dscudabindtexturetoarrayid_1();
extern  dscudaResult * dscudabindtexturetoarrayid_1_svc();
#define dscudaUnbindTextureId 1405
extern  dscudaResult * dscudaunbindtextureid_1();
extern  dscudaResult * dscudaunbindtextureid_1_svc();
#define dscufftPlan3dId 2002
extern  dscufftPlanResult * dscufftplan3did_1();
extern  dscufftPlanResult * dscufftplan3did_1_svc();
#define dscufftDestroyId 2004
extern  dscufftResult * dscufftdestroyid_1();
extern  dscufftResult * dscufftdestroyid_1_svc();
#define dscufftExecC2CId 2005
extern  dscufftResult * dscufftexecc2cid_1();
extern  dscufftResult * dscufftexecc2cid_1_svc();
extern int dscuda_prog_1_freeresult ();
#endif 



#if defined(__STDC__) || defined(__cplusplus)
extern  bool_t xdr_RCadr (XDR *, RCadr*);
extern  bool_t xdr_RCstream (XDR *, RCstream*);
extern  bool_t xdr_RCevent (XDR *, RCevent*);
extern  bool_t xdr_RCipaddr (XDR *, RCipaddr*);
extern  bool_t xdr_RCsize (XDR *, RCsize*);
extern  bool_t xdr_RCerror (XDR *, RCerror*);
extern  bool_t xdr_RCbuf (XDR *, RCbuf*);
extern  bool_t xdr_RCchannelformat (XDR *, RCchannelformat*);
extern  bool_t xdr_RCpid (XDR *, RCpid*);
extern  bool_t xdr_RCchanneldesc_t (XDR *, RCchanneldesc_t*);
extern  bool_t xdr_RCchanneldesc (XDR *, RCchanneldesc*);
extern  bool_t xdr_RCtexture_t (XDR *, RCtexture_t*);
extern  bool_t xdr_RCtexture (XDR *, RCtexture*);
extern  bool_t xdr_RCfuncattr_t (XDR *, RCfuncattr_t*);
extern  bool_t xdr_RCfuncattr (XDR *, RCfuncattr*);
extern  bool_t xdr_RCargType (XDR *, RCargType*);
extern  bool_t xdr_RCargVal (XDR *, RCargVal*);
extern  bool_t xdr_RCarg (XDR *, RCarg*);
extern  bool_t xdr_RCargs (XDR *, RCargs*);
extern  bool_t xdr_dscudaResult (XDR *, dscudaResult*);
extern  bool_t xdr_dscudaThreadGetLimitResult (XDR *, dscudaThreadGetLimitResult*);
extern  bool_t xdr_dscudaThreadGetCacheConfigResult (XDR *, dscudaThreadGetCacheConfigResult*);
extern  bool_t xdr_dscudaMallocResult (XDR *, dscudaMallocResult*);
extern  bool_t xdr_dscudaHostAllocResult (XDR *, dscudaHostAllocResult*);
extern  bool_t xdr_dscudaMallocHostResult (XDR *, dscudaMallocHostResult*);
extern  bool_t xdr_dscudaMallocArrayResult (XDR *, dscudaMallocArrayResult*);
extern  bool_t xdr_dscudaMallocPitchResult (XDR *, dscudaMallocPitchResult*);
extern  bool_t xdr_dscudaMemcpyD2HResult (XDR *, dscudaMemcpyD2HResult*);
extern  bool_t xdr_dscudaMemcpyH2HResult (XDR *, dscudaMemcpyH2HResult*);
extern  bool_t xdr_dscudaMemcpyToArrayD2HResult (XDR *, dscudaMemcpyToArrayD2HResult*);
extern  bool_t xdr_dscudaMemcpyToArrayH2HResult (XDR *, dscudaMemcpyToArrayH2HResult*);
extern  bool_t xdr_dscudaMemcpy2DToArrayD2HResult (XDR *, dscudaMemcpy2DToArrayD2HResult*);
extern  bool_t xdr_dscudaMemcpy2DToArrayH2HResult (XDR *, dscudaMemcpy2DToArrayH2HResult*);
extern  bool_t xdr_dscudaMemcpy2DD2HResult (XDR *, dscudaMemcpy2DD2HResult*);
extern  bool_t xdr_dscudaMemcpy2DH2HResult (XDR *, dscudaMemcpy2DH2HResult*);
extern  bool_t xdr_dscudaGetDeviceResult (XDR *, dscudaGetDeviceResult*);
extern  bool_t xdr_dscudaGetDeviceCountResult (XDR *, dscudaGetDeviceCountResult*);
extern  bool_t xdr_dscudaGetDevicePropertiesResult (XDR *, dscudaGetDevicePropertiesResult*);
extern  bool_t xdr_dscudaDriverGetVersionResult (XDR *, dscudaDriverGetVersionResult*);
extern  bool_t xdr_dscudaRuntimeGetVersionResult (XDR *, dscudaRuntimeGetVersionResult*);
extern  bool_t xdr_dscudaGetErrorStringResult (XDR *, dscudaGetErrorStringResult*);
extern  bool_t xdr_dscudaCreateChannelDescResult (XDR *, dscudaCreateChannelDescResult*);
extern  bool_t xdr_dscudaGetChannelDescResult (XDR *, dscudaGetChannelDescResult*);
extern  bool_t xdr_dscudaChooseDeviceResult (XDR *, dscudaChooseDeviceResult*);
extern  bool_t xdr_dscudaMemcpyAsyncD2HResult (XDR *, dscudaMemcpyAsyncD2HResult*);
extern  bool_t xdr_dscudaMemcpyAsyncH2HResult (XDR *, dscudaMemcpyAsyncH2HResult*);
extern  bool_t xdr_dscudaMemcpyFromSymbolD2HResult (XDR *, dscudaMemcpyFromSymbolD2HResult*);
extern  bool_t xdr_dscudaMemcpyFromSymbolAsyncD2HResult (XDR *, dscudaMemcpyFromSymbolAsyncD2HResult*);
extern  bool_t xdr_dscudaStreamCreateResult (XDR *, dscudaStreamCreateResult*);
extern  bool_t xdr_dscudaEventCreateResult (XDR *, dscudaEventCreateResult*);
extern  bool_t xdr_dscudaEventElapsedTimeResult (XDR *, dscudaEventElapsedTimeResult*);
extern  bool_t xdr_dscudaHostGetDevicePointerResult (XDR *, dscudaHostGetDevicePointerResult*);
extern  bool_t xdr_dscudaHostGetFlagsResult (XDR *, dscudaHostGetFlagsResult*);
extern  bool_t xdr_dscudaLoadModuleResult (XDR *, dscudaLoadModuleResult*);
extern  bool_t xdr_dscudaFuncGetAttributesResult (XDR *, dscudaFuncGetAttributesResult*);
extern  bool_t xdr_dscudaBindTextureResult (XDR *, dscudaBindTextureResult*);
extern  bool_t xdr_dscudaBindTexture2DResult (XDR *, dscudaBindTexture2DResult*);
extern  bool_t xdr_dscufftResult (XDR *, dscufftResult*);
extern  bool_t xdr_dscufftPlanResult (XDR *, dscufftPlanResult*);
extern  bool_t xdr_dscublasResult (XDR *, dscublasResult*);
extern  bool_t xdr_dscublasCreateResult (XDR *, dscublasCreateResult*);
extern  bool_t xdr_dscublasGetVectorResult (XDR *, dscublasGetVectorResult*);
extern  bool_t xdr_RCdim3 (XDR *, RCdim3*);
extern  bool_t xdr_dscudathreadsetlimitid_1_argument (XDR *, dscudathreadsetlimitid_1_argument*);
extern  bool_t xdr_dscudastreamwaiteventid_1_argument (XDR *, dscudastreamwaiteventid_1_argument*);
extern  bool_t xdr_dscudaeventelapsedtimeid_1_argument (XDR *, dscudaeventelapsedtimeid_1_argument*);
extern  bool_t xdr_dscudaeventrecordid_1_argument (XDR *, dscudaeventrecordid_1_argument*);
extern  bool_t xdr_dscudalaunchkernelid_1_argument (XDR *, dscudalaunchkernelid_1_argument*);
extern  bool_t xdr_dscudaloadmoduleid_1_argument (XDR *, dscudaloadmoduleid_1_argument*);
extern  bool_t xdr_dscudafuncgetattributesid_1_argument (XDR *, dscudafuncgetattributesid_1_argument*);
extern  bool_t xdr_dscudamemcpyh2hid_1_argument (XDR *, dscudamemcpyh2hid_1_argument*);
extern  bool_t xdr_dscudamemcpyh2did_1_argument (XDR *, dscudamemcpyh2did_1_argument*);
extern  bool_t xdr_dscudamemcpyd2hid_1_argument (XDR *, dscudamemcpyd2hid_1_argument*);
extern  bool_t xdr_dscudamemcpyd2did_1_argument (XDR *, dscudamemcpyd2did_1_argument*);
extern  bool_t xdr_dscudamemcpyasynch2hid_1_argument (XDR *, dscudamemcpyasynch2hid_1_argument*);
extern  bool_t xdr_dscudamemcpyasynch2did_1_argument (XDR *, dscudamemcpyasynch2did_1_argument*);
extern  bool_t xdr_dscudamemcpyasyncd2hid_1_argument (XDR *, dscudamemcpyasyncd2hid_1_argument*);
extern  bool_t xdr_dscudamemcpyasyncd2did_1_argument (XDR *, dscudamemcpyasyncd2did_1_argument*);
extern  bool_t xdr_dscudamemcpytosymbolh2did_1_argument (XDR *, dscudamemcpytosymbolh2did_1_argument*);
extern  bool_t xdr_dscudamemcpytosymbold2did_1_argument (XDR *, dscudamemcpytosymbold2did_1_argument*);
extern  bool_t xdr_dscudamemcpyfromsymbold2hid_1_argument (XDR *, dscudamemcpyfromsymbold2hid_1_argument*);
extern  bool_t xdr_dscudamemcpyfromsymbold2did_1_argument (XDR *, dscudamemcpyfromsymbold2did_1_argument*);
extern  bool_t xdr_dscudamemsetid_1_argument (XDR *, dscudamemsetid_1_argument*);
extern  bool_t xdr_dscudahostallocid_1_argument (XDR *, dscudahostallocid_1_argument*);
extern  bool_t xdr_dscudahostgetdevicepointerid_1_argument (XDR *, dscudahostgetdevicepointerid_1_argument*);
extern  bool_t xdr_dscudamallocarrayid_1_argument (XDR *, dscudamallocarrayid_1_argument*);
extern  bool_t xdr_dscudamemcpytoarrayh2hid_1_argument (XDR *, dscudamemcpytoarrayh2hid_1_argument*);
extern  bool_t xdr_dscudamemcpytoarrayh2did_1_argument (XDR *, dscudamemcpytoarrayh2did_1_argument*);
extern  bool_t xdr_dscudamemcpytoarrayd2hid_1_argument (XDR *, dscudamemcpytoarrayd2hid_1_argument*);
extern  bool_t xdr_dscudamemcpytoarrayd2did_1_argument (XDR *, dscudamemcpytoarrayd2did_1_argument*);
extern  bool_t xdr_dscudamallocpitchid_1_argument (XDR *, dscudamallocpitchid_1_argument*);
extern  bool_t xdr_dscudamemcpy2dtoarrayh2hid_1_argument (XDR *, dscudamemcpy2dtoarrayh2hid_1_argument*);
extern  bool_t xdr_dscudamemcpy2dtoarrayh2did_1_argument (XDR *, dscudamemcpy2dtoarrayh2did_1_argument*);
extern  bool_t xdr_dscudamemcpy2dtoarrayd2hid_1_argument (XDR *, dscudamemcpy2dtoarrayd2hid_1_argument*);
extern  bool_t xdr_dscudamemcpy2dtoarrayd2did_1_argument (XDR *, dscudamemcpy2dtoarrayd2did_1_argument*);
extern  bool_t xdr_dscudamemcpy2dh2hid_1_argument (XDR *, dscudamemcpy2dh2hid_1_argument*);
extern  bool_t xdr_dscudamemcpy2dh2did_1_argument (XDR *, dscudamemcpy2dh2did_1_argument*);
extern  bool_t xdr_dscudamemcpy2dd2hid_1_argument (XDR *, dscudamemcpy2dd2hid_1_argument*);
extern  bool_t xdr_dscudamemcpy2dd2did_1_argument (XDR *, dscudamemcpy2dd2did_1_argument*);
extern  bool_t xdr_dscudamemset2did_1_argument (XDR *, dscudamemset2did_1_argument*);
extern  bool_t xdr_dscudamemcpytosymbolasynch2did_1_argument (XDR *, dscudamemcpytosymbolasynch2did_1_argument*);
extern  bool_t xdr_dscudamemcpytosymbolasyncd2did_1_argument (XDR *, dscudamemcpytosymbolasyncd2did_1_argument*);
extern  bool_t xdr_dscudamemcpyfromsymbolasyncd2hid_1_argument (XDR *, dscudamemcpyfromsymbolasyncd2hid_1_argument*);
extern  bool_t xdr_dscudamemcpyfromsymbolasyncd2did_1_argument (XDR *, dscudamemcpyfromsymbolasyncd2did_1_argument*);
extern  bool_t xdr_dscudacreatechanneldescid_1_argument (XDR *, dscudacreatechanneldescid_1_argument*);
extern  bool_t xdr_dscudabindtextureid_1_argument (XDR *, dscudabindtextureid_1_argument*);
extern  bool_t xdr_dscudabindtexture2did_1_argument (XDR *, dscudabindtexture2did_1_argument*);
extern  bool_t xdr_dscudabindtexturetoarrayid_1_argument (XDR *, dscudabindtexturetoarrayid_1_argument*);
extern  bool_t xdr_dscufftplan3did_1_argument (XDR *, dscufftplan3did_1_argument*);
extern  bool_t xdr_dscufftexecc2cid_1_argument (XDR *, dscufftexecc2cid_1_argument*);

#else 
extern bool_t xdr_RCadr ();
extern bool_t xdr_RCstream ();
extern bool_t xdr_RCevent ();
extern bool_t xdr_RCipaddr ();
extern bool_t xdr_RCsize ();
extern bool_t xdr_RCerror ();
extern bool_t xdr_RCbuf ();
extern bool_t xdr_RCchannelformat ();
extern bool_t xdr_RCpid ();
extern bool_t xdr_RCchanneldesc_t ();
extern bool_t xdr_RCchanneldesc ();
extern bool_t xdr_RCtexture_t ();
extern bool_t xdr_RCtexture ();
extern bool_t xdr_RCfuncattr_t ();
extern bool_t xdr_RCfuncattr ();
extern bool_t xdr_RCargType ();
extern bool_t xdr_RCargVal ();
extern bool_t xdr_RCarg ();
extern bool_t xdr_RCargs ();
extern bool_t xdr_dscudaResult ();
extern bool_t xdr_dscudaThreadGetLimitResult ();
extern bool_t xdr_dscudaThreadGetCacheConfigResult ();
extern bool_t xdr_dscudaMallocResult ();
extern bool_t xdr_dscudaHostAllocResult ();
extern bool_t xdr_dscudaMallocHostResult ();
extern bool_t xdr_dscudaMallocArrayResult ();
extern bool_t xdr_dscudaMallocPitchResult ();
extern bool_t xdr_dscudaMemcpyD2HResult ();
extern bool_t xdr_dscudaMemcpyH2HResult ();
extern bool_t xdr_dscudaMemcpyToArrayD2HResult ();
extern bool_t xdr_dscudaMemcpyToArrayH2HResult ();
extern bool_t xdr_dscudaMemcpy2DToArrayD2HResult ();
extern bool_t xdr_dscudaMemcpy2DToArrayH2HResult ();
extern bool_t xdr_dscudaMemcpy2DD2HResult ();
extern bool_t xdr_dscudaMemcpy2DH2HResult ();
extern bool_t xdr_dscudaGetDeviceResult ();
extern bool_t xdr_dscudaGetDeviceCountResult ();
extern bool_t xdr_dscudaGetDevicePropertiesResult ();
extern bool_t xdr_dscudaDriverGetVersionResult ();
extern bool_t xdr_dscudaRuntimeGetVersionResult ();
extern bool_t xdr_dscudaGetErrorStringResult ();
extern bool_t xdr_dscudaCreateChannelDescResult ();
extern bool_t xdr_dscudaGetChannelDescResult ();
extern bool_t xdr_dscudaChooseDeviceResult ();
extern bool_t xdr_dscudaMemcpyAsyncD2HResult ();
extern bool_t xdr_dscudaMemcpyAsyncH2HResult ();
extern bool_t xdr_dscudaMemcpyFromSymbolD2HResult ();
extern bool_t xdr_dscudaMemcpyFromSymbolAsyncD2HResult ();
extern bool_t xdr_dscudaStreamCreateResult ();
extern bool_t xdr_dscudaEventCreateResult ();
extern bool_t xdr_dscudaEventElapsedTimeResult ();
extern bool_t xdr_dscudaHostGetDevicePointerResult ();
extern bool_t xdr_dscudaHostGetFlagsResult ();
extern bool_t xdr_dscudaLoadModuleResult ();
extern bool_t xdr_dscudaFuncGetAttributesResult ();
extern bool_t xdr_dscudaBindTextureResult ();
extern bool_t xdr_dscudaBindTexture2DResult ();
extern bool_t xdr_dscufftResult ();
extern bool_t xdr_dscufftPlanResult ();
extern bool_t xdr_dscublasResult ();
extern bool_t xdr_dscublasCreateResult ();
extern bool_t xdr_dscublasGetVectorResult ();
extern bool_t xdr_RCdim3 ();
extern bool_t xdr_dscudathreadsetlimitid_1_argument ();
extern bool_t xdr_dscudastreamwaiteventid_1_argument ();
extern bool_t xdr_dscudaeventelapsedtimeid_1_argument ();
extern bool_t xdr_dscudaeventrecordid_1_argument ();
extern bool_t xdr_dscudalaunchkernelid_1_argument ();
extern bool_t xdr_dscudaloadmoduleid_1_argument ();
extern bool_t xdr_dscudafuncgetattributesid_1_argument ();
extern bool_t xdr_dscudamemcpyh2hid_1_argument ();
extern bool_t xdr_dscudamemcpyh2did_1_argument ();
extern bool_t xdr_dscudamemcpyd2hid_1_argument ();
extern bool_t xdr_dscudamemcpyd2did_1_argument ();
extern bool_t xdr_dscudamemcpyasynch2hid_1_argument ();
extern bool_t xdr_dscudamemcpyasynch2did_1_argument ();
extern bool_t xdr_dscudamemcpyasyncd2hid_1_argument ();
extern bool_t xdr_dscudamemcpyasyncd2did_1_argument ();
extern bool_t xdr_dscudamemcpytosymbolh2did_1_argument ();
extern bool_t xdr_dscudamemcpytosymbold2did_1_argument ();
extern bool_t xdr_dscudamemcpyfromsymbold2hid_1_argument ();
extern bool_t xdr_dscudamemcpyfromsymbold2did_1_argument ();
extern bool_t xdr_dscudamemsetid_1_argument ();
extern bool_t xdr_dscudahostallocid_1_argument ();
extern bool_t xdr_dscudahostgetdevicepointerid_1_argument ();
extern bool_t xdr_dscudamallocarrayid_1_argument ();
extern bool_t xdr_dscudamemcpytoarrayh2hid_1_argument ();
extern bool_t xdr_dscudamemcpytoarrayh2did_1_argument ();
extern bool_t xdr_dscudamemcpytoarrayd2hid_1_argument ();
extern bool_t xdr_dscudamemcpytoarrayd2did_1_argument ();
extern bool_t xdr_dscudamallocpitchid_1_argument ();
extern bool_t xdr_dscudamemcpy2dtoarrayh2hid_1_argument ();
extern bool_t xdr_dscudamemcpy2dtoarrayh2did_1_argument ();
extern bool_t xdr_dscudamemcpy2dtoarrayd2hid_1_argument ();
extern bool_t xdr_dscudamemcpy2dtoarrayd2did_1_argument ();
extern bool_t xdr_dscudamemcpy2dh2hid_1_argument ();
extern bool_t xdr_dscudamemcpy2dh2did_1_argument ();
extern bool_t xdr_dscudamemcpy2dd2hid_1_argument ();
extern bool_t xdr_dscudamemcpy2dd2did_1_argument ();
extern bool_t xdr_dscudamemset2did_1_argument ();
extern bool_t xdr_dscudamemcpytosymbolasynch2did_1_argument ();
extern bool_t xdr_dscudamemcpytosymbolasyncd2did_1_argument ();
extern bool_t xdr_dscudamemcpyfromsymbolasyncd2hid_1_argument ();
extern bool_t xdr_dscudamemcpyfromsymbolasyncd2did_1_argument ();
extern bool_t xdr_dscudacreatechanneldescid_1_argument ();
extern bool_t xdr_dscudabindtextureid_1_argument ();
extern bool_t xdr_dscudabindtexture2did_1_argument ();
extern bool_t xdr_dscudabindtexturetoarrayid_1_argument ();
extern bool_t xdr_dscufftplan3did_1_argument ();
extern bool_t xdr_dscufftexecc2cid_1_argument ();

#endif 

#ifdef __cplusplus
}
#endif

#endif 
#pragma end dscudarpc.h
#pragma begin dscudadefs.h
#ifndef _DSCUDADEFS_H
#define _DSCUDADEFS_H

#define RC_NSERVERMAX 32    
#define RC_NDEVICEMAX 32    
#define RC_NREDUNDANCYMAX 4 
#define RC_NVDEVMAX 64      
#define RC_NPTHREADMAX 64   

#define RC_BUFSIZE (1024*1024) 
#define RC_NKMODULEMAX 128  
#define RC_NKFUNCMAX   128  
#define RC_KARGMAX     64   
#define RC_KMODULENAMELEN 64   
#define RC_KNAMELEN       64   
#define RC_KMODULEIMAGELEN (1024*1024*2)   
#define RC_SNAMELEN       64   

#define RC_CACHE_MODULE (1) 
#define RC_CLIENT_CACHE_LIFETIME (30) 
#define RC_SERVER_CACHE_LIFETIME (RC_CLIENT_CACHE_LIFETIME+30) 

#define RC_SUPPORT_PAGELOCK (0)  
#define RC_SUPPORT_STREAM (0)
#define RC_SUPPORT_CONCURRENT_EXEC (0)

#define RC_DAEMON_IP_PORT  (65432)
#define RC_SERVER_IP_PORT  (RC_DAEMON_IP_PORT+1)

#endif 
#pragma end dscudadefs.h
#pragma begin dscudamacros.h
#ifndef DSCUDA_MACROS_H
#define DSCUDA_MACROS_H

#define WARN(lv, fmt, args...) if (lv <= dscudaWarnLevel()) fprintf(stderr, fmt, ## args);
#define WARNONCE(lv, fmt, args...) if (lv <= dscudaWarnLevel()) { \
        static int firstcall = 1;                                 \
        if (firstcall) {                                          \
            firstcall = 0;                                        \
            fprintf(stderr, fmt, ## args);                        \
        }                                                         \
    }

#define ALIGN_UP(off, align) (off) = ((off) + (align) - 1) & ~((align) - 1)
int dscudaWarnLevel(void);
void dscudaSetWarnLevel(int level);

#endif 
#pragma end dscudamacros.h
#pragma begin ibv_rdma.h
#ifndef RDMA_COMMON_H
#define RDMA_COMMON_H

#ifdef RPC_ONLY

typedef struct {
    int type;
    union {
        uint64_t pointerval;
        unsigned int intval;
        float floatval;
        char customval[RC_KARGMAX];
    } val;
    unsigned int offset;
    unsigned int size;
} IbvArg;

#else

#include <netdb.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <rdma/rdma_cma.h>
#include <hip/hip_runtime_api.h>
#pragma begin dscudadefs.h
#ifndef _DSCUDADEFS_H
#define _DSCUDADEFS_H

#define RC_NSERVERMAX 32    
#define RC_NDEVICEMAX 32    
#define RC_NREDUNDANCYMAX 4 
#define RC_NVDEVMAX 64      
#define RC_NPTHREADMAX 64   

#define RC_BUFSIZE (1024*1024) 
#define RC_NKMODULEMAX 128  
#define RC_NKFUNCMAX   128  
#define RC_KARGMAX     64   
#define RC_KMODULENAMELEN 64   
#define RC_KNAMELEN       64   
#define RC_KMODULEIMAGELEN (1024*1024*2)   
#define RC_SNAMELEN       64   

#define RC_CACHE_MODULE (1) 
#define RC_CLIENT_CACHE_LIFETIME (30) 
#define RC_SERVER_CACHE_LIFETIME (RC_CLIENT_CACHE_LIFETIME+30) 

#define RC_SUPPORT_PAGELOCK (0)  
#define RC_SUPPORT_STREAM (0)
#define RC_SUPPORT_CONCURRENT_EXEC (0)

#define RC_DAEMON_IP_PORT  (65432)
#define RC_SERVER_IP_PORT  (RC_DAEMON_IP_PORT+1)

#endif 
#pragma end dscudadefs.h
#pragma begin dscudarpc.h


#ifndef _DSCUDARPC_H_RPCGEN
#define _DSCUDARPC_H_RPCGEN

#include <rpc/rpc.h>


#ifdef __cplusplus
extern "C" {
#endif


typedef u_quad_t RCadr;

typedef u_quad_t RCstream;

typedef u_quad_t RCevent;

typedef u_quad_t RCipaddr;

typedef u_int RCsize;

typedef u_int RCerror;

typedef struct {
	u_int RCbuf_len;
	char *RCbuf_val;
} RCbuf;

typedef u_int RCchannelformat;

typedef u_long RCpid;

struct RCchanneldesc_t {
	RCchannelformat f;
	int w;
	int x;
	int y;
	int z;
};
typedef struct RCchanneldesc_t RCchanneldesc_t;

typedef RCchanneldesc_t RCchanneldesc;

struct RCtexture_t {
	int normalized;
	int filterMode;
	int addressMode[3];
	RCchannelformat f;
	int w;
	int x;
	int y;
	int z;
};
typedef struct RCtexture_t RCtexture_t;

typedef RCtexture_t RCtexture;

struct RCfuncattr_t {
	int binaryVersion;
	RCsize constSizeBytes;
	RCsize localSizeBytes;
	int maxThreadsPerBlock;
	int numRegs;
	int ptxVersion;
	RCsize sharedSizeBytes;
};
typedef struct RCfuncattr_t RCfuncattr_t;

typedef RCfuncattr_t RCfuncattr;

enum RCargType {
	dscudaArgTypeP = 0,
	dscudaArgTypeI = 1,
	dscudaArgTypeF = 2,
	dscudaArgTypeV = 3,
};
typedef enum RCargType RCargType;

struct RCargVal {
	RCargType type;
	union {
		RCadr address;
		u_int valuei;
		float valuef;
		char valuev[64];
	} RCargVal_u;
};
typedef struct RCargVal RCargVal;

struct RCarg {
	RCargVal val;
	u_int offset;
	u_int size;
};
typedef struct RCarg RCarg;

typedef struct {
	u_int RCargs_len;
	RCarg *RCargs_val;
} RCargs;

struct dscudaResult {
	RCerror err;
};
typedef struct dscudaResult dscudaResult;

struct dscudaThreadGetLimitResult {
	RCerror err;
	RCsize value;
};
typedef struct dscudaThreadGetLimitResult dscudaThreadGetLimitResult;

struct dscudaThreadGetCacheConfigResult {
	RCerror err;
	int cacheConfig;
};
typedef struct dscudaThreadGetCacheConfigResult dscudaThreadGetCacheConfigResult;

struct dscudaMallocResult {
	RCerror err;
	RCadr devAdr;
};
typedef struct dscudaMallocResult dscudaMallocResult;

struct dscudaHostAllocResult {
	RCerror err;
	RCadr pHost;
};
typedef struct dscudaHostAllocResult dscudaHostAllocResult;

struct dscudaMallocHostResult {
	RCerror err;
	RCadr ptr;
};
typedef struct dscudaMallocHostResult dscudaMallocHostResult;

struct dscudaMallocArrayResult {
	RCerror err;
	RCadr array;
};
typedef struct dscudaMallocArrayResult dscudaMallocArrayResult;

struct dscudaMallocPitchResult {
	RCerror err;
	RCadr devPtr;
	RCsize pitch;
};
typedef struct dscudaMallocPitchResult dscudaMallocPitchResult;

struct dscudaMemcpyD2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyD2HResult dscudaMemcpyD2HResult;

struct dscudaMemcpyH2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyH2HResult dscudaMemcpyH2HResult;

struct dscudaMemcpyToArrayD2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyToArrayD2HResult dscudaMemcpyToArrayD2HResult;

struct dscudaMemcpyToArrayH2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyToArrayH2HResult dscudaMemcpyToArrayH2HResult;

struct dscudaMemcpy2DToArrayD2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpy2DToArrayD2HResult dscudaMemcpy2DToArrayD2HResult;

struct dscudaMemcpy2DToArrayH2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpy2DToArrayH2HResult dscudaMemcpy2DToArrayH2HResult;

struct dscudaMemcpy2DD2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpy2DD2HResult dscudaMemcpy2DD2HResult;

struct dscudaMemcpy2DH2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpy2DH2HResult dscudaMemcpy2DH2HResult;

struct dscudaGetDeviceResult {
	RCerror err;
	int device;
};
typedef struct dscudaGetDeviceResult dscudaGetDeviceResult;

struct dscudaGetDeviceCountResult {
	RCerror err;
	int count;
};
typedef struct dscudaGetDeviceCountResult dscudaGetDeviceCountResult;

struct dscudaGetDevicePropertiesResult {
	RCerror err;
	RCbuf prop;
};
typedef struct dscudaGetDevicePropertiesResult dscudaGetDevicePropertiesResult;

struct dscudaDriverGetVersionResult {
	RCerror err;
	int ver;
};
typedef struct dscudaDriverGetVersionResult dscudaDriverGetVersionResult;

struct dscudaRuntimeGetVersionResult {
	RCerror err;
	int ver;
};
typedef struct dscudaRuntimeGetVersionResult dscudaRuntimeGetVersionResult;

struct dscudaGetErrorStringResult {
	char *errmsg;
};
typedef struct dscudaGetErrorStringResult dscudaGetErrorStringResult;

struct dscudaCreateChannelDescResult {
	int x;
	int y;
	int z;
	int w;
	RCchannelformat f;
};
typedef struct dscudaCreateChannelDescResult dscudaCreateChannelDescResult;

struct dscudaGetChannelDescResult {
	RCerror err;
	int x;
	int y;
	int z;
	int w;
	RCchannelformat f;
};
typedef struct dscudaGetChannelDescResult dscudaGetChannelDescResult;

struct dscudaChooseDeviceResult {
	RCerror err;
	int device;
};
typedef struct dscudaChooseDeviceResult dscudaChooseDeviceResult;

struct dscudaMemcpyAsyncD2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyAsyncD2HResult dscudaMemcpyAsyncD2HResult;

struct dscudaMemcpyAsyncH2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyAsyncH2HResult dscudaMemcpyAsyncH2HResult;

struct dscudaMemcpyFromSymbolD2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyFromSymbolD2HResult dscudaMemcpyFromSymbolD2HResult;

struct dscudaMemcpyFromSymbolAsyncD2HResult {
	RCerror err;
	RCbuf buf;
};
typedef struct dscudaMemcpyFromSymbolAsyncD2HResult dscudaMemcpyFromSymbolAsyncD2HResult;

struct dscudaStreamCreateResult {
	RCerror err;
	RCadr stream;
};
typedef struct dscudaStreamCreateResult dscudaStreamCreateResult;

struct dscudaEventCreateResult {
	RCerror err;
	RCadr event;
};
typedef struct dscudaEventCreateResult dscudaEventCreateResult;

struct dscudaEventElapsedTimeResult {
	RCerror err;
	float ms;
};
typedef struct dscudaEventElapsedTimeResult dscudaEventElapsedTimeResult;

struct dscudaHostGetDevicePointerResult {
	RCerror err;
	RCadr pDevice;
};
typedef struct dscudaHostGetDevicePointerResult dscudaHostGetDevicePointerResult;

struct dscudaHostGetFlagsResult {
	RCerror err;
	u_int flags;
};
typedef struct dscudaHostGetFlagsResult dscudaHostGetFlagsResult;

struct dscudaLoadModuleResult {
	u_int id;
};
typedef struct dscudaLoadModuleResult dscudaLoadModuleResult;

struct dscudaFuncGetAttributesResult {
	RCerror err;
	RCfuncattr attr;
};
typedef struct dscudaFuncGetAttributesResult dscudaFuncGetAttributesResult;

struct dscudaBindTextureResult {
	RCerror err;
	RCsize offset;
};
typedef struct dscudaBindTextureResult dscudaBindTextureResult;

struct dscudaBindTexture2DResult {
	RCerror err;
	RCsize offset;
};
typedef struct dscudaBindTexture2DResult dscudaBindTexture2DResult;

struct dscufftResult {
	RCerror err;
};
typedef struct dscufftResult dscufftResult;

struct dscufftPlanResult {
	RCerror err;
	u_int plan;
};
typedef struct dscufftPlanResult dscufftPlanResult;

struct dscublasResult {
	RCerror err;
	u_int stat;
};
typedef struct dscublasResult dscublasResult;

struct dscublasCreateResult {
	RCerror err;
	u_int stat;
	RCadr handle;
};
typedef struct dscublasCreateResult dscublasCreateResult;

struct dscublasGetVectorResult {
	RCerror err;
	u_int stat;
	RCbuf y;
};
typedef struct dscublasGetVectorResult dscublasGetVectorResult;

struct RCdim3 {
	u_int x;
	u_int y;
	u_int z;
};
typedef struct RCdim3 RCdim3;

struct dscudathreadsetlimitid_1_argument {
	int limit;
	RCsize value;
};
typedef struct dscudathreadsetlimitid_1_argument dscudathreadsetlimitid_1_argument;

struct dscudastreamwaiteventid_1_argument {
	RCstream stream;
	RCevent event;
	u_int flags;
};
typedef struct dscudastreamwaiteventid_1_argument dscudastreamwaiteventid_1_argument;

struct dscudaeventelapsedtimeid_1_argument {
	RCevent start;
	RCevent end;
};
typedef struct dscudaeventelapsedtimeid_1_argument dscudaeventelapsedtimeid_1_argument;

struct dscudaeventrecordid_1_argument {
	RCevent event;
	RCstream stream;
};
typedef struct dscudaeventrecordid_1_argument dscudaeventrecordid_1_argument;

struct dscudalaunchkernelid_1_argument {
	int moduleid;
	int kid;
	char *kname;
	RCdim3 gdim;
	RCdim3 bdim;
	RCsize smemsize;
	RCstream stream;
	RCargs args;
};
typedef struct dscudalaunchkernelid_1_argument dscudalaunchkernelid_1_argument;

struct dscudaloadmoduleid_1_argument {
	RCipaddr ipaddr;
	RCpid pid;
	char *mname;
	char *image;
};
typedef struct dscudaloadmoduleid_1_argument dscudaloadmoduleid_1_argument;

struct dscudafuncgetattributesid_1_argument {
	int moduleid;
	char *kname;
};
typedef struct dscudafuncgetattributesid_1_argument dscudafuncgetattributesid_1_argument;

struct dscudamemcpyh2hid_1_argument {
	RCadr dst;
	RCbuf src;
	RCsize count;
};
typedef struct dscudamemcpyh2hid_1_argument dscudamemcpyh2hid_1_argument;

struct dscudamemcpyh2did_1_argument {
	RCadr dst;
	RCbuf src;
	RCsize count;
};
typedef struct dscudamemcpyh2did_1_argument dscudamemcpyh2did_1_argument;

struct dscudamemcpyd2hid_1_argument {
	RCadr src;
	RCsize count;
};
typedef struct dscudamemcpyd2hid_1_argument dscudamemcpyd2hid_1_argument;

struct dscudamemcpyd2did_1_argument {
	RCadr dst;
	RCadr src;
	RCsize count;
};
typedef struct dscudamemcpyd2did_1_argument dscudamemcpyd2did_1_argument;

struct dscudamemcpyasynch2hid_1_argument {
	RCadr dst;
	RCbuf src;
	RCsize count;
	RCstream stream;
};
typedef struct dscudamemcpyasynch2hid_1_argument dscudamemcpyasynch2hid_1_argument;

struct dscudamemcpyasynch2did_1_argument {
	RCadr dst;
	RCbuf src;
	RCsize count;
	RCstream stream;
};
typedef struct dscudamemcpyasynch2did_1_argument dscudamemcpyasynch2did_1_argument;

struct dscudamemcpyasyncd2hid_1_argument {
	RCadr src;
	RCsize count;
	RCstream stream;
};
typedef struct dscudamemcpyasyncd2hid_1_argument dscudamemcpyasyncd2hid_1_argument;

struct dscudamemcpyasyncd2did_1_argument {
	RCadr dst;
	RCadr src;
	RCsize count;
	RCstream stream;
};
typedef struct dscudamemcpyasyncd2did_1_argument dscudamemcpyasyncd2did_1_argument;

struct dscudamemcpytosymbolh2did_1_argument {
	int moduleid;
	char *symbol;
	RCbuf src;
	RCsize count;
	RCsize offset;
};
typedef struct dscudamemcpytosymbolh2did_1_argument dscudamemcpytosymbolh2did_1_argument;

struct dscudamemcpytosymbold2did_1_argument {
	int moduleid;
	char *symbol;
	RCadr src;
	RCsize count;
	RCsize offset;
};
typedef struct dscudamemcpytosymbold2did_1_argument dscudamemcpytosymbold2did_1_argument;

struct dscudamemcpyfromsymbold2hid_1_argument {
	int moduleid;
	char *symbol;
	RCsize count;
	RCsize offset;
};
typedef struct dscudamemcpyfromsymbold2hid_1_argument dscudamemcpyfromsymbold2hid_1_argument;

struct dscudamemcpyfromsymbold2did_1_argument {
	int moduleid;
	RCadr dst;
	char *symbol;
	RCsize count;
	RCsize offset;
};
typedef struct dscudamemcpyfromsymbold2did_1_argument dscudamemcpyfromsymbold2did_1_argument;

struct dscudamemsetid_1_argument {
	RCadr dst;
	int value;
	RCsize count;
};
typedef struct dscudamemsetid_1_argument dscudamemsetid_1_argument;

struct dscudahostallocid_1_argument {
	RCsize size;
	u_int flags;
};
typedef struct dscudahostallocid_1_argument dscudahostallocid_1_argument;

struct dscudahostgetdevicepointerid_1_argument {
	RCadr pHost;
	u_int flags;
};
typedef struct dscudahostgetdevicepointerid_1_argument dscudahostgetdevicepointerid_1_argument;

struct dscudamallocarrayid_1_argument {
	RCchanneldesc desc;
	RCsize width;
	RCsize height;
	u_int flags;
};
typedef struct dscudamallocarrayid_1_argument dscudamallocarrayid_1_argument;

struct dscudamemcpytoarrayh2hid_1_argument {
	RCadr dst;
	RCsize wOffset;
	RCsize hOffset;
	RCbuf src;
	RCsize count;
};
typedef struct dscudamemcpytoarrayh2hid_1_argument dscudamemcpytoarrayh2hid_1_argument;

struct dscudamemcpytoarrayh2did_1_argument {
	RCadr dst;
	RCsize wOffset;
	RCsize hOffset;
	RCbuf src;
	RCsize count;
};
typedef struct dscudamemcpytoarrayh2did_1_argument dscudamemcpytoarrayh2did_1_argument;

struct dscudamemcpytoarrayd2hid_1_argument {
	RCsize wOffset;
	RCsize hOffset;
	RCadr src;
	RCsize count;
};
typedef struct dscudamemcpytoarrayd2hid_1_argument dscudamemcpytoarrayd2hid_1_argument;

struct dscudamemcpytoarrayd2did_1_argument {
	RCadr dst;
	RCsize wOffset;
	RCsize hOffset;
	RCadr src;
	RCsize count;
};
typedef struct dscudamemcpytoarrayd2did_1_argument dscudamemcpytoarrayd2did_1_argument;

struct dscudamallocpitchid_1_argument {
	RCsize width;
	RCsize height;
};
typedef struct dscudamallocpitchid_1_argument dscudamallocpitchid_1_argument;

struct dscudamemcpy2dtoarrayh2hid_1_argument {
	RCadr dst;
	RCsize wOffset;
	RCsize hOffset;
	RCbuf src;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dtoarrayh2hid_1_argument dscudamemcpy2dtoarrayh2hid_1_argument;

struct dscudamemcpy2dtoarrayh2did_1_argument {
	RCadr dst;
	RCsize wOffset;
	RCsize hOffset;
	RCbuf srcbuf;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dtoarrayh2did_1_argument dscudamemcpy2dtoarrayh2did_1_argument;

struct dscudamemcpy2dtoarrayd2hid_1_argument {
	RCsize wOffset;
	RCsize hOffset;
	RCadr src;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dtoarrayd2hid_1_argument dscudamemcpy2dtoarrayd2hid_1_argument;

struct dscudamemcpy2dtoarrayd2did_1_argument {
	RCadr dst;
	RCsize wOffset;
	RCsize hOffset;
	RCadr src;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dtoarrayd2did_1_argument dscudamemcpy2dtoarrayd2did_1_argument;

struct dscudamemcpy2dh2hid_1_argument {
	RCadr dst;
	RCsize dpitch;
	RCbuf src;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dh2hid_1_argument dscudamemcpy2dh2hid_1_argument;

struct dscudamemcpy2dh2did_1_argument {
	RCadr dst;
	RCsize dpitch;
	RCbuf src;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dh2did_1_argument dscudamemcpy2dh2did_1_argument;

struct dscudamemcpy2dd2hid_1_argument {
	RCsize dpitch;
	RCadr src;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dd2hid_1_argument dscudamemcpy2dd2hid_1_argument;

struct dscudamemcpy2dd2did_1_argument {
	RCadr dst;
	RCsize dpitch;
	RCadr src;
	RCsize spitch;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemcpy2dd2did_1_argument dscudamemcpy2dd2did_1_argument;

struct dscudamemset2did_1_argument {
	RCadr dst;
	RCsize pitch;
	int value;
	RCsize width;
	RCsize height;
};
typedef struct dscudamemset2did_1_argument dscudamemset2did_1_argument;

struct dscudamemcpytosymbolasynch2did_1_argument {
	int moduleid;
	char *symbol;
	RCbuf src;
	RCsize count;
	RCsize offset;
	RCstream stream;
};
typedef struct dscudamemcpytosymbolasynch2did_1_argument dscudamemcpytosymbolasynch2did_1_argument;

struct dscudamemcpytosymbolasyncd2did_1_argument {
	int moduleid;
	char *symbol;
	RCadr src;
	RCsize count;
	RCsize offset;
	RCstream stream;
};
typedef struct dscudamemcpytosymbolasyncd2did_1_argument dscudamemcpytosymbolasyncd2did_1_argument;

struct dscudamemcpyfromsymbolasyncd2hid_1_argument {
	int moduleid;
	char *symbol;
	RCsize count;
	RCsize offset;
	RCstream stream;
};
typedef struct dscudamemcpyfromsymbolasyncd2hid_1_argument dscudamemcpyfromsymbolasyncd2hid_1_argument;

struct dscudamemcpyfromsymbolasyncd2did_1_argument {
	int moduleid;
	RCadr dst;
	char *symbol;
	RCsize count;
	RCsize offset;
	RCstream stream;
};
typedef struct dscudamemcpyfromsymbolasyncd2did_1_argument dscudamemcpyfromsymbolasyncd2did_1_argument;

struct dscudacreatechanneldescid_1_argument {
	int x;
	int y;
	int z;
	int w;
	RCchannelformat f;
};
typedef struct dscudacreatechanneldescid_1_argument dscudacreatechanneldescid_1_argument;

struct dscudabindtextureid_1_argument {
	int moduleid;
	char *texname;
	RCadr devPtr;
	RCsize size;
	RCtexture texbuf;
};
typedef struct dscudabindtextureid_1_argument dscudabindtextureid_1_argument;

struct dscudabindtexture2did_1_argument {
	int moduleid;
	char *texname;
	RCadr devPtr;
	RCsize width;
	RCsize height;
	RCsize pitch;
	RCtexture texbuf;
};
typedef struct dscudabindtexture2did_1_argument dscudabindtexture2did_1_argument;

struct dscudabindtexturetoarrayid_1_argument {
	int moduleid;
	char *texname;
	RCadr array;
	RCtexture texbuf;
};
typedef struct dscudabindtexturetoarrayid_1_argument dscudabindtexturetoarrayid_1_argument;

struct dscufftplan3did_1_argument {
	int nx;
	int ny;
	int nz;
	u_int type;
};
typedef struct dscufftplan3did_1_argument dscufftplan3did_1_argument;

struct dscufftexecc2cid_1_argument {
	u_int plan;
	RCadr idata;
	RCadr odata;
	int direction;
};
typedef struct dscufftexecc2cid_1_argument dscufftexecc2cid_1_argument;

#define DSCUDA_PROG 60000
#define DSCUDA_VER 1

#if defined(__STDC__) || defined(__cplusplus)
#define dscudaThreadExitId 100
extern  dscudaResult * dscudathreadexitid_1(CLIENT *);
extern  dscudaResult * dscudathreadexitid_1_svc(struct svc_req *);
#define dscudaThreadSynchronizeId 101
extern  dscudaResult * dscudathreadsynchronizeid_1(CLIENT *);
extern  dscudaResult * dscudathreadsynchronizeid_1_svc(struct svc_req *);
#define dscudaThreadSetLimitId 102
extern  dscudaResult * dscudathreadsetlimitid_1(int , RCsize , CLIENT *);
extern  dscudaResult * dscudathreadsetlimitid_1_svc(int , RCsize , struct svc_req *);
#define dscudaThreadGetLimitId 103
extern  dscudaThreadGetLimitResult * dscudathreadgetlimitid_1(int , CLIENT *);
extern  dscudaThreadGetLimitResult * dscudathreadgetlimitid_1_svc(int , struct svc_req *);
#define dscudaThreadSetCacheConfigId 104
extern  dscudaResult * dscudathreadsetcacheconfigid_1(int , CLIENT *);
extern  dscudaResult * dscudathreadsetcacheconfigid_1_svc(int , struct svc_req *);
#define dscudaThreadGetCacheConfigId 105
extern  dscudaThreadGetCacheConfigResult * dscudathreadgetcacheconfigid_1(CLIENT *);
extern  dscudaThreadGetCacheConfigResult * dscudathreadgetcacheconfigid_1_svc(struct svc_req *);
#define dscudaGetLastErrorId 200
extern  dscudaResult * dscudagetlasterrorid_1(CLIENT *);
extern  dscudaResult * dscudagetlasterrorid_1_svc(struct svc_req *);
#define dscudaPeekAtLastErrorId 201
extern  dscudaResult * dscudapeekatlasterrorid_1(CLIENT *);
extern  dscudaResult * dscudapeekatlasterrorid_1_svc(struct svc_req *);
#define dscudaGetErrorStringId 202
extern  dscudaGetErrorStringResult * dscudageterrorstringid_1(int , CLIENT *);
extern  dscudaGetErrorStringResult * dscudageterrorstringid_1_svc(int , struct svc_req *);
#define dscudaGetDeviceId 300
extern  dscudaGetDeviceResult * dscudagetdeviceid_1(CLIENT *);
extern  dscudaGetDeviceResult * dscudagetdeviceid_1_svc(struct svc_req *);
#define dscudaGetDeviceCountId 301
extern  dscudaGetDeviceCountResult * dscudagetdevicecountid_1(CLIENT *);
extern  dscudaGetDeviceCountResult * dscudagetdevicecountid_1_svc(struct svc_req *);
#define dscudaGetDevicePropertiesId 302
extern  dscudaGetDevicePropertiesResult * dscudagetdevicepropertiesid_1(int , CLIENT *);
extern  dscudaGetDevicePropertiesResult * dscudagetdevicepropertiesid_1_svc(int , struct svc_req *);
#define dscudaDriverGetVersionId 303
extern  dscudaDriverGetVersionResult * dscudadrivergetversionid_1(CLIENT *);
extern  dscudaDriverGetVersionResult * dscudadrivergetversionid_1_svc(struct svc_req *);
#define dscudaRuntimeGetVersionId 304
extern  dscudaRuntimeGetVersionResult * dscudaruntimegetversionid_1(CLIENT *);
extern  dscudaRuntimeGetVersionResult * dscudaruntimegetversionid_1_svc(struct svc_req *);
#define dscudaSetDeviceId 305
extern  dscudaResult * dscudasetdeviceid_1(int , CLIENT *);
extern  dscudaResult * dscudasetdeviceid_1_svc(int , struct svc_req *);
#define dscudaSetDeviceFlagsId 306
extern  dscudaResult * dscudasetdeviceflagsid_1(u_int , CLIENT *);
extern  dscudaResult * dscudasetdeviceflagsid_1_svc(u_int , struct svc_req *);
#define dscudaChooseDeviceId 307
extern  dscudaChooseDeviceResult * dscudachoosedeviceid_1(RCbuf , CLIENT *);
extern  dscudaChooseDeviceResult * dscudachoosedeviceid_1_svc(RCbuf , struct svc_req *);
#define dscudaDeviceSynchronize 308
extern  dscudaResult * dscudadevicesynchronize_1(CLIENT *);
extern  dscudaResult * dscudadevicesynchronize_1_svc(struct svc_req *);
#define dscudaDeviceReset 309
extern  dscudaResult * dscudadevicereset_1(CLIENT *);
extern  dscudaResult * dscudadevicereset_1_svc(struct svc_req *);
#define dscudaStreamCreateId 400
extern  dscudaStreamCreateResult * dscudastreamcreateid_1(CLIENT *);
extern  dscudaStreamCreateResult * dscudastreamcreateid_1_svc(struct svc_req *);
#define dscudaStreamDestroyId 401
extern  dscudaResult * dscudastreamdestroyid_1(RCstream , CLIENT *);
extern  dscudaResult * dscudastreamdestroyid_1_svc(RCstream , struct svc_req *);
#define dscudaStreamSynchronizeId 402
extern  dscudaResult * dscudastreamsynchronizeid_1(RCstream , CLIENT *);
extern  dscudaResult * dscudastreamsynchronizeid_1_svc(RCstream , struct svc_req *);
#define dscudaStreamQueryId 403
extern  dscudaResult * dscudastreamqueryid_1(RCstream , CLIENT *);
extern  dscudaResult * dscudastreamqueryid_1_svc(RCstream , struct svc_req *);
#define dscudaStreamWaitEventId 404
extern  dscudaResult * dscudastreamwaiteventid_1(RCstream , RCevent , u_int , CLIENT *);
extern  dscudaResult * dscudastreamwaiteventid_1_svc(RCstream , RCevent , u_int , struct svc_req *);
#define dscudaEventCreateId 500
extern  dscudaEventCreateResult * dscudaeventcreateid_1(CLIENT *);
extern  dscudaEventCreateResult * dscudaeventcreateid_1_svc(struct svc_req *);
#define dscudaEventCreateWithFlagsId 501
extern  dscudaEventCreateResult * dscudaeventcreatewithflagsid_1(u_int , CLIENT *);
extern  dscudaEventCreateResult * dscudaeventcreatewithflagsid_1_svc(u_int , struct svc_req *);
#define dscudaEventDestroyId 502
extern  dscudaResult * dscudaeventdestroyid_1(RCevent , CLIENT *);
extern  dscudaResult * dscudaeventdestroyid_1_svc(RCevent , struct svc_req *);
#define dscudaEventElapsedTimeId 503
extern  dscudaEventElapsedTimeResult * dscudaeventelapsedtimeid_1(RCevent , RCevent , CLIENT *);
extern  dscudaEventElapsedTimeResult * dscudaeventelapsedtimeid_1_svc(RCevent , RCevent , struct svc_req *);
#define dscudaEventRecordId 504
extern  dscudaResult * dscudaeventrecordid_1(RCevent , RCstream , CLIENT *);
extern  dscudaResult * dscudaeventrecordid_1_svc(RCevent , RCstream , struct svc_req *);
#define dscudaEventSynchronizeId 505
extern  dscudaResult * dscudaeventsynchronizeid_1(RCevent , CLIENT *);
extern  dscudaResult * dscudaeventsynchronizeid_1_svc(RCevent , struct svc_req *);
#define dscudaEventQueryId 506
extern  dscudaResult * dscudaeventqueryid_1(RCevent , CLIENT *);
extern  dscudaResult * dscudaeventqueryid_1_svc(RCevent , struct svc_req *);
#define dscudaLaunchKernelId 600
extern  void * dscudalaunchkernelid_1(int , int , char *, RCdim3 , RCdim3 , RCsize , RCstream , RCargs , CLIENT *);
extern  void * dscudalaunchkernelid_1_svc(int , int , char *, RCdim3 , RCdim3 , RCsize , RCstream , RCargs , struct svc_req *);
#define dscudaLoadModuleId 601
extern  dscudaLoadModuleResult * dscudaloadmoduleid_1(RCipaddr , RCpid , char *, char *, CLIENT *);
extern  dscudaLoadModuleResult * dscudaloadmoduleid_1_svc(RCipaddr , RCpid , char *, char *, struct svc_req *);
#define dscudaFuncGetAttributesId 602
extern  dscudaFuncGetAttributesResult * dscudafuncgetattributesid_1(int , char *, CLIENT *);
extern  dscudaFuncGetAttributesResult * dscudafuncgetattributesid_1_svc(int , char *, struct svc_req *);
#define dscudaMallocId 700
extern  dscudaMallocResult * dscudamallocid_1(RCsize , CLIENT *);
extern  dscudaMallocResult * dscudamallocid_1_svc(RCsize , struct svc_req *);
#define dscudaFreeId 701
extern  dscudaResult * dscudafreeid_1(RCadr , CLIENT *);
extern  dscudaResult * dscudafreeid_1_svc(RCadr , struct svc_req *);
#define dscudaMemcpyH2HId 702
extern  dscudaMemcpyH2HResult * dscudamemcpyh2hid_1(RCadr , RCbuf , RCsize , CLIENT *);
extern  dscudaMemcpyH2HResult * dscudamemcpyh2hid_1_svc(RCadr , RCbuf , RCsize , struct svc_req *);
#define dscudaMemcpyH2DId 703
extern  dscudaResult * dscudamemcpyh2did_1(RCadr , RCbuf , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpyh2did_1_svc(RCadr , RCbuf , RCsize , struct svc_req *);
#define dscudaMemcpyD2HId 704
extern  dscudaMemcpyD2HResult * dscudamemcpyd2hid_1(RCadr , RCsize , CLIENT *);
extern  dscudaMemcpyD2HResult * dscudamemcpyd2hid_1_svc(RCadr , RCsize , struct svc_req *);
#define dscudaMemcpyD2DId 705
extern  dscudaResult * dscudamemcpyd2did_1(RCadr , RCadr , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpyd2did_1_svc(RCadr , RCadr , RCsize , struct svc_req *);
#define dscudaMemcpyAsyncH2HId 706
extern  dscudaMemcpyAsyncH2HResult * dscudamemcpyasynch2hid_1(RCadr , RCbuf , RCsize , RCstream , CLIENT *);
extern  dscudaMemcpyAsyncH2HResult * dscudamemcpyasynch2hid_1_svc(RCadr , RCbuf , RCsize , RCstream , struct svc_req *);
#define dscudaMemcpyAsyncH2DId 707
extern  dscudaResult * dscudamemcpyasynch2did_1(RCadr , RCbuf , RCsize , RCstream , CLIENT *);
extern  dscudaResult * dscudamemcpyasynch2did_1_svc(RCadr , RCbuf , RCsize , RCstream , struct svc_req *);
#define dscudaMemcpyAsyncD2HId 708
extern  dscudaMemcpyAsyncD2HResult * dscudamemcpyasyncd2hid_1(RCadr , RCsize , RCstream , CLIENT *);
extern  dscudaMemcpyAsyncD2HResult * dscudamemcpyasyncd2hid_1_svc(RCadr , RCsize , RCstream , struct svc_req *);
#define dscudaMemcpyAsyncD2DId 709
extern  dscudaResult * dscudamemcpyasyncd2did_1(RCadr , RCadr , RCsize , RCstream , CLIENT *);
extern  dscudaResult * dscudamemcpyasyncd2did_1_svc(RCadr , RCadr , RCsize , RCstream , struct svc_req *);
#define dscudaMemcpyToSymbolH2DId 710
extern  dscudaResult * dscudamemcpytosymbolh2did_1(int , char *, RCbuf , RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpytosymbolh2did_1_svc(int , char *, RCbuf , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpyToSymbolD2DId 711
extern  dscudaResult * dscudamemcpytosymbold2did_1(int , char *, RCadr , RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpytosymbold2did_1_svc(int , char *, RCadr , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpyFromSymbolD2HId 712
extern  dscudaMemcpyFromSymbolD2HResult * dscudamemcpyfromsymbold2hid_1(int , char *, RCsize , RCsize , CLIENT *);
extern  dscudaMemcpyFromSymbolD2HResult * dscudamemcpyfromsymbold2hid_1_svc(int , char *, RCsize , RCsize , struct svc_req *);
#define dscudaMemcpyFromSymbolD2DId 713
extern  dscudaResult * dscudamemcpyfromsymbold2did_1(int , RCadr , char *, RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpyfromsymbold2did_1_svc(int , RCadr , char *, RCsize , RCsize , struct svc_req *);
#define dscudaMemsetId 714
extern  dscudaResult * dscudamemsetid_1(RCadr , int , RCsize , CLIENT *);
extern  dscudaResult * dscudamemsetid_1_svc(RCadr , int , RCsize , struct svc_req *);
#define dscudaHostAllocId 715
extern  dscudaHostAllocResult * dscudahostallocid_1(RCsize , u_int , CLIENT *);
extern  dscudaHostAllocResult * dscudahostallocid_1_svc(RCsize , u_int , struct svc_req *);
#define dscudaMallocHostId 716
extern  dscudaMallocHostResult * dscudamallochostid_1(RCsize , CLIENT *);
extern  dscudaMallocHostResult * dscudamallochostid_1_svc(RCsize , struct svc_req *);
#define dscudaFreeHostId 717
extern  dscudaResult * dscudafreehostid_1(RCadr , CLIENT *);
extern  dscudaResult * dscudafreehostid_1_svc(RCadr , struct svc_req *);
#define dscudaHostGetDevicePointerId 718
extern  dscudaHostGetDevicePointerResult * dscudahostgetdevicepointerid_1(RCadr , u_int , CLIENT *);
extern  dscudaHostGetDevicePointerResult * dscudahostgetdevicepointerid_1_svc(RCadr , u_int , struct svc_req *);
#define dscudaHostGetFlagsID 719
extern  dscudaHostGetFlagsResult * dscudahostgetflagsid_1(RCadr , CLIENT *);
extern  dscudaHostGetFlagsResult * dscudahostgetflagsid_1_svc(RCadr , struct svc_req *);
#define dscudaMallocArrayId 720
extern  dscudaMallocArrayResult * dscudamallocarrayid_1(RCchanneldesc , RCsize , RCsize , u_int , CLIENT *);
extern  dscudaMallocArrayResult * dscudamallocarrayid_1_svc(RCchanneldesc , RCsize , RCsize , u_int , struct svc_req *);
#define dscudaFreeArrayId 721
extern  dscudaResult * dscudafreearrayid_1(RCadr , CLIENT *);
extern  dscudaResult * dscudafreearrayid_1_svc(RCadr , struct svc_req *);
#define dscudaMemcpyToArrayH2HId 722
extern  dscudaMemcpyToArrayH2HResult * dscudamemcpytoarrayh2hid_1(RCadr , RCsize , RCsize , RCbuf , RCsize , CLIENT *);
extern  dscudaMemcpyToArrayH2HResult * dscudamemcpytoarrayh2hid_1_svc(RCadr , RCsize , RCsize , RCbuf , RCsize , struct svc_req *);
#define dscudaMemcpyToArrayH2DId 723
extern  dscudaResult * dscudamemcpytoarrayh2did_1(RCadr , RCsize , RCsize , RCbuf , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpytoarrayh2did_1_svc(RCadr , RCsize , RCsize , RCbuf , RCsize , struct svc_req *);
#define dscudaMemcpyToArrayD2HId 724
extern  dscudaMemcpyToArrayD2HResult * dscudamemcpytoarrayd2hid_1(RCsize , RCsize , RCadr , RCsize , CLIENT *);
extern  dscudaMemcpyToArrayD2HResult * dscudamemcpytoarrayd2hid_1_svc(RCsize , RCsize , RCadr , RCsize , struct svc_req *);
#define dscudaMemcpyToArrayD2DId 725
extern  dscudaResult * dscudamemcpytoarrayd2did_1(RCadr , RCsize , RCsize , RCadr , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpytoarrayd2did_1_svc(RCadr , RCsize , RCsize , RCadr , RCsize , struct svc_req *);
#define dscudaMallocPitchId 726
extern  dscudaMallocPitchResult * dscudamallocpitchid_1(RCsize , RCsize , CLIENT *);
extern  dscudaMallocPitchResult * dscudamallocpitchid_1_svc(RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DToArrayH2HId 727
extern  dscudaMemcpy2DToArrayH2HResult * dscudamemcpy2dtoarrayh2hid_1(RCadr , RCsize , RCsize , RCbuf , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaMemcpy2DToArrayH2HResult * dscudamemcpy2dtoarrayh2hid_1_svc(RCadr , RCsize , RCsize , RCbuf , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DToArrayH2DId 728
extern  dscudaResult * dscudamemcpy2dtoarrayh2did_1(RCadr , RCsize , RCsize , RCbuf , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpy2dtoarrayh2did_1_svc(RCadr , RCsize , RCsize , RCbuf , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DToArrayD2HId 729
extern  dscudaMemcpy2DToArrayD2HResult * dscudamemcpy2dtoarrayd2hid_1(RCsize , RCsize , RCadr , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaMemcpy2DToArrayD2HResult * dscudamemcpy2dtoarrayd2hid_1_svc(RCsize , RCsize , RCadr , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DToArrayD2DId 730
extern  dscudaResult * dscudamemcpy2dtoarrayd2did_1(RCadr , RCsize , RCsize , RCadr , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpy2dtoarrayd2did_1_svc(RCadr , RCsize , RCsize , RCadr , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DH2HId 731
extern  dscudaMemcpy2DH2HResult * dscudamemcpy2dh2hid_1(RCadr , RCsize , RCbuf , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaMemcpy2DH2HResult * dscudamemcpy2dh2hid_1_svc(RCadr , RCsize , RCbuf , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DH2DId 732
extern  dscudaResult * dscudamemcpy2dh2did_1(RCadr , RCsize , RCbuf , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpy2dh2did_1_svc(RCadr , RCsize , RCbuf , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DD2HId 733
extern  dscudaMemcpy2DD2HResult * dscudamemcpy2dd2hid_1(RCsize , RCadr , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaMemcpy2DD2HResult * dscudamemcpy2dd2hid_1_svc(RCsize , RCadr , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpy2DD2DId 734
extern  dscudaResult * dscudamemcpy2dd2did_1(RCadr , RCsize , RCadr , RCsize , RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemcpy2dd2did_1_svc(RCadr , RCsize , RCadr , RCsize , RCsize , RCsize , struct svc_req *);
#define dscudaMemset2DId 735
extern  dscudaResult * dscudamemset2did_1(RCadr , RCsize , int , RCsize , RCsize , CLIENT *);
extern  dscudaResult * dscudamemset2did_1_svc(RCadr , RCsize , int , RCsize , RCsize , struct svc_req *);
#define dscudaMemcpyToSymbolAsyncH2DId 736
extern  dscudaResult * dscudamemcpytosymbolasynch2did_1(int , char *, RCbuf , RCsize , RCsize , RCstream , CLIENT *);
extern  dscudaResult * dscudamemcpytosymbolasynch2did_1_svc(int , char *, RCbuf , RCsize , RCsize , RCstream , struct svc_req *);
#define dscudaMemcpyToSymbolAsyncD2DId 737
extern  dscudaResult * dscudamemcpytosymbolasyncd2did_1(int , char *, RCadr , RCsize , RCsize , RCstream , CLIENT *);
extern  dscudaResult * dscudamemcpytosymbolasyncd2did_1_svc(int , char *, RCadr , RCsize , RCsize , RCstream , struct svc_req *);
#define dscudaMemcpyFromSymbolAsyncD2HId 738
extern  dscudaMemcpyFromSymbolAsyncD2HResult * dscudamemcpyfromsymbolasyncd2hid_1(int , char *, RCsize , RCsize , RCstream , CLIENT *);
extern  dscudaMemcpyFromSymbolAsyncD2HResult * dscudamemcpyfromsymbolasyncd2hid_1_svc(int , char *, RCsize , RCsize , RCstream , struct svc_req *);
#define dscudaMemcpyFromSymbolAsyncD2DId 739
extern  dscudaResult * dscudamemcpyfromsymbolasyncd2did_1(int , RCadr , char *, RCsize , RCsize , RCstream , CLIENT *);
extern  dscudaResult * dscudamemcpyfromsymbolasyncd2did_1_svc(int , RCadr , char *, RCsize , RCsize , RCstream , struct svc_req *);
#define dscudaCreateChannelDescId 1400
extern  dscudaCreateChannelDescResult * dscudacreatechanneldescid_1(int , int , int , int , RCchannelformat , CLIENT *);
extern  dscudaCreateChannelDescResult * dscudacreatechanneldescid_1_svc(int , int , int , int , RCchannelformat , struct svc_req *);
#define dscudaGetChannelDescId 1401
extern  dscudaGetChannelDescResult * dscudagetchanneldescid_1(RCadr , CLIENT *);
extern  dscudaGetChannelDescResult * dscudagetchanneldescid_1_svc(RCadr , struct svc_req *);
#define dscudaBindTextureId 1402
extern  dscudaBindTextureResult * dscudabindtextureid_1(int , char *, RCadr , RCsize , RCtexture , CLIENT *);
extern  dscudaBindTextureResult * dscudabindtextureid_1_svc(int , char *, RCadr , RCsize , RCtexture , struct svc_req *);
#define dscudaBindTexture2DId 1403
extern  dscudaBindTexture2DResult * dscudabindtexture2did_1(int , char *, RCadr , RCsize , RCsize , RCsize , RCtexture , CLIENT *);
extern  dscudaBindTexture2DResult * dscudabindtexture2did_1_svc(int , char *, RCadr , RCsize , RCsize , RCsize , RCtexture , struct svc_req *);
#define dscudaBindTextureToArrayId 1404
extern  dscudaResult * dscudabindtexturetoarrayid_1(int , char *, RCadr , RCtexture , CLIENT *);
extern  dscudaResult * dscudabindtexturetoarrayid_1_svc(int , char *, RCadr , RCtexture , struct svc_req *);
#define dscudaUnbindTextureId 1405
extern  dscudaResult * dscudaunbindtextureid_1(RCtexture , CLIENT *);
extern  dscudaResult * dscudaunbindtextureid_1_svc(RCtexture , struct svc_req *);
#define dscufftPlan3dId 2002
extern  dscufftPlanResult * dscufftplan3did_1(int , int , int , u_int , CLIENT *);
extern  dscufftPlanResult * dscufftplan3did_1_svc(int , int , int , u_int , struct svc_req *);
#define dscufftDestroyId 2004
extern  dscufftResult * dscufftdestroyid_1(u_int , CLIENT *);
extern  dscufftResult * dscufftdestroyid_1_svc(u_int , struct svc_req *);
#define dscufftExecC2CId 2005
extern  dscufftResult * dscufftexecc2cid_1(u_int , RCadr , RCadr , int , CLIENT *);
extern  dscufftResult * dscufftexecc2cid_1_svc(u_int , RCadr , RCadr , int , struct svc_req *);
extern int dscuda_prog_1_freeresult (SVCXPRT *, xdrproc_t, caddr_t);

#else 
#define dscudaThreadExitId 100
extern  dscudaResult * dscudathreadexitid_1();
extern  dscudaResult * dscudathreadexitid_1_svc();
#define dscudaThreadSynchronizeId 101
extern  dscudaResult * dscudathreadsynchronizeid_1();
extern  dscudaResult * dscudathreadsynchronizeid_1_svc();
#define dscudaThreadSetLimitId 102
extern  dscudaResult * dscudathreadsetlimitid_1();
extern  dscudaResult * dscudathreadsetlimitid_1_svc();
#define dscudaThreadGetLimitId 103
extern  dscudaThreadGetLimitResult * dscudathreadgetlimitid_1();
extern  dscudaThreadGetLimitResult * dscudathreadgetlimitid_1_svc();
#define dscudaThreadSetCacheConfigId 104
extern  dscudaResult * dscudathreadsetcacheconfigid_1();
extern  dscudaResult * dscudathreadsetcacheconfigid_1_svc();
#define dscudaThreadGetCacheConfigId 105
extern  dscudaThreadGetCacheConfigResult * dscudathreadgetcacheconfigid_1();
extern  dscudaThreadGetCacheConfigResult * dscudathreadgetcacheconfigid_1_svc();
#define dscudaGetLastErrorId 200
extern  dscudaResult * dscudagetlasterrorid_1();
extern  dscudaResult * dscudagetlasterrorid_1_svc();
#define dscudaPeekAtLastErrorId 201
extern  dscudaResult * dscudapeekatlasterrorid_1();
extern  dscudaResult * dscudapeekatlasterrorid_1_svc();
#define dscudaGetErrorStringId 202
extern  dscudaGetErrorStringResult * dscudageterrorstringid_1();
extern  dscudaGetErrorStringResult * dscudageterrorstringid_1_svc();
#define dscudaGetDeviceId 300
extern  dscudaGetDeviceResult * dscudagetdeviceid_1();
extern  dscudaGetDeviceResult * dscudagetdeviceid_1_svc();
#define dscudaGetDeviceCountId 301
extern  dscudaGetDeviceCountResult * dscudagetdevicecountid_1();
extern  dscudaGetDeviceCountResult * dscudagetdevicecountid_1_svc();
#define dscudaGetDevicePropertiesId 302
extern  dscudaGetDevicePropertiesResult * dscudagetdevicepropertiesid_1();
extern  dscudaGetDevicePropertiesResult * dscudagetdevicepropertiesid_1_svc();
#define dscudaDriverGetVersionId 303
extern  dscudaDriverGetVersionResult * dscudadrivergetversionid_1();
extern  dscudaDriverGetVersionResult * dscudadrivergetversionid_1_svc();
#define dscudaRuntimeGetVersionId 304
extern  dscudaRuntimeGetVersionResult * dscudaruntimegetversionid_1();
extern  dscudaRuntimeGetVersionResult * dscudaruntimegetversionid_1_svc();
#define dscudaSetDeviceId 305
extern  dscudaResult * dscudasetdeviceid_1();
extern  dscudaResult * dscudasetdeviceid_1_svc();
#define dscudaSetDeviceFlagsId 306
extern  dscudaResult * dscudasetdeviceflagsid_1();
extern  dscudaResult * dscudasetdeviceflagsid_1_svc();
#define dscudaChooseDeviceId 307
extern  dscudaChooseDeviceResult * dscudachoosedeviceid_1();
extern  dscudaChooseDeviceResult * dscudachoosedeviceid_1_svc();
#define dscudaDeviceSynchronize 308
extern  dscudaResult * dscudadevicesynchronize_1();
extern  dscudaResult * dscudadevicesynchronize_1_svc();
#define dscudaDeviceReset 309
extern  dscudaResult * dscudadevicereset_1();
extern  dscudaResult * dscudadevicereset_1_svc();
#define dscudaStreamCreateId 400
extern  dscudaStreamCreateResult * dscudastreamcreateid_1();
extern  dscudaStreamCreateResult * dscudastreamcreateid_1_svc();
#define dscudaStreamDestroyId 401
extern  dscudaResult * dscudastreamdestroyid_1();
extern  dscudaResult * dscudastreamdestroyid_1_svc();
#define dscudaStreamSynchronizeId 402
extern  dscudaResult * dscudastreamsynchronizeid_1();
extern  dscudaResult * dscudastreamsynchronizeid_1_svc();
#define dscudaStreamQueryId 403
extern  dscudaResult * dscudastreamqueryid_1();
extern  dscudaResult * dscudastreamqueryid_1_svc();
#define dscudaStreamWaitEventId 404
extern  dscudaResult * dscudastreamwaiteventid_1();
extern  dscudaResult * dscudastreamwaiteventid_1_svc();
#define dscudaEventCreateId 500
extern  dscudaEventCreateResult * dscudaeventcreateid_1();
extern  dscudaEventCreateResult * dscudaeventcreateid_1_svc();
#define dscudaEventCreateWithFlagsId 501
extern  dscudaEventCreateResult * dscudaeventcreatewithflagsid_1();
extern  dscudaEventCreateResult * dscudaeventcreatewithflagsid_1_svc();
#define dscudaEventDestroyId 502
extern  dscudaResult * dscudaeventdestroyid_1();
extern  dscudaResult * dscudaeventdestroyid_1_svc();
#define dscudaEventElapsedTimeId 503
extern  dscudaEventElapsedTimeResult * dscudaeventelapsedtimeid_1();
extern  dscudaEventElapsedTimeResult * dscudaeventelapsedtimeid_1_svc();
#define dscudaEventRecordId 504
extern  dscudaResult * dscudaeventrecordid_1();
extern  dscudaResult * dscudaeventrecordid_1_svc();
#define dscudaEventSynchronizeId 505
extern  dscudaResult * dscudaeventsynchronizeid_1();
extern  dscudaResult * dscudaeventsynchronizeid_1_svc();
#define dscudaEventQueryId 506
extern  dscudaResult * dscudaeventqueryid_1();
extern  dscudaResult * dscudaeventqueryid_1_svc();
#define dscudaLaunchKernelId 600
extern  void * dscudalaunchkernelid_1();
extern  void * dscudalaunchkernelid_1_svc();
#define dscudaLoadModuleId 601
extern  dscudaLoadModuleResult * dscudaloadmoduleid_1();
extern  dscudaLoadModuleResult * dscudaloadmoduleid_1_svc();
#define dscudaFuncGetAttributesId 602
extern  dscudaFuncGetAttributesResult * dscudafuncgetattributesid_1();
extern  dscudaFuncGetAttributesResult * dscudafuncgetattributesid_1_svc();
#define dscudaMallocId 700
extern  dscudaMallocResult * dscudamallocid_1();
extern  dscudaMallocResult * dscudamallocid_1_svc();
#define dscudaFreeId 701
extern  dscudaResult * dscudafreeid_1();
extern  dscudaResult * dscudafreeid_1_svc();
#define dscudaMemcpyH2HId 702
extern  dscudaMemcpyH2HResult * dscudamemcpyh2hid_1();
extern  dscudaMemcpyH2HResult * dscudamemcpyh2hid_1_svc();
#define dscudaMemcpyH2DId 703
extern  dscudaResult * dscudamemcpyh2did_1();
extern  dscudaResult * dscudamemcpyh2did_1_svc();
#define dscudaMemcpyD2HId 704
extern  dscudaMemcpyD2HResult * dscudamemcpyd2hid_1();
extern  dscudaMemcpyD2HResult * dscudamemcpyd2hid_1_svc();
#define dscudaMemcpyD2DId 705
extern  dscudaResult * dscudamemcpyd2did_1();
extern  dscudaResult * dscudamemcpyd2did_1_svc();
#define dscudaMemcpyAsyncH2HId 706
extern  dscudaMemcpyAsyncH2HResult * dscudamemcpyasynch2hid_1();
extern  dscudaMemcpyAsyncH2HResult * dscudamemcpyasynch2hid_1_svc();
#define dscudaMemcpyAsyncH2DId 707
extern  dscudaResult * dscudamemcpyasynch2did_1();
extern  dscudaResult * dscudamemcpyasynch2did_1_svc();
#define dscudaMemcpyAsyncD2HId 708
extern  dscudaMemcpyAsyncD2HResult * dscudamemcpyasyncd2hid_1();
extern  dscudaMemcpyAsyncD2HResult * dscudamemcpyasyncd2hid_1_svc();
#define dscudaMemcpyAsyncD2DId 709
extern  dscudaResult * dscudamemcpyasyncd2did_1();
extern  dscudaResult * dscudamemcpyasyncd2did_1_svc();
#define dscudaMemcpyToSymbolH2DId 710
extern  dscudaResult * dscudamemcpytosymbolh2did_1();
extern  dscudaResult * dscudamemcpytosymbolh2did_1_svc();
#define dscudaMemcpyToSymbolD2DId 711
extern  dscudaResult * dscudamemcpytosymbold2did_1();
extern  dscudaResult * dscudamemcpytosymbold2did_1_svc();
#define dscudaMemcpyFromSymbolD2HId 712
extern  dscudaMemcpyFromSymbolD2HResult * dscudamemcpyfromsymbold2hid_1();
extern  dscudaMemcpyFromSymbolD2HResult * dscudamemcpyfromsymbold2hid_1_svc();
#define dscudaMemcpyFromSymbolD2DId 713
extern  dscudaResult * dscudamemcpyfromsymbold2did_1();
extern  dscudaResult * dscudamemcpyfromsymbold2did_1_svc();
#define dscudaMemsetId 714
extern  dscudaResult * dscudamemsetid_1();
extern  dscudaResult * dscudamemsetid_1_svc();
#define dscudaHostAllocId 715
extern  dscudaHostAllocResult * dscudahostallocid_1();
extern  dscudaHostAllocResult * dscudahostallocid_1_svc();
#define dscudaMallocHostId 716
extern  dscudaMallocHostResult * dscudamallochostid_1();
extern  dscudaMallocHostResult * dscudamallochostid_1_svc();
#define dscudaFreeHostId 717
extern  dscudaResult * dscudafreehostid_1();
extern  dscudaResult * dscudafreehostid_1_svc();
#define dscudaHostGetDevicePointerId 718
extern  dscudaHostGetDevicePointerResult * dscudahostgetdevicepointerid_1();
extern  dscudaHostGetDevicePointerResult * dscudahostgetdevicepointerid_1_svc();
#define dscudaHostGetFlagsID 719
extern  dscudaHostGetFlagsResult * dscudahostgetflagsid_1();
extern  dscudaHostGetFlagsResult * dscudahostgetflagsid_1_svc();
#define dscudaMallocArrayId 720
extern  dscudaMallocArrayResult * dscudamallocarrayid_1();
extern  dscudaMallocArrayResult * dscudamallocarrayid_1_svc();
#define dscudaFreeArrayId 721
extern  dscudaResult * dscudafreearrayid_1();
extern  dscudaResult * dscudafreearrayid_1_svc();
#define dscudaMemcpyToArrayH2HId 722
extern  dscudaMemcpyToArrayH2HResult * dscudamemcpytoarrayh2hid_1();
extern  dscudaMemcpyToArrayH2HResult * dscudamemcpytoarrayh2hid_1_svc();
#define dscudaMemcpyToArrayH2DId 723
extern  dscudaResult * dscudamemcpytoarrayh2did_1();
extern  dscudaResult * dscudamemcpytoarrayh2did_1_svc();
#define dscudaMemcpyToArrayD2HId 724
extern  dscudaMemcpyToArrayD2HResult * dscudamemcpytoarrayd2hid_1();
extern  dscudaMemcpyToArrayD2HResult * dscudamemcpytoarrayd2hid_1_svc();
#define dscudaMemcpyToArrayD2DId 725
extern  dscudaResult * dscudamemcpytoarrayd2did_1();
extern  dscudaResult * dscudamemcpytoarrayd2did_1_svc();
#define dscudaMallocPitchId 726
extern  dscudaMallocPitchResult * dscudamallocpitchid_1();
extern  dscudaMallocPitchResult * dscudamallocpitchid_1_svc();
#define dscudaMemcpy2DToArrayH2HId 727
extern  dscudaMemcpy2DToArrayH2HResult * dscudamemcpy2dtoarrayh2hid_1();
extern  dscudaMemcpy2DToArrayH2HResult * dscudamemcpy2dtoarrayh2hid_1_svc();
#define dscudaMemcpy2DToArrayH2DId 728
extern  dscudaResult * dscudamemcpy2dtoarrayh2did_1();
extern  dscudaResult * dscudamemcpy2dtoarrayh2did_1_svc();
#define dscudaMemcpy2DToArrayD2HId 729
extern  dscudaMemcpy2DToArrayD2HResult * dscudamemcpy2dtoarrayd2hid_1();
extern  dscudaMemcpy2DToArrayD2HResult * dscudamemcpy2dtoarrayd2hid_1_svc();
#define dscudaMemcpy2DToArrayD2DId 730
extern  dscudaResult * dscudamemcpy2dtoarrayd2did_1();
extern  dscudaResult * dscudamemcpy2dtoarrayd2did_1_svc();
#define dscudaMemcpy2DH2HId 731
extern  dscudaMemcpy2DH2HResult * dscudamemcpy2dh2hid_1();
extern  dscudaMemcpy2DH2HResult * dscudamemcpy2dh2hid_1_svc();
#define dscudaMemcpy2DH2DId 732
extern  dscudaResult * dscudamemcpy2dh2did_1();
extern  dscudaResult * dscudamemcpy2dh2did_1_svc();
#define dscudaMemcpy2DD2HId 733
extern  dscudaMemcpy2DD2HResult * dscudamemcpy2dd2hid_1();
extern  dscudaMemcpy2DD2HResult * dscudamemcpy2dd2hid_1_svc();
#define dscudaMemcpy2DD2DId 734
extern  dscudaResult * dscudamemcpy2dd2did_1();
extern  dscudaResult * dscudamemcpy2dd2did_1_svc();
#define dscudaMemset2DId 735
extern  dscudaResult * dscudamemset2did_1();
extern  dscudaResult * dscudamemset2did_1_svc();
#define dscudaMemcpyToSymbolAsyncH2DId 736
extern  dscudaResult * dscudamemcpytosymbolasynch2did_1();
extern  dscudaResult * dscudamemcpytosymbolasynch2did_1_svc();
#define dscudaMemcpyToSymbolAsyncD2DId 737
extern  dscudaResult * dscudamemcpytosymbolasyncd2did_1();
extern  dscudaResult * dscudamemcpytosymbolasyncd2did_1_svc();
#define dscudaMemcpyFromSymbolAsyncD2HId 738
extern  dscudaMemcpyFromSymbolAsyncD2HResult * dscudamemcpyfromsymbolasyncd2hid_1();
extern  dscudaMemcpyFromSymbolAsyncD2HResult * dscudamemcpyfromsymbolasyncd2hid_1_svc();
#define dscudaMemcpyFromSymbolAsyncD2DId 739
extern  dscudaResult * dscudamemcpyfromsymbolasyncd2did_1();
extern  dscudaResult * dscudamemcpyfromsymbolasyncd2did_1_svc();
#define dscudaCreateChannelDescId 1400
extern  dscudaCreateChannelDescResult * dscudacreatechanneldescid_1();
extern  dscudaCreateChannelDescResult * dscudacreatechanneldescid_1_svc();
#define dscudaGetChannelDescId 1401
extern  dscudaGetChannelDescResult * dscudagetchanneldescid_1();
extern  dscudaGetChannelDescResult * dscudagetchanneldescid_1_svc();
#define dscudaBindTextureId 1402
extern  dscudaBindTextureResult * dscudabindtextureid_1();
extern  dscudaBindTextureResult * dscudabindtextureid_1_svc();
#define dscudaBindTexture2DId 1403
extern  dscudaBindTexture2DResult * dscudabindtexture2did_1();
extern  dscudaBindTexture2DResult * dscudabindtexture2did_1_svc();
#define dscudaBindTextureToArrayId 1404
extern  dscudaResult * dscudabindtexturetoarrayid_1();
extern  dscudaResult * dscudabindtexturetoarrayid_1_svc();
#define dscudaUnbindTextureId 1405
extern  dscudaResult * dscudaunbindtextureid_1();
extern  dscudaResult * dscudaunbindtextureid_1_svc();
#define dscufftPlan3dId 2002
extern  dscufftPlanResult * dscufftplan3did_1();
extern  dscufftPlanResult * dscufftplan3did_1_svc();
#define dscufftDestroyId 2004
extern  dscufftResult * dscufftdestroyid_1();
extern  dscufftResult * dscufftdestroyid_1_svc();
#define dscufftExecC2CId 2005
extern  dscufftResult * dscufftexecc2cid_1();
extern  dscufftResult * dscufftexecc2cid_1_svc();
extern int dscuda_prog_1_freeresult ();
#endif 



#if defined(__STDC__) || defined(__cplusplus)
extern  bool_t xdr_RCadr (XDR *, RCadr*);
extern  bool_t xdr_RCstream (XDR *, RCstream*);
extern  bool_t xdr_RCevent (XDR *, RCevent*);
extern  bool_t xdr_RCipaddr (XDR *, RCipaddr*);
extern  bool_t xdr_RCsize (XDR *, RCsize*);
extern  bool_t xdr_RCerror (XDR *, RCerror*);
extern  bool_t xdr_RCbuf (XDR *, RCbuf*);
extern  bool_t xdr_RCchannelformat (XDR *, RCchannelformat*);
extern  bool_t xdr_RCpid (XDR *, RCpid*);
extern  bool_t xdr_RCchanneldesc_t (XDR *, RCchanneldesc_t*);
extern  bool_t xdr_RCchanneldesc (XDR *, RCchanneldesc*);
extern  bool_t xdr_RCtexture_t (XDR *, RCtexture_t*);
extern  bool_t xdr_RCtexture (XDR *, RCtexture*);
extern  bool_t xdr_RCfuncattr_t (XDR *, RCfuncattr_t*);
extern  bool_t xdr_RCfuncattr (XDR *, RCfuncattr*);
extern  bool_t xdr_RCargType (XDR *, RCargType*);
extern  bool_t xdr_RCargVal (XDR *, RCargVal*);
extern  bool_t xdr_RCarg (XDR *, RCarg*);
extern  bool_t xdr_RCargs (XDR *, RCargs*);
extern  bool_t xdr_dscudaResult (XDR *, dscudaResult*);
extern  bool_t xdr_dscudaThreadGetLimitResult (XDR *, dscudaThreadGetLimitResult*);
extern  bool_t xdr_dscudaThreadGetCacheConfigResult (XDR *, dscudaThreadGetCacheConfigResult*);
extern  bool_t xdr_dscudaMallocResult (XDR *, dscudaMallocResult*);
extern  bool_t xdr_dscudaHostAllocResult (XDR *, dscudaHostAllocResult*);
extern  bool_t xdr_dscudaMallocHostResult (XDR *, dscudaMallocHostResult*);
extern  bool_t xdr_dscudaMallocArrayResult (XDR *, dscudaMallocArrayResult*);
extern  bool_t xdr_dscudaMallocPitchResult (XDR *, dscudaMallocPitchResult*);
extern  bool_t xdr_dscudaMemcpyD2HResult (XDR *, dscudaMemcpyD2HResult*);
extern  bool_t xdr_dscudaMemcpyH2HResult (XDR *, dscudaMemcpyH2HResult*);
extern  bool_t xdr_dscudaMemcpyToArrayD2HResult (XDR *, dscudaMemcpyToArrayD2HResult*);
extern  bool_t xdr_dscudaMemcpyToArrayH2HResult (XDR *, dscudaMemcpyToArrayH2HResult*);
extern  bool_t xdr_dscudaMemcpy2DToArrayD2HResult (XDR *, dscudaMemcpy2DToArrayD2HResult*);
extern  bool_t xdr_dscudaMemcpy2DToArrayH2HResult (XDR *, dscudaMemcpy2DToArrayH2HResult*);
extern  bool_t xdr_dscudaMemcpy2DD2HResult (XDR *, dscudaMemcpy2DD2HResult*);
extern  bool_t xdr_dscudaMemcpy2DH2HResult (XDR *, dscudaMemcpy2DH2HResult*);
extern  bool_t xdr_dscudaGetDeviceResult (XDR *, dscudaGetDeviceResult*);
extern  bool_t xdr_dscudaGetDeviceCountResult (XDR *, dscudaGetDeviceCountResult*);
extern  bool_t xdr_dscudaGetDevicePropertiesResult (XDR *, dscudaGetDevicePropertiesResult*);
extern  bool_t xdr_dscudaDriverGetVersionResult (XDR *, dscudaDriverGetVersionResult*);
extern  bool_t xdr_dscudaRuntimeGetVersionResult (XDR *, dscudaRuntimeGetVersionResult*);
extern  bool_t xdr_dscudaGetErrorStringResult (XDR *, dscudaGetErrorStringResult*);
extern  bool_t xdr_dscudaCreateChannelDescResult (XDR *, dscudaCreateChannelDescResult*);
extern  bool_t xdr_dscudaGetChannelDescResult (XDR *, dscudaGetChannelDescResult*);
extern  bool_t xdr_dscudaChooseDeviceResult (XDR *, dscudaChooseDeviceResult*);
extern  bool_t xdr_dscudaMemcpyAsyncD2HResult (XDR *, dscudaMemcpyAsyncD2HResult*);
extern  bool_t xdr_dscudaMemcpyAsyncH2HResult (XDR *, dscudaMemcpyAsyncH2HResult*);
extern  bool_t xdr_dscudaMemcpyFromSymbolD2HResult (XDR *, dscudaMemcpyFromSymbolD2HResult*);
extern  bool_t xdr_dscudaMemcpyFromSymbolAsyncD2HResult (XDR *, dscudaMemcpyFromSymbolAsyncD2HResult*);
extern  bool_t xdr_dscudaStreamCreateResult (XDR *, dscudaStreamCreateResult*);
extern  bool_t xdr_dscudaEventCreateResult (XDR *, dscudaEventCreateResult*);
extern  bool_t xdr_dscudaEventElapsedTimeResult (XDR *, dscudaEventElapsedTimeResult*);
extern  bool_t xdr_dscudaHostGetDevicePointerResult (XDR *, dscudaHostGetDevicePointerResult*);
extern  bool_t xdr_dscudaHostGetFlagsResult (XDR *, dscudaHostGetFlagsResult*);
extern  bool_t xdr_dscudaLoadModuleResult (XDR *, dscudaLoadModuleResult*);
extern  bool_t xdr_dscudaFuncGetAttributesResult (XDR *, dscudaFuncGetAttributesResult*);
extern  bool_t xdr_dscudaBindTextureResult (XDR *, dscudaBindTextureResult*);
extern  bool_t xdr_dscudaBindTexture2DResult (XDR *, dscudaBindTexture2DResult*);
extern  bool_t xdr_dscufftResult (XDR *, dscufftResult*);
extern  bool_t xdr_dscufftPlanResult (XDR *, dscufftPlanResult*);
extern  bool_t xdr_dscublasResult (XDR *, dscublasResult*);
extern  bool_t xdr_dscublasCreateResult (XDR *, dscublasCreateResult*);
extern  bool_t xdr_dscublasGetVectorResult (XDR *, dscublasGetVectorResult*);
extern  bool_t xdr_RCdim3 (XDR *, RCdim3*);
extern  bool_t xdr_dscudathreadsetlimitid_1_argument (XDR *, dscudathreadsetlimitid_1_argument*);
extern  bool_t xdr_dscudastreamwaiteventid_1_argument (XDR *, dscudastreamwaiteventid_1_argument*);
extern  bool_t xdr_dscudaeventelapsedtimeid_1_argument (XDR *, dscudaeventelapsedtimeid_1_argument*);
extern  bool_t xdr_dscudaeventrecordid_1_argument (XDR *, dscudaeventrecordid_1_argument*);
extern  bool_t xdr_dscudalaunchkernelid_1_argument (XDR *, dscudalaunchkernelid_1_argument*);
extern  bool_t xdr_dscudaloadmoduleid_1_argument (XDR *, dscudaloadmoduleid_1_argument*);
extern  bool_t xdr_dscudafuncgetattributesid_1_argument (XDR *, dscudafuncgetattributesid_1_argument*);
extern  bool_t xdr_dscudamemcpyh2hid_1_argument (XDR *, dscudamemcpyh2hid_1_argument*);
extern  bool_t xdr_dscudamemcpyh2did_1_argument (XDR *, dscudamemcpyh2did_1_argument*);
extern  bool_t xdr_dscudamemcpyd2hid_1_argument (XDR *, dscudamemcpyd2hid_1_argument*);
extern  bool_t xdr_dscudamemcpyd2did_1_argument (XDR *, dscudamemcpyd2did_1_argument*);
extern  bool_t xdr_dscudamemcpyasynch2hid_1_argument (XDR *, dscudamemcpyasynch2hid_1_argument*);
extern  bool_t xdr_dscudamemcpyasynch2did_1_argument (XDR *, dscudamemcpyasynch2did_1_argument*);
extern  bool_t xdr_dscudamemcpyasyncd2hid_1_argument (XDR *, dscudamemcpyasyncd2hid_1_argument*);
extern  bool_t xdr_dscudamemcpyasyncd2did_1_argument (XDR *, dscudamemcpyasyncd2did_1_argument*);
extern  bool_t xdr_dscudamemcpytosymbolh2did_1_argument (XDR *, dscudamemcpytosymbolh2did_1_argument*);
extern  bool_t xdr_dscudamemcpytosymbold2did_1_argument (XDR *, dscudamemcpytosymbold2did_1_argument*);
extern  bool_t xdr_dscudamemcpyfromsymbold2hid_1_argument (XDR *, dscudamemcpyfromsymbold2hid_1_argument*);
extern  bool_t xdr_dscudamemcpyfromsymbold2did_1_argument (XDR *, dscudamemcpyfromsymbold2did_1_argument*);
extern  bool_t xdr_dscudamemsetid_1_argument (XDR *, dscudamemsetid_1_argument*);
extern  bool_t xdr_dscudahostallocid_1_argument (XDR *, dscudahostallocid_1_argument*);
extern  bool_t xdr_dscudahostgetdevicepointerid_1_argument (XDR *, dscudahostgetdevicepointerid_1_argument*);
extern  bool_t xdr_dscudamallocarrayid_1_argument (XDR *, dscudamallocarrayid_1_argument*);
extern  bool_t xdr_dscudamemcpytoarrayh2hid_1_argument (XDR *, dscudamemcpytoarrayh2hid_1_argument*);
extern  bool_t xdr_dscudamemcpytoarrayh2did_1_argument (XDR *, dscudamemcpytoarrayh2did_1_argument*);
extern  bool_t xdr_dscudamemcpytoarrayd2hid_1_argument (XDR *, dscudamemcpytoarrayd2hid_1_argument*);
extern  bool_t xdr_dscudamemcpytoarrayd2did_1_argument (XDR *, dscudamemcpytoarrayd2did_1_argument*);
extern  bool_t xdr_dscudamallocpitchid_1_argument (XDR *, dscudamallocpitchid_1_argument*);
extern  bool_t xdr_dscudamemcpy2dtoarrayh2hid_1_argument (XDR *, dscudamemcpy2dtoarrayh2hid_1_argument*);
extern  bool_t xdr_dscudamemcpy2dtoarrayh2did_1_argument (XDR *, dscudamemcpy2dtoarrayh2did_1_argument*);
extern  bool_t xdr_dscudamemcpy2dtoarrayd2hid_1_argument (XDR *, dscudamemcpy2dtoarrayd2hid_1_argument*);
extern  bool_t xdr_dscudamemcpy2dtoarrayd2did_1_argument (XDR *, dscudamemcpy2dtoarrayd2did_1_argument*);
extern  bool_t xdr_dscudamemcpy2dh2hid_1_argument (XDR *, dscudamemcpy2dh2hid_1_argument*);
extern  bool_t xdr_dscudamemcpy2dh2did_1_argument (XDR *, dscudamemcpy2dh2did_1_argument*);
extern  bool_t xdr_dscudamemcpy2dd2hid_1_argument (XDR *, dscudamemcpy2dd2hid_1_argument*);
extern  bool_t xdr_dscudamemcpy2dd2did_1_argument (XDR *, dscudamemcpy2dd2did_1_argument*);
extern  bool_t xdr_dscudamemset2did_1_argument (XDR *, dscudamemset2did_1_argument*);
extern  bool_t xdr_dscudamemcpytosymbolasynch2did_1_argument (XDR *, dscudamemcpytosymbolasynch2did_1_argument*);
extern  bool_t xdr_dscudamemcpytosymbolasyncd2did_1_argument (XDR *, dscudamemcpytosymbolasyncd2did_1_argument*);
extern  bool_t xdr_dscudamemcpyfromsymbolasyncd2hid_1_argument (XDR *, dscudamemcpyfromsymbolasyncd2hid_1_argument*);
extern  bool_t xdr_dscudamemcpyfromsymbolasyncd2did_1_argument (XDR *, dscudamemcpyfromsymbolasyncd2did_1_argument*);
extern  bool_t xdr_dscudacreatechanneldescid_1_argument (XDR *, dscudacreatechanneldescid_1_argument*);
extern  bool_t xdr_dscudabindtextureid_1_argument (XDR *, dscudabindtextureid_1_argument*);
extern  bool_t xdr_dscudabindtexture2did_1_argument (XDR *, dscudabindtexture2did_1_argument*);
extern  bool_t xdr_dscudabindtexturetoarrayid_1_argument (XDR *, dscudabindtexturetoarrayid_1_argument*);
extern  bool_t xdr_dscufftplan3did_1_argument (XDR *, dscufftplan3did_1_argument*);
extern  bool_t xdr_dscufftexecc2cid_1_argument (XDR *, dscufftexecc2cid_1_argument*);

#else 
extern bool_t xdr_RCadr ();
extern bool_t xdr_RCstream ();
extern bool_t xdr_RCevent ();
extern bool_t xdr_RCipaddr ();
extern bool_t xdr_RCsize ();
extern bool_t xdr_RCerror ();
extern bool_t xdr_RCbuf ();
extern bool_t xdr_RCchannelformat ();
extern bool_t xdr_RCpid ();
extern bool_t xdr_RCchanneldesc_t ();
extern bool_t xdr_RCchanneldesc ();
extern bool_t xdr_RCtexture_t ();
extern bool_t xdr_RCtexture ();
extern bool_t xdr_RCfuncattr_t ();
extern bool_t xdr_RCfuncattr ();
extern bool_t xdr_RCargType ();
extern bool_t xdr_RCargVal ();
extern bool_t xdr_RCarg ();
extern bool_t xdr_RCargs ();
extern bool_t xdr_dscudaResult ();
extern bool_t xdr_dscudaThreadGetLimitResult ();
extern bool_t xdr_dscudaThreadGetCacheConfigResult ();
extern bool_t xdr_dscudaMallocResult ();
extern bool_t xdr_dscudaHostAllocResult ();
extern bool_t xdr_dscudaMallocHostResult ();
extern bool_t xdr_dscudaMallocArrayResult ();
extern bool_t xdr_dscudaMallocPitchResult ();
extern bool_t xdr_dscudaMemcpyD2HResult ();
extern bool_t xdr_dscudaMemcpyH2HResult ();
extern bool_t xdr_dscudaMemcpyToArrayD2HResult ();
extern bool_t xdr_dscudaMemcpyToArrayH2HResult ();
extern bool_t xdr_dscudaMemcpy2DToArrayD2HResult ();
extern bool_t xdr_dscudaMemcpy2DToArrayH2HResult ();
extern bool_t xdr_dscudaMemcpy2DD2HResult ();
extern bool_t xdr_dscudaMemcpy2DH2HResult ();
extern bool_t xdr_dscudaGetDeviceResult ();
extern bool_t xdr_dscudaGetDeviceCountResult ();
extern bool_t xdr_dscudaGetDevicePropertiesResult ();
extern bool_t xdr_dscudaDriverGetVersionResult ();
extern bool_t xdr_dscudaRuntimeGetVersionResult ();
extern bool_t xdr_dscudaGetErrorStringResult ();
extern bool_t xdr_dscudaCreateChannelDescResult ();
extern bool_t xdr_dscudaGetChannelDescResult ();
extern bool_t xdr_dscudaChooseDeviceResult ();
extern bool_t xdr_dscudaMemcpyAsyncD2HResult ();
extern bool_t xdr_dscudaMemcpyAsyncH2HResult ();
extern bool_t xdr_dscudaMemcpyFromSymbolD2HResult ();
extern bool_t xdr_dscudaMemcpyFromSymbolAsyncD2HResult ();
extern bool_t xdr_dscudaStreamCreateResult ();
extern bool_t xdr_dscudaEventCreateResult ();
extern bool_t xdr_dscudaEventElapsedTimeResult ();
extern bool_t xdr_dscudaHostGetDevicePointerResult ();
extern bool_t xdr_dscudaHostGetFlagsResult ();
extern bool_t xdr_dscudaLoadModuleResult ();
extern bool_t xdr_dscudaFuncGetAttributesResult ();
extern bool_t xdr_dscudaBindTextureResult ();
extern bool_t xdr_dscudaBindTexture2DResult ();
extern bool_t xdr_dscufftResult ();
extern bool_t xdr_dscufftPlanResult ();
extern bool_t xdr_dscublasResult ();
extern bool_t xdr_dscublasCreateResult ();
extern bool_t xdr_dscublasGetVectorResult ();
extern bool_t xdr_RCdim3 ();
extern bool_t xdr_dscudathreadsetlimitid_1_argument ();
extern bool_t xdr_dscudastreamwaiteventid_1_argument ();
extern bool_t xdr_dscudaeventelapsedtimeid_1_argument ();
extern bool_t xdr_dscudaeventrecordid_1_argument ();
extern bool_t xdr_dscudalaunchkernelid_1_argument ();
extern bool_t xdr_dscudaloadmoduleid_1_argument ();
extern bool_t xdr_dscudafuncgetattributesid_1_argument ();
extern bool_t xdr_dscudamemcpyh2hid_1_argument ();
extern bool_t xdr_dscudamemcpyh2did_1_argument ();
extern bool_t xdr_dscudamemcpyd2hid_1_argument ();
extern bool_t xdr_dscudamemcpyd2did_1_argument ();
extern bool_t xdr_dscudamemcpyasynch2hid_1_argument ();
extern bool_t xdr_dscudamemcpyasynch2did_1_argument ();
extern bool_t xdr_dscudamemcpyasyncd2hid_1_argument ();
extern bool_t xdr_dscudamemcpyasyncd2did_1_argument ();
extern bool_t xdr_dscudamemcpytosymbolh2did_1_argument ();
extern bool_t xdr_dscudamemcpytosymbold2did_1_argument ();
extern bool_t xdr_dscudamemcpyfromsymbold2hid_1_argument ();
extern bool_t xdr_dscudamemcpyfromsymbold2did_1_argument ();
extern bool_t xdr_dscudamemsetid_1_argument ();
extern bool_t xdr_dscudahostallocid_1_argument ();
extern bool_t xdr_dscudahostgetdevicepointerid_1_argument ();
extern bool_t xdr_dscudamallocarrayid_1_argument ();
extern bool_t xdr_dscudamemcpytoarrayh2hid_1_argument ();
extern bool_t xdr_dscudamemcpytoarrayh2did_1_argument ();
extern bool_t xdr_dscudamemcpytoarrayd2hid_1_argument ();
extern bool_t xdr_dscudamemcpytoarrayd2did_1_argument ();
extern bool_t xdr_dscudamallocpitchid_1_argument ();
extern bool_t xdr_dscudamemcpy2dtoarrayh2hid_1_argument ();
extern bool_t xdr_dscudamemcpy2dtoarrayh2did_1_argument ();
extern bool_t xdr_dscudamemcpy2dtoarrayd2hid_1_argument ();
extern bool_t xdr_dscudamemcpy2dtoarrayd2did_1_argument ();
extern bool_t xdr_dscudamemcpy2dh2hid_1_argument ();
extern bool_t xdr_dscudamemcpy2dh2did_1_argument ();
extern bool_t xdr_dscudamemcpy2dd2hid_1_argument ();
extern bool_t xdr_dscudamemcpy2dd2did_1_argument ();
extern bool_t xdr_dscudamemset2did_1_argument ();
extern bool_t xdr_dscudamemcpytosymbolasynch2did_1_argument ();
extern bool_t xdr_dscudamemcpytosymbolasyncd2did_1_argument ();
extern bool_t xdr_dscudamemcpyfromsymbolasyncd2hid_1_argument ();
extern bool_t xdr_dscudamemcpyfromsymbolasyncd2did_1_argument ();
extern bool_t xdr_dscudacreatechanneldescid_1_argument ();
extern bool_t xdr_dscudabindtextureid_1_argument ();
extern bool_t xdr_dscudabindtexture2did_1_argument ();
extern bool_t xdr_dscudabindtexturetoarrayid_1_argument ();
extern bool_t xdr_dscufftplan3did_1_argument ();
extern bool_t xdr_dscufftexecc2cid_1_argument ();

#endif 

#ifdef __cplusplus
}
#endif

#endif 
#pragma end dscudarpc.h
#pragma begin dscudamacros.h
#ifndef DSCUDA_MACROS_H
#define DSCUDA_MACROS_H

#define WARN(lv, fmt, args...) if (lv <= dscudaWarnLevel()) fprintf(stderr, fmt, ## args);
#define WARNONCE(lv, fmt, args...) if (lv <= dscudaWarnLevel()) { \
        static int firstcall = 1;                                 \
        if (firstcall) {                                          \
            firstcall = 0;                                        \
            fprintf(stderr, fmt, ## args);                        \
        }                                                         \
    }

#define ALIGN_UP(off, align) (off) = ((off) + (align) - 1) & ~((align) - 1)
int dscudaWarnLevel(void);
void dscudaSetWarnLevel(int level);

#endif 
#pragma end dscudamacros.h

#define TEST_NZ(x) do { if ( (x)) {WARN(0, #x " failed (returned non-zero).\n" ); exit(EXIT_FAILURE); } } while (0)
#define TEST_Z(x)  do { if (!(x)) {WARN(0, #x " failed (returned zero/null).\n"); exit(EXIT_FAILURE); } } while (0)


#define RC_NWR_PER_POST (16) 
#define RC_SGE_SIZE (1024 * 1024 * 2) 
#define RC_WR_MAX (RC_NWR_PER_POST * 16) 
#define RC_RDMA_BUF_SIZE (RC_NWR_PER_POST * RC_SGE_SIZE) 


#if RC_RDMA_BUF_SIZE  < RC_KMODULEIMAGELEN
#error "RC_RDMA_BUF_SIZE too small."

#endif

#define RC_SERVER_IBV_CQ_SIZE (RC_WR_MAX)
#define RC_CLIENT_IBV_CQ_SIZE (65536)

#define RC_IBV_IP_PORT_BASE  (65432)
#define RC_IBV_TIMEOUT (500)  

struct message {
    struct ibv_mr mr[RC_NWR_PER_POST];
};

enum rdma_state_t {
    STATE_INIT,
    STATE_READY,
    STATE_BUSY,
};

typedef struct {
    
    struct rdma_cm_id *id;
    struct ibv_qp *qp;
    struct ibv_context *ibvctx;
    struct ibv_pd *pd;
    struct ibv_cq *cq;
    struct ibv_comp_channel *comp_channel;

    
    struct message *recv_msg;
    struct message *send_msg;

    
    char *rdma_local_region;
    char *rdma_remote_region;

    
    struct ibv_mr *recv_mr;
    struct ibv_mr *send_mr;
    struct ibv_mr peer_mr[RC_NWR_PER_POST];

    
    struct ibv_mr *rdma_local_mr[RC_NWR_PER_POST];
    struct ibv_mr *rdma_remote_mr[RC_NWR_PER_POST];

    
    pthread_t cq_poller_thread;
    int connected;
    enum rdma_state_t rdma_state;
    int rdma_nreq_pending;
} IbvConnection;

typedef enum {
    RCMethodNone = 0,
    RCMethodMemcpyH2D,
    RCMethodMemcpyD2H,
    RCMethodMemcpyD2D,
    RCMethodMalloc,
    RCMethodFree,
    RCMethodGetErrorString,
    RCMethodGetDeviceProperties,
    RCMethodRuntimeGetVersion,
    RCMethodThreadSynchronize,
    RCMethodThreadExit,
    RCMethodDeviceSynchronize,
    RCMethodDscudaMemcpyToSymbolH2D,
    RCMethodDscudaMemcpyToSymbolD2D,
    RCMethodDscudaMemcpyFromSymbolD2H,
    RCMethodDscudaMemcpyFromSymbolD2D,
    RCMethodDscudaMemcpyToSymbolAsyncH2D,
    RCMethodDscudaMemcpyToSymbolAsyncD2D,
    RCMethodDscudaMemcpyFromSymbolAsyncD2H,
    RCMethodDscudaMemcpyFromSymbolAsyncD2D,
    RCMethodDscudaLoadModule,
    RCMethodDscudaLaunchKernel,

    

    RCMethodEnd
} RCMethod;


typedef struct {
    RCMethod method;
    int payload;
} IbvHdr;


typedef struct {
    RCMethod method;
    size_t count;
    RCadr dstadr;
    void *srcbuf;
} IbvMemcpyH2DInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
} IbvMemcpyH2DReturnHdr;


typedef struct {
    RCMethod method;
    size_t count;
    RCadr srcadr;
} IbvMemcpyD2HInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
    void *dstbuf;
} IbvMemcpyD2HReturnHdr;


typedef struct {
    RCMethod method;
    size_t count;
    RCadr dstadr;
    RCadr srcadr;
} IbvMemcpyD2DInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
} IbvMemcpyD2DReturnHdr;


typedef struct {
    RCMethod method;
    size_t size;
} IbvMallocInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
    RCadr devAdr;
} IbvMallocReturnHdr;


typedef struct {
    RCMethod method;
    RCadr devAdr;
} IbvFreeInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
} IbvFreeReturnHdr;


typedef struct {
    RCMethod method;
    int device;
    hipError_t err;
} IbvGetErrorStringInvokeHdr;

typedef struct {
    RCMethod method;
    char *errmsg;
} IbvGetErrorStringReturnHdr;


typedef struct {
    RCMethod method;
    int device;
} IbvGetDevicePropertiesInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
    hipDeviceProp_t prop;
} IbvGetDevicePropertiesReturnHdr;


typedef struct {
    RCMethod method;
    char dummy[8];
} IbvRuntimeGetVersionInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
    int version;
} IbvRuntimeGetVersionReturnHdr;


typedef struct {
    RCMethod method;
    char dummy[8];
} IbvThreadSynchronizeInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
} IbvThreadSynchronizeReturnHdr;


typedef struct {
    RCMethod method;
    char dummy[8];
} IbvThreadExitInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
} IbvThreadExitReturnHdr;


typedef struct {
    RCMethod method;
    char dummy[8];
} IbvDeviceSynchronizeInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
} IbvDeviceSynchronizeReturnHdr;


typedef struct {
    RCMethod method;
    int moduleid;
    char symbol[RC_SNAMELEN];
    size_t count;
    size_t offset;
    void *src;
} IbvDscudaMemcpyToSymbolH2DInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
} IbvDscudaMemcpyToSymbolH2DReturnHdr;


typedef struct {
    RCMethod method;
    int moduleid;
    char symbol[RC_SNAMELEN];
    size_t count;
    size_t offset;
    RCadr srcadr;
} IbvDscudaMemcpyToSymbolD2DInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
} IbvDscudaMemcpyToSymbolD2DReturnHdr;



typedef struct {
    RCMethod method;
    int moduleid;
    char symbol[RC_SNAMELEN];
    size_t count;
    size_t offset;
} IbvDscudaMemcpyFromSymbolD2HInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
    void *dst;
} IbvDscudaMemcpyFromSymbolD2HReturnHdr;


typedef struct {
    RCMethod method;
    int moduleid;
    char symbol[RC_SNAMELEN];
    size_t count;
    size_t offset;
    RCadr dstadr;
} IbvDscudaMemcpyFromSymbolD2DInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
} IbvDscudaMemcpyFromSymbolD2DReturnHdr;


typedef struct {
    RCMethod method;
    int moduleid;
    char symbol[RC_SNAMELEN];
    size_t count;
    size_t offset;
    RCstream stream;
    void *src;
} IbvDscudaMemcpyToSymbolAsyncH2DInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
} IbvDscudaMemcpyToSymbolAsyncH2DReturnHdr;


typedef struct {
    RCMethod method;
    int moduleid;
    char symbol[RC_SNAMELEN];
    size_t count;
    size_t offset;
    RCstream stream;
    RCadr srcadr;
} IbvDscudaMemcpyToSymbolAsyncD2DInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
} IbvDscudaMemcpyToSymbolAsyncD2DReturnHdr;



typedef struct {
    RCMethod method;
    int moduleid;
    char symbol[RC_SNAMELEN];
    size_t count;
    size_t offset;
    RCstream stream;
} IbvDscudaMemcpyFromSymbolAsyncD2HInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
    void *dst;
} IbvDscudaMemcpyFromSymbolAsyncD2HReturnHdr;


typedef struct {
    RCMethod method;
    int moduleid;
    char symbol[RC_SNAMELEN];
    size_t count;
    size_t offset;
    RCstream stream;
    RCadr dstadr;
} IbvDscudaMemcpyFromSymbolAsyncD2DInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
} IbvDscudaMemcpyFromSymbolAsyncD2DReturnHdr;



typedef struct {
    RCMethod method;
    uint64_t ipaddr;
    unsigned long int pid;
    char modulename[RC_KMODULENAMELEN];
    void *moduleimage;
} IbvDscudaLoadModuleInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
    int moduleid;
} IbvDscudaLoadModuleReturnHdr;


typedef struct {
    RCMethod method;
    int moduleid;
    int kernelid;
    char kernelname[RC_KNAMELEN];
    unsigned int gdim[3];
    unsigned int bdim[3];
    unsigned int smemsize;
    RCstream stream;
    int narg;
    void *args;
} IbvDscudaLaunchKernelInvokeHdr;

typedef struct {
    RCMethod method;
    hipError_t err;
} IbvDscudaLaunchKernelReturnHdr;

typedef struct {
    int type;
    union {
        uint64_t pointerval;
        unsigned int intval;
        float floatval;
        char customval[RC_KARGMAX];
    } val;
    unsigned int offset;
    unsigned int size;
} IbvArg;

void rdmaBuildConnection(struct rdma_cm_id *id, bool is_server);
void rdmaBuildParams(struct rdma_conn_param *params);
void rdmaDestroyConnection(IbvConnection *conn);
void rdmaSetOnCompletionHandler(void (*handler)(struct ibv_wc *));
void rdmaOnCompletionClient(struct ibv_wc *);
void rdmaOnCompletionServer(struct ibv_wc *);
void rdmaWaitEvent(struct rdma_event_channel *ec, rdma_cm_event_type et, int (*handler)(struct rdma_cm_id *id));
void rdmaWaitReadyToKickoff(IbvConnection *conn);
void rdmaWaitReadyToDisconnect(IbvConnection *conn);
void rdmaKickoff(IbvConnection *conn, int length);
void rdmaPipelinedKickoff(IbvConnection *conn, int length, char *payload_buf, char *payload_src, int payload_size);
void rdmaSendMr(IbvConnection *conn);

#endif 

#endif 
#pragma end ibv_rdma.h

enum {
    RC_REMOTECALL_TYPE_RPC,
    RC_REMOTECALL_TYPE_IBV,
};


int dscudaWarnLevel(void);
void dscudaSetWarnLevel(int level);
char *dscudaMemcpyKindName(hipMemcpyKind kind);
const char *dscudaGetIpaddrString(unsigned int addr);
double RCgetCputime(double *t0);


void *dscudaUvaOfAdr(void *adr, int devid);
int dscudaDevidOfUva(void *adr);
void *dscudaAdrOfUva(void *adr);
int dscudaNredundancy(void);
void dscudaSetAutoVerb(int verb);
int dscudaRemoteCallType(void);
void dscudaSetErrorHandler(void (*handler)(void *), void *handler_arg);
void dscudaGetMangledFunctionName(char *name, const char *funcif, const char *ptxdata);
int *dscudaLoadModule(char *srcname, char *strdata);
void rpcDscudaLaunchKernelWrapper(int *moduleid, int kid, char *kname,
                              RCdim3 gdim, RCdim3 bdim, RCsize smemsize, RCstream stream,
                              RCargs args);
void ibvDscudaLaunchKernelWrapper(int *moduleid, int kid, char *kname,
                                 int *gdim, int *bdim, RCsize smemsize, RCstream stream,
                                 int narg, IbvArg *arg);

hipError_t dscudaFuncGetAttributesWrapper(int *moduleid, struct hipFuncAttributes *attr, const char *func);

hipError_t dscudaMemcpyToSymbolWrapper(int *moduleid, const char *symbol, const void *src,
                                       size_t count, size_t offset = 0,
                                       enum hipMemcpyKind kind = hipMemcpyHostToDevice);

hipError_t dscudaMemcpyToSymbolAsyncWrapper(int *moduleid, const char *symbol, const void *src,
					    size_t count, size_t offset = 0,
					    enum hipMemcpyKind kind = hipMemcpyHostToDevice, hipStream_t stream = 0);

hipError_t dscudaMemcpyFromSymbolWrapper(int *moduleid, void *dst, const char *symbol,
					 size_t count, size_t offset = 0,
					 enum hipMemcpyKind kind = hipMemcpyDeviceToHost);

hipError_t dscudaMemcpyFromSymbolAsyncWrapper(int *moduleid, void *dst, const char *symbol,
					      size_t count, size_t offset = 0,
					      enum hipMemcpyKind kind = hipMemcpyDeviceToHost, hipStream_t stream = 0);

hipError_t dscudaBindTextureWrapper(int *moduleid, char *texname,
                                    size_t *offset,
                                    const struct textureReference *tex,
                                    const void *devPtr,
                                    const struct hipChannelFormatDesc *desc,
                                    size_t size = UINT_MAX);

template<class T, int dim, enum hipTextureReadMode readMode>
hipError_t dscudaBindTextureWrapper(int *moduleid, char *texname,
                                    size_t *offset,
                                    const struct texture<T, dim, readMode> &tex,
                                    const void *devPtr,
                                    const struct hipChannelFormatDesc &desc,
                                    size_t size = UINT_MAX)
{
  return     dscudaBindTextureWrapper(dscudaLoadModule("./dscudatmp/matrixMul.cu.ptx", Ptxdata), "tex", offset, &tex, devPtr, &desc, size);
}

template<class T, int dim, enum hipTextureReadMode readMode>
hipError_t dscudaBindTextureWrapper(int *moduleid, char *texname,
                                    size_t *offset,
                                    const struct texture<T, dim, readMode> &tex,
                                    const void *devPtr,
                                    size_t size = UINT_MAX)
{
  return     dscudaBindTextureWrapper(dscudaLoadModule("./dscudatmp/matrixMul.cu.ptx", Ptxdata), "tex", offset, tex, devPtr, tex.channelDesc, size);
}


hipError_t dscudaBindTexture2DWrapper(int *moduleid, char *texname,
                                      size_t *offset,
                                      const struct textureReference *tex,
                                      const void *devPtr,
                                      const struct hipChannelFormatDesc *desc,
                                      size_t width, size_t height, size_t pitch);

template<class T, int dim, enum hipTextureReadMode readMode>
hipError_t dscudaBindTexture2DWrapper(int *moduleid, char *texname,
                                      size_t *offset,
                                      const struct texture<T, dim, readMode> &tex,
                                      const void *devPtr,
                                      const struct hipChannelFormatDesc &desc,
                                      size_t width, size_t height, size_t pitch)
{
    return dscudaBindTexture2DWrapper(moduleid, texname,
                                     offset, &tex, devPtr, &desc, width, height, pitch);
}

template<class T, int dim, enum hipTextureReadMode readMode>
hipError_t dscudaBindTexture2DWrapper(int *moduleid, char *texname,
                                      size_t *offset,
                                      const struct texture<T, dim, readMode> &tex,
                                      const void *devPtr,
                                      size_t width, size_t height, size_t pitch)
{
    return dscudaBindTexture2DWrapper(moduleid, texname,
                                     offset, &tex, devPtr, &tex.channelDesc, width, height, pitch);
}

hipError_t dscudaBindTextureToArrayWrapper(int *moduleid, char *texname,
                                           const struct textureReference *tex,
                                           const struct hipArray * array,
                                           const struct hipChannelFormatDesc *desc);

template<class T, int dim, enum hipTextureReadMode readMode>
hipError_t dscudaBindTextureToArrayWrapper(int *moduleid, char *texname,
                                           const struct texture<T, dim, readMode> &tex,
                                           const struct hipArray * array,
                                           const struct hipChannelFormatDesc & desc)
{
    return dscudaBindTextureToArrayWrapper(moduleid, texname, &tex, array, &desc);
}

template<class T, int dim, enum hipTextureReadMode readMode>
hipError_t dscudaBindTextureToArrayWrapper(int *moduleid, char *texname,
                                           const struct texture<T, dim, readMode> &tex,
                                           const struct hipArray * array)
{
    struct hipChannelFormatDesc desc;
    hipError_t err = hipGetChannelDesc(&desc, array);
    return err == hipSuccess ? dscudaBindTextureToArrayWrapper(moduleid, texname, &tex, array, &desc) : err;
}

#endif 
#pragma end dscuda.h




#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>


#include <hip/hip_runtime.h>

#include <cutil.h>

#ifdef MIN
#undef MIN
#endif
#ifdef MAX
#undef MAX
#endif
#include <cutil_inline.h>







#pragma begin matrixMulkernel.cu
#include <stdio.h>
#define CHECK_BANK_CONFLICTS 0
#if CHECK_BANK_CONFLICTS
#define AS(i, j) cutilBankChecker(((float*)&As[0][0]), (BLOCK_SIZE * i + j))
#define BS(i, j) cutilBankChecker(((float*)&Bs[0][0]), (BLOCK_SIZE * i + j))
#else
#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]
#endif



/*
 * stub for remote call to matrixMul.
 */
template <int BLOCK_SIZE> void
dscudamatrixMul(dim3 _gdim, dim3 _bdim, size_t _smemsize, hipStream_t _stream , float* C, float* A, float* B, int wA, int wB)
{
    int _narg = 5;
    int _ibvgdim[3], _ibvbdim[3];
    IbvArg _ibvarg[5], *_ibvargp;
    RCargs _rcargs;
    RCarg _rcarg[5], *_rcargp;
    RCdim3 _gdimrc, _bdimrc;
    int _off = 0;
    int _rcargc = 0;
    void *_devptr;
    _rcargs.RCargs_val = _rcarg;
    _rcargs.RCargs_len = _narg;
    static char mangledname_[512] = {0,};
    if (!mangledname_[0]) {
        if (1) {
          dscudaGetMangledFunctionName(mangledname_, __PRETTY_FUNCTION__, Ptxdata);
        }
        else {
          char buf_[256];
          sprintf(buf_, "%s", __FUNCTION__);
          strcpy(mangledname_, buf_ + strlen("dscuda")); // obtain original function name.
        }
        WARN(3, "mangled name : %s\n", mangledname_);
    }

    if (dscudaRemoteCallType() == RC_REMOTECALL_TYPE_IBV) {

        // a pointer to a device-address 'dscudaAdrOfUva(C)'.
        _ibvargp = _ibvarg + _rcargc;
        _rcargc++;
        _devptr = (void*)(size_t)dscudaAdrOfUva(C);
        ALIGN_UP(_off, __alignof(_devptr));
        _ibvargp->type = dscudaArgTypeP;
        _ibvargp->offset = _off;
        _ibvargp->val.pointerval = (RCadr)_devptr;
        _ibvargp->size = sizeof(_devptr);
        _off += _ibvargp->size;


        // a pointer to a device-address 'dscudaAdrOfUva(A)'.
        _ibvargp = _ibvarg + _rcargc;
        _rcargc++;
        _devptr = (void*)(size_t)dscudaAdrOfUva(A);
        ALIGN_UP(_off, __alignof(_devptr));
        _ibvargp->type = dscudaArgTypeP;
        _ibvargp->offset = _off;
        _ibvargp->val.pointerval = (RCadr)_devptr;
        _ibvargp->size = sizeof(_devptr);
        _off += _ibvargp->size;


        // a pointer to a device-address 'dscudaAdrOfUva(B)'.
        _ibvargp = _ibvarg + _rcargc;
        _rcargc++;
        _devptr = (void*)(size_t)dscudaAdrOfUva(B);
        ALIGN_UP(_off, __alignof(_devptr));
        _ibvargp->type = dscudaArgTypeP;
        _ibvargp->offset = _off;
        _ibvargp->val.pointerval = (RCadr)_devptr;
        _ibvargp->size = sizeof(_devptr);
        _off += _ibvargp->size;


        // an integer 'wA'.
        _ibvargp = _ibvarg + _rcargc;
        _rcargc++;
        ALIGN_UP(_off, __alignof(int));
        _ibvargp->type = dscudaArgTypeI;
        _ibvargp->offset = _off;
        _ibvargp->val.intval = wA;
        _ibvargp->size = sizeof(int);
        _off += _ibvargp->size;


        // an integer 'wB'.
        _ibvargp = _ibvarg + _rcargc;
        _rcargc++;
        ALIGN_UP(_off, __alignof(int));
        _ibvargp->type = dscudaArgTypeI;
        _ibvargp->offset = _off;
        _ibvargp->val.intval = wB;
        _ibvargp->size = sizeof(int);
        _off += _ibvargp->size;

        _ibvgdim[0] = _gdim.x; _ibvgdim[1] = _gdim.y; _ibvgdim[2] = _gdim.z;
        _ibvbdim[0] = _bdim.x; _ibvbdim[1] = _bdim.y; _ibvbdim[2] = _gdim.z;
#if !RPC_ONLY
        ibvDscudaLaunchKernelWrapper(dscudaLoadModule("./dscudatmp/matrixMul.cu.ptx", Ptxdata), 0, mangledname_,
                                 _ibvgdim, _ibvbdim, _smemsize, (RCstream)_stream,
                                 _narg, _ibvarg);
#endif
    }
    else {

        // a pointer to a device-address 'dscudaAdrOfUva(C)'.
        _rcargp = &(_rcargs.RCargs_val[_rcargc++]);
        _devptr = (void*)(size_t)dscudaAdrOfUva(C);
        ALIGN_UP(_off, __alignof(_devptr));
        _rcargp->val.type = dscudaArgTypeP;
        _rcargp->offset = _off;
        _rcargp->val.RCargVal_u.address = (RCadr)_devptr;
        _rcargp->size = sizeof(_devptr);
        _off += _rcargp->size;

        // a pointer to a device-address 'dscudaAdrOfUva(A)'.
        _rcargp = &(_rcargs.RCargs_val[_rcargc++]);
        _devptr = (void*)(size_t)dscudaAdrOfUva(A);
        ALIGN_UP(_off, __alignof(_devptr));
        _rcargp->val.type = dscudaArgTypeP;
        _rcargp->offset = _off;
        _rcargp->val.RCargVal_u.address = (RCadr)_devptr;
        _rcargp->size = sizeof(_devptr);
        _off += _rcargp->size;

        // a pointer to a device-address 'dscudaAdrOfUva(B)'.
        _rcargp = &(_rcargs.RCargs_val[_rcargc++]);
        _devptr = (void*)(size_t)dscudaAdrOfUva(B);
        ALIGN_UP(_off, __alignof(_devptr));
        _rcargp->val.type = dscudaArgTypeP;
        _rcargp->offset = _off;
        _rcargp->val.RCargVal_u.address = (RCadr)_devptr;
        _rcargp->size = sizeof(_devptr);
        _off += _rcargp->size;

        // an integer 'wA'.
        _rcargp = &(_rcargs.RCargs_val[_rcargc++]);
        ALIGN_UP(_off, __alignof(int));
        _rcargp->val.type = dscudaArgTypeI;
        _rcargp->offset = _off;
        _rcargp->val.RCargVal_u.valuei = wA;
        _rcargp->size = sizeof(int);
        _off += _rcargp->size;

        // an integer 'wB'.
        _rcargp = &(_rcargs.RCargs_val[_rcargc++]);
        ALIGN_UP(_off, __alignof(int));
        _rcargp->val.type = dscudaArgTypeI;
        _rcargp->offset = _off;
        _rcargp->val.RCargVal_u.valuei = wB;
        _rcargp->size = sizeof(int);
        _off += _rcargp->size;
        _gdimrc.x = _gdim.x; _gdimrc.y = _gdim.y; _gdimrc.z = _gdim.z;
        _bdimrc.x = _bdim.x; _bdimrc.y = _bdim.y; _bdimrc.z = _bdim.z;
        rpcDscudaLaunchKernelWrapper(dscudaLoadModule("./dscudatmp/matrixMul.cu.ptx", Ptxdata), 0, mangledname_,
                                 _gdimrc, _bdimrc, _smemsize, (RCstream)_stream,
                                 _rcargs);
    }
}
 template <int BLOCK_SIZE>  void
matrixMul(float* C, float* A, float* B, int wA, int wB)
{
    /* nop */
}








/*
 * stub for remote call to matrixMulDS.
 */
void
dscudamatrixMulDS(dim3 _gdim, dim3 _bdim, size_t _smemsize, hipStream_t _stream , float *C, float *A, float *B, int wA, int wB)
{
    int _narg = 5;
    int _ibvgdim[3], _ibvbdim[3];
    IbvArg _ibvarg[5], *_ibvargp;
    RCargs _rcargs;
    RCarg _rcarg[5], *_rcargp;
    RCdim3 _gdimrc, _bdimrc;
    int _off = 0;
    int _rcargc = 0;
    void *_devptr;
    _rcargs.RCargs_val = _rcarg;
    _rcargs.RCargs_len = _narg;
    static char mangledname_[512] = {0,};
    if (!mangledname_[0]) {
        if (1) {
          dscudaGetMangledFunctionName(mangledname_, __PRETTY_FUNCTION__, Ptxdata);
        }
        else {
          char buf_[256];
          sprintf(buf_, "%s", __FUNCTION__);
          strcpy(mangledname_, buf_ + strlen("dscuda")); // obtain original function name.
        }
        WARN(3, "mangled name : %s\n", mangledname_);
    }

    if (dscudaRemoteCallType() == RC_REMOTECALL_TYPE_IBV) {

        // a pointer to a device-address 'dscudaAdrOfUva(C)'.
        _ibvargp = _ibvarg + _rcargc;
        _rcargc++;
        _devptr = (void*)(size_t)dscudaAdrOfUva(C);
        ALIGN_UP(_off, __alignof(_devptr));
        _ibvargp->type = dscudaArgTypeP;
        _ibvargp->offset = _off;
        _ibvargp->val.pointerval = (RCadr)_devptr;
        _ibvargp->size = sizeof(_devptr);
        _off += _ibvargp->size;


        // a pointer to a device-address 'dscudaAdrOfUva(A)'.
        _ibvargp = _ibvarg + _rcargc;
        _rcargc++;
        _devptr = (void*)(size_t)dscudaAdrOfUva(A);
        ALIGN_UP(_off, __alignof(_devptr));
        _ibvargp->type = dscudaArgTypeP;
        _ibvargp->offset = _off;
        _ibvargp->val.pointerval = (RCadr)_devptr;
        _ibvargp->size = sizeof(_devptr);
        _off += _ibvargp->size;


        // a pointer to a device-address 'dscudaAdrOfUva(B)'.
        _ibvargp = _ibvarg + _rcargc;
        _rcargc++;
        _devptr = (void*)(size_t)dscudaAdrOfUva(B);
        ALIGN_UP(_off, __alignof(_devptr));
        _ibvargp->type = dscudaArgTypeP;
        _ibvargp->offset = _off;
        _ibvargp->val.pointerval = (RCadr)_devptr;
        _ibvargp->size = sizeof(_devptr);
        _off += _ibvargp->size;


        // an integer 'wA'.
        _ibvargp = _ibvarg + _rcargc;
        _rcargc++;
        ALIGN_UP(_off, __alignof(int));
        _ibvargp->type = dscudaArgTypeI;
        _ibvargp->offset = _off;
        _ibvargp->val.intval = wA;
        _ibvargp->size = sizeof(int);
        _off += _ibvargp->size;


        // an integer 'wB'.
        _ibvargp = _ibvarg + _rcargc;
        _rcargc++;
        ALIGN_UP(_off, __alignof(int));
        _ibvargp->type = dscudaArgTypeI;
        _ibvargp->offset = _off;
        _ibvargp->val.intval = wB;
        _ibvargp->size = sizeof(int);
        _off += _ibvargp->size;

        _ibvgdim[0] = _gdim.x; _ibvgdim[1] = _gdim.y; _ibvgdim[2] = _gdim.z;
        _ibvbdim[0] = _bdim.x; _ibvbdim[1] = _bdim.y; _ibvbdim[2] = _gdim.z;
#if !RPC_ONLY
        ibvDscudaLaunchKernelWrapper(dscudaLoadModule("./dscudatmp/matrixMul.cu.ptx", Ptxdata), 1, mangledname_,
                                 _ibvgdim, _ibvbdim, _smemsize, (RCstream)_stream,
                                 _narg, _ibvarg);
#endif
    }
    else {

        // a pointer to a device-address 'dscudaAdrOfUva(C)'.
        _rcargp = &(_rcargs.RCargs_val[_rcargc++]);
        _devptr = (void*)(size_t)dscudaAdrOfUva(C);
        ALIGN_UP(_off, __alignof(_devptr));
        _rcargp->val.type = dscudaArgTypeP;
        _rcargp->offset = _off;
        _rcargp->val.RCargVal_u.address = (RCadr)_devptr;
        _rcargp->size = sizeof(_devptr);
        _off += _rcargp->size;

        // a pointer to a device-address 'dscudaAdrOfUva(A)'.
        _rcargp = &(_rcargs.RCargs_val[_rcargc++]);
        _devptr = (void*)(size_t)dscudaAdrOfUva(A);
        ALIGN_UP(_off, __alignof(_devptr));
        _rcargp->val.type = dscudaArgTypeP;
        _rcargp->offset = _off;
        _rcargp->val.RCargVal_u.address = (RCadr)_devptr;
        _rcargp->size = sizeof(_devptr);
        _off += _rcargp->size;

        // a pointer to a device-address 'dscudaAdrOfUva(B)'.
        _rcargp = &(_rcargs.RCargs_val[_rcargc++]);
        _devptr = (void*)(size_t)dscudaAdrOfUva(B);
        ALIGN_UP(_off, __alignof(_devptr));
        _rcargp->val.type = dscudaArgTypeP;
        _rcargp->offset = _off;
        _rcargp->val.RCargVal_u.address = (RCadr)_devptr;
        _rcargp->size = sizeof(_devptr);
        _off += _rcargp->size;

        // an integer 'wA'.
        _rcargp = &(_rcargs.RCargs_val[_rcargc++]);
        ALIGN_UP(_off, __alignof(int));
        _rcargp->val.type = dscudaArgTypeI;
        _rcargp->offset = _off;
        _rcargp->val.RCargVal_u.valuei = wA;
        _rcargp->size = sizeof(int);
        _off += _rcargp->size;

        // an integer 'wB'.
        _rcargp = &(_rcargs.RCargs_val[_rcargc++]);
        ALIGN_UP(_off, __alignof(int));
        _rcargp->val.type = dscudaArgTypeI;
        _rcargp->offset = _off;
        _rcargp->val.RCargVal_u.valuei = wB;
        _rcargp->size = sizeof(int);
        _off += _rcargp->size;
        _gdimrc.x = _gdim.x; _gdimrc.y = _gdim.y; _gdimrc.z = _gdim.z;
        _bdimrc.x = _bdim.x; _bdimrc.y = _bdim.y; _bdimrc.z = _bdim.z;
        rpcDscudaLaunchKernelWrapper(dscudaLoadModule("./dscudatmp/matrixMul.cu.ptx", Ptxdata), 1, mangledname_,
                                 _gdimrc, _bdimrc, _smemsize, (RCstream)_stream,
                                 _rcargs);
    }
}
  void matrixMulDS(float *C, float *A, float *B, int wA, int wB)
{
    /* nop */
}





#pragma end matrixMulkernel.cu

#define WA (4 * block_size) 
#define HA (6 * block_size) 
#define WB (4 * block_size) 
#define HB WA  
#define WC WB  
#define HC HA  
#define MEGA 1024*1024





static void
get_cputime(double *splittime, double *laptime)
{
    struct timeval x;

    gettimeofday(&x, NULL);

    *splittime = x.tv_sec + x.tv_usec/1000000.0 - *laptime;
    *laptime = x.tv_sec + x.tv_usec/1000000.0;
}


void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}


void KernelCPU(float* C, const float* A, const float* B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    for (unsigned int i = 0; i < hA; ++i)
        for (unsigned int j = 0; j < wB; ++j) {
            double sum = 0;
            for (unsigned int k = 0; k < wA; ++k) {
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }
            C[i * wB + j] = (float)sum;
        }
}



int main(int argc, char** argv)
{
	printf("\n\n[matrixMul starting...]\n");

	char *as=NULL;
	char num ='1';

	int devID=0;
	int iSizeMultiple=5;
	hipDeviceProp_t props;
	double lt=0.0, st=0.0;

	cutilSafeCall(hipGetDeviceCount(&devID));
	printf("\n# %d device%s found.\n", devID, devID > 1 ? "s" : "");
	devID=0;
	cutilSafeCall(hipSetDevice(devID));
	cutilSafeCall(hipGetDeviceProperties(&props, devID));
	int block_size = 32;
	
	printf("\n|||||||||||||||||||||||||||||||||||||||||DS_CUDA mulMatrix....|||||");
	printf("\n\nDevice %d: \"%s\" with Compute %d.%d capability\n", 0, props.name, props.major, props.minor);
	
	
	unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;

	
	
	if ( argc < 2 ) 
		{
			iSizeMultiple = 5;
		}
		else
		{
			
			as = argv[1];

			for(int i=1;i<11;i++)
				{
					if(num == as[0])
					{
						iSizeMultiple = i;
						break;
					}
					num++;
				}
		}
	
	
	
	if (props.multiProcessorCount <= 4) {
		uiWA = 2 * block_size * iSizeMultiple;
		uiHA = 4 * block_size * iSizeMultiple;
		uiWB = 2 * block_size * iSizeMultiple;
		uiHB = 4 * block_size * iSizeMultiple;
		uiWC = 2 * block_size * iSizeMultiple;
		uiHC = 4 * block_size * iSizeMultiple;
	} else {
		uiWA = WA * iSizeMultiple;
		uiHA = HA * iSizeMultiple;
		uiWB = WB * iSizeMultiple;
		uiHB = HB * iSizeMultiple;
		uiWC = WC * iSizeMultiple;
		uiHC = HC * iSizeMultiple;
	}


	
	
	printf("\nMatrix Sizes:A(%u x %u), B(%u x %u), C(%u x %u)\n\n",uiWA, uiHA, uiWB, uiHB, uiWC, uiHC);
	dim3 threads(block_size, block_size);
	dim3 grid(uiWC / threads.x, uiHC / threads.y);
	
	printf("\nRunning kernels......\n");
	int nIter = 60;
	printf("\nNumber of iterations for each kernel %i",nIter);
	
	
	
	unsigned int size_A = uiWA * uiHA;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float* h_A = (float*)malloc(mem_size_A);
	
	unsigned int size_B = uiWB * uiHB;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float* h_B = (float*)malloc(mem_size_B);
	
	unsigned int size_C = uiWC * uiHC;	
	unsigned int mem_size_C = sizeof(float) * size_C;
	float* h_C      = (float*) malloc(mem_size_C);
	
	
	srand(2013);
	randomInit(h_A, size_A);
	randomInit(h_B, size_B);

	
	
	float* d_A, *d_B, *d_C;
	
	cutilSafeCall(hipMalloc((void**) &d_A, mem_size_A));
	cutilSafeCall(hipMalloc((void**) &d_B, mem_size_B));
	cutilSafeCall(hipMalloc((void**) &d_C, mem_size_C));
	
	
	cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) );
	cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) );
	printf("\nTotal amount of memory to be sent from CPU to GPU: %d Bytes",mem_size_A+mem_size_B);

	
	
	
	dscudamatrixMulDS(grid, threads , 0, NULL, d_C, d_A, d_B, uiWA, uiWB);
	
	hipDeviceSynchronize();
	
	
	
	get_cputime(&lt,&st);
	for (int j = 0;j< nIter;j++){
		dscudamatrixMulDS(grid, threads , 0, NULL, d_C, d_A, d_B, uiWA, uiWB);
		
	}
	hipDeviceSynchronize();
	get_cputime(&lt,&st);

	
	
	cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost) );
	

	
	printf("\nTotal amount of memory to be sent from GPU to CPU: %d Bytes",mem_size_C);
	double dSeconds = lt/((double)nIter);
	double dNumOps = 2.0 * (double)uiWA * (double)uiHA * (double)uiWB;
	double gflops = 1.0e-9 * dNumOps/dSeconds;
	printf("\n>DSCUDA\t%.4f GFlop/s,\tTime:%.8f s,\tSize:%.0f Ops ",gflops,dSeconds,dNumOps);

#if 1	
	
	lt = 0.0;
	st = 0.0;
	nIter = 2;
	dSeconds = 0.0;
	dNumOps = 0.0;
	gflops = 0.0;
	bool correct = true;
	float* reference = (float*)malloc(mem_size_C);
	
	
	get_cputime(&lt,&st);
	
	for(int i=0;i<nIter;i++)
	{
		KernelCPU(reference, h_A, h_B, uiHA, uiWA, uiWB);
		
	}
	get_cputime(&lt,&st);
	
	
	
	dSeconds = lt/((double)nIter);
	dNumOps = 2.0 * (double)uiWA * (double)uiHA * (double)uiWB;
	gflops = 1.0e-9 * dNumOps/dSeconds;
	printf("\n> CPU\t\t%.4f GFlop/s\t, Time:%.5f s, Size:%.0f Ops ",gflops,dSeconds,dNumOps);

	printf("\n\nComparing GPU results with CPU calculation...");
	
	for (int i = 0; i < size_C; i++)
	    {
	        if (fabs(h_C[i] - reference[i]) > 1e-3)
	        {
	            printf("Error! GPUmem[%05d]=%.8f, CPUmem=%.8f error term is %.8f > 1e-3\n", i, h_C[i],reference[i],fabs(h_C[i] - reference[i]));
	            correct = false;
	        }
	    }

	printf("%s\n", correct ? "OK" : "FAIL");
	if (correct){
		printf("\nMatrix");
		for (int i=0;i<10;i++){
			printf("\nGPUmem[%05d]=%.8f, CPUmem=%.8f ---- Difference... %.8f < 1e-3", i, h_C[i],reference[i],fabs(h_C[i] - reference[i]));
		}
	}
	
	
	free(reference);
	
#endif

	
	free(h_A);
	free(h_B);
	free(h_C);
	
	cutilSafeCall(hipFree(d_A));
	cutilSafeCall(hipFree(d_B));
	cutilSafeCall(hipFree(d_C));
	printf("\n\nExit mulMatrix Program...\n\n");

	
    exit(0);
    return 0;
}
