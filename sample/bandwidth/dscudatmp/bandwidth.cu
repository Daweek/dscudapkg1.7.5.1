static char *Ptxdata = 
    "	.version 1.4\n"
    "	.target sm_10, map_f64_to_f32\n"
    "	// compiled with /usr/local/cuda-6.0/open64/lib//be\n"
    "	// nvopencc 4.1 built on 2014-03-13\n"
    "\n"
    "	//-----------------------------------------------------------\n"
    "	// Compiling /tmp/tmpxft_00004525_00000000-9_bandwidth.cpp3.i (/tmp/ccBI#.XDM1kw)\n"
    "	//-----------------------------------------------------------\n"
    "\n"
    "	//-----------------------------------------------------------\n"
    "	// Options:\n"
    "	//-----------------------------------------------------------\n"
    "	//  Target:ptx, ISA:sm_10, Endian:little, Pointer Size:32\n"
    "	//  -O3	(Optimization level)\n"
    "	//  -g0	(Debug level)\n"
    "	//  -m2	(Report advisories)\n"
    "	//-----------------------------------------------------------\n"
    "\n"
    "	.file	1	\"<command-line>\"\n"
    "	.file	2	\"/usr/include/stdc-predef.h\"\n"
    "	.file	3	\"/tmp/tmpxft_00004525_00000000-8_bandwidth.cudafe2.gpu\"\n"
    "	.file	4	\"/usr/lib/gcc/i686-linux-gnu/4.8/include/stddef.h\"\n"
    "	.file	5	\"/usr/local/cuda/include/crt/device_runtime.h\"\n"
    "	.file	6	\"/usr/local/cuda/include/host_defines.h\"\n"
    "	.file	7	\"/usr/local/cuda/include/builtin_types.h\"\n"
    "	.file	8	\"/usr/local/cuda/include/device_types.h\"\n"
    "	.file	9	\"/usr/local/cuda/include/hip/driver_types.h\"\n"
    "	.file	10	\"/usr/local/cuda/include/surface_types.h\"\n"
    "	.file	11	\"/usr/local/cuda/include/texture_types.h\"\n"
    "	.file	12	\"/usr/local/cuda/include/hip/hip_vector_types.h\"\n"
    "	.file	13	\"/usr/local/cuda/include/\"\n"
    "	.file	14	\"/usr/local/cuda/include/crt/storage_class.h\"\n"
    "	.file	15	\"/usr/local/cuda/include/common_functions.h\"\n"
    "	.file	16	\"/usr/local/cuda/include/math_functions.h\"\n"
    "	.file	17	\"/usr/local/cuda/include/hip/hip_math_constants.h\"\n"
    "	.file	18	\"/usr/local/cuda/include/hip/device_functions.h\"\n"
    "	.file	19	\"/usr/local/cuda/include/sm_11_atomic_functions.h\"\n"
    "	.file	20	\"/usr/local/cuda/include/sm_12_atomic_functions.h\"\n"
    "	.file	21	\"/usr/local/cuda/include/sm_13_double_functions.h\"\n"
    "	.file	22	\"/usr/local/cuda/include/sm_20_atomic_functions.h\"\n"
    "	.file	23	\"/usr/local/cuda/include/sm_32_atomic_functions.h\"\n"
    "	.file	24	\"/usr/local/cuda/include/sm_35_atomic_functions.h\"\n"
    "	.file	25	\"/usr/local/cuda/include/sm_20_intrinsics.h\"\n"
    "	.file	26	\"/usr/local/cuda/include/sm_30_intrinsics.h\"\n"
    "	.file	27	\"/usr/local/cuda/include/sm_32_intrinsics.h\"\n"
    "	.file	28	\"/usr/local/cuda/include/sm_35_intrinsics.h\"\n"
    "	.file	29	\"/usr/local/cuda/include/surface_functions.h\"\n"
    "	.file	30	\"/usr/local/cuda/include/\"\n"
    "	.file	31	\"/usr/local/cuda/include/texture_indirect_functions.h\"\n"
    "	.file	32	\"/usr/local/cuda/include/surface_indirect_functions.h\"\n"
    "	.file	33	\"/usr/local/cuda/include/math_functions_dbl_ptx1.h\"\n"
    "\n"
    "\n";
#pragma dscuda endofptx
#include "dscuda.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <cutil.h>
#include <cutil_inline.h>

#define MAXDEV 2
#define NLOOP 8e8
#define PKG	  1024
#define MAXSIZE 1024*1024*300
static const double MEGA  = 1e6;
static const double MICRO = 1e-6;

static void
get_cputime(double *nowp, double *deltap)
{
    struct timeval t;
    double now0;

    gettimeofday(&t, NULL);
    now0 = t.tv_sec + t.tv_usec/1000000.0;
    *deltap = now0 - *nowp;
    *nowp   = now0;
}

static void
sendperf(int argc, char **argv)
{
    int maxsize = MAXSIZE;
    int i, j;
    size_t size;
    double sized;
    double now = 0.0, dt = 0.0;
    double ratio = 2;
    double nloop = NLOOP;
    char *src[MAXDEV];
    char *dst[MAXDEV];
    int ndev;

    ndev = 1; // !!!

    printf("# %d device%s found.\n", ndev, ndev > 1 ? "s" : "");
    for (i = 0; i < ndev; i++) {
        cutilSafeCall(hipMalloc((void**) &dst[i], sizeof(char) * maxsize));
        src[i] = (char *)malloc(sizeof(char) * maxsize);
    }
    printf("\n#\n# hipMemcpy (HostToDevice)\n#\n");

    for (sized = PKG; sized < maxsize; sized *= ratio) {
        size = (size_t)sized;
        get_cputime(&now, &dt);
        for (j = 0; j < nloop/size; j++) {
        	for (i = 0; i < ndev; i++) {
        		hipMemcpy(dst[i], src[i], size, hipMemcpyHostToDevice);
        	}
        }
        hipDeviceSynchronize();
        get_cputime(&now, &dt);
        printf("%d byte    %f sec    %f MB/s\n", size, dt, nloop/MEGA/dt);
    }
    cutilSafeCall(hipFree(dst[0]));
}

static void
receiveperf(int argc, char **argv)
{
    int maxsize = MAXSIZE;
    int i, j;
    size_t size;
    double sized;
    double now = 0.0, dt = 0.0;
    double ratio = 2;
    double nloop = NLOOP;
    char *src[MAXDEV];
    char *dst[MAXDEV];
    int ndev;

    ndev = 1; // !!!

    printf("# %d device%s found.\n", ndev, ndev > 1 ? "s" : "");
    for (i = 0; i < ndev; i++) {
    	cutilSafeCall(hipMalloc((void**) &src[i], sizeof(char) * maxsize));
    	dst[i] = (char *)malloc(sizeof(char) * maxsize);
    }
    printf("\n#\n# hipMemcpy (DeviceToHost)\n#\n");

    for (sized = PKG; sized < maxsize; sized *= ratio) {
    	size = (size_t)sized;
		get_cputime(&now, &dt);
		for (j = 0; j < nloop/size; j++) {
			for (i = 0; i < ndev; i++) {
				hipMemcpy(dst[i], src[i], size, hipMemcpyDeviceToHost);
		}
	}
		hipDeviceSynchronize();
		get_cputime(&now, &dt);
		printf("%d byte    %f sec    %f MB/s\n",size, dt, nloop/MEGA/dt);
	}
    cutilSafeCall(hipFree(src[0]));
}

int main(int argc, char **argv)
{
	fprintf(stderr,"Starting Bandwidth Test...\n");
	printf("Info:\nMax size:%d Byte\nPKGsize:%d Byte\nLOOP:%d\n\n",(int)MAXSIZE,(int)PKG,(int)NLOOP);
    sendperf(argc, argv);
    receiveperf(argc, argv);

    fprintf(stderr, "Finishing Bandwidth Test...\n");
    return 0;
}
