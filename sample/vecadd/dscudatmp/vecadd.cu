static char *Ptxdata = 
    "	.version 1.4\n"
    "	.target sm_10, map_f64_to_f32\n"
    "	// compiled with /usr/local/cuda-6.0/open64/lib//be\n"
    "	// nvopencc 4.1 built on 2014-03-13\n"
    "\n"
    "	//-----------------------------------------------------------\n"
    "	// Compiling /tmp/tmpxft_00006b0d_00000000-9_vecadd.cpp3.i (/tmp/ccBI#.N85Oiu)\n"
    "	//-----------------------------------------------------------\n"
    "\n"
    "	//-----------------------------------------------------------\n"
    "	// Options:\n"
    "	//-----------------------------------------------------------\n"
    "	//  Target:ptx, ISA:sm_10, Endian:little, Pointer Size:32\n"
    "	//  -O3	(Optimization level)\n"
    "	//  -g0	(Debug level)\n"
    "	//  -m2	(Report advisories)\n"
    "	//-----------------------------------------------------------\n"
    "\n"
    "	.file	1	\"<command-line>\"\n"
    "	.file	2	\"/usr/include/stdc-predef.h\"\n"
    "	.file	3	\"/tmp/tmpxft_00006b0d_00000000-8_vecadd.cudafe2.gpu\"\n"
    "	.file	4	\"/usr/lib/gcc/i686-linux-gnu/4.8/include/stddef.h\"\n"
    "	.file	5	\"/usr/local/cuda/include/crt/device_runtime.h\"\n"
    "	.file	6	\"/usr/local/cuda/include/host_defines.h\"\n"
    "	.file	7	\"/usr/local/cuda/include/builtin_types.h\"\n"
    "	.file	8	\"/usr/local/cuda/include/device_types.h\"\n"
    "	.file	9	\"/usr/local/cuda/include/hip/driver_types.h\"\n"
    "	.file	10	\"/usr/local/cuda/include/surface_types.h\"\n"
    "	.file	11	\"/usr/local/cuda/include/texture_types.h\"\n"
    "	.file	12	\"/usr/local/cuda/include/hip/hip_vector_types.h\"\n"
    "	.file	13	\"/usr/local/cuda/include/\"\n"
    "	.file	14	\"/usr/local/cuda/include/crt/storage_class.h\"\n"
    "	.file	15	\"vecadd.cuh\"\n"
    "	.file	16	\"/usr/local/cuda/include/common_functions.h\"\n"
    "	.file	17	\"/usr/local/cuda/include/math_functions.h\"\n"
    "	.file	18	\"/usr/local/cuda/include/hip/hip_math_constants.h\"\n"
    "	.file	19	\"/usr/local/cuda/include/hip/device_functions.h\"\n"
    "	.file	20	\"/usr/local/cuda/include/sm_11_atomic_functions.h\"\n"
    "	.file	21	\"/usr/local/cuda/include/sm_12_atomic_functions.h\"\n"
    "	.file	22	\"/usr/local/cuda/include/sm_13_double_functions.h\"\n"
    "	.file	23	\"/usr/local/cuda/include/sm_20_atomic_functions.h\"\n"
    "	.file	24	\"/usr/local/cuda/include/sm_32_atomic_functions.h\"\n"
    "	.file	25	\"/usr/local/cuda/include/sm_35_atomic_functions.h\"\n"
    "	.file	26	\"/usr/local/cuda/include/sm_20_intrinsics.h\"\n"
    "	.file	27	\"/usr/local/cuda/include/sm_30_intrinsics.h\"\n"
    "	.file	28	\"/usr/local/cuda/include/sm_32_intrinsics.h\"\n"
    "	.file	29	\"/usr/local/cuda/include/sm_35_intrinsics.h\"\n"
    "	.file	30	\"/usr/local/cuda/include/surface_functions.h\"\n"
    "	.file	31	\"/usr/local/cuda/include/\"\n"
    "	.file	32	\"/usr/local/cuda/include/texture_indirect_functions.h\"\n"
    "	.file	33	\"/usr/local/cuda/include/surface_indirect_functions.h\"\n"
    "	.file	34	\"/usr/local/cuda/include/math_functions_dbl_ptx1.h\"\n"
    "\n"
    "\n"
    "	.entry _Z6vecAddPfS_S_ (\n"
    "		.param .u32 __cudaparm__Z6vecAddPfS_S__a,\n"
    "		.param .u32 __cudaparm__Z6vecAddPfS_S__b,\n"
    "		.param .u32 __cudaparm__Z6vecAddPfS_S__c)\n"
    "	{\n"
    "	.reg .u16 %rh<4>;\n"
    "	.reg .u32 %r<12>;\n"
    "	.reg .f32 %f<5>;\n"
    "	.loc	15	2	0\n"
    "$LDWbegin__Z6vecAddPfS_S_:\n"
    "	.loc	15	5	0\n"
    "	mov.u16 	%rh1, %ctaid.x;\n"
    "	mov.u16 	%rh2, %ntid.x;\n"
    "	mul.wide.u16 	%r1, %rh1, %rh2;\n"
    "	cvt.u32.u16 	%r2, %tid.x;\n"
    "	add.u32 	%r3, %r2, %r1;\n"
    "	mul.lo.u32 	%r4, %r3, 4;\n"
    "	ld.param.u32 	%r5, [__cudaparm__Z6vecAddPfS_S__a];\n"
    "	add.u32 	%r6, %r5, %r4;\n"
    "	ld.global.f32 	%f1, [%r6+0];\n"
    "	ld.param.u32 	%r7, [__cudaparm__Z6vecAddPfS_S__b];\n"
    "	add.u32 	%r8, %r7, %r4;\n"
    "	ld.global.f32 	%f2, [%r8+0];\n"
    "	add.f32 	%f3, %f1, %f2;\n"
    "	ld.param.u32 	%r9, [__cudaparm__Z6vecAddPfS_S__c];\n"
    "	add.u32 	%r10, %r9, %r4;\n"
    "	st.global.f32 	[%r10+0], %f3;\n"
    "	.loc	15	6	0\n"
    "	exit;\n"
    "$LDWend__Z6vecAddPfS_S_:\n"
    "	} // _Z6vecAddPfS_S_\n"
    "\n";
#pragma dscuda endofptx
#include "dscuda.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cutil.h>
#include <cutil_inline.h>

#include "vecadd.cuh"

#define N (8)

int
main(void)
{
	int i, t;
    float a[N], b[N], c[N];

    float *d_a, *d_b, *d_c;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_a, sizeof(float) * N));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_b, sizeof(float) * N));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_c, sizeof(float) * N));

    for (t = 0; t < 3; t++) {
        printf("try %d\n", t);
        for (i = 0; i < N; i++) {
            a[i] = rand()%64;
            b[i] = rand()%64;
        }
        CUDA_SAFE_CALL(hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice));
        int nth = 4;
        dim3 threads(nth, 1, 1);
        dim3 grids((N+nth-1)/nth, 1, 1);
        vecAdd<<<grids, threads>>>(d_a, d_b, d_c);
        CUDA_SAFE_CALL(hipMemcpy(c, d_c, sizeof(float) * N, hipMemcpyDeviceToHost));
        for (i = 0; i < N; i++) {
            printf("% 6.2f + % 6.2f = % 7.2f",
                   a[i], b[i], c[i]);
            if (a[i] + b[i] != c[i]) printf("   NG");
            printf("\n");
        }
        printf("\n");
    }

    exit(0);
}
