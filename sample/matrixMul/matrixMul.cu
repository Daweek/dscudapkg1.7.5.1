#include "hip/hip_runtime.h"
// Martinez Noriega Edgar Josafat     14/05/2013
// Based on CUDA SDK 4.1 MatrixMul

// C includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

// CUDA Includes
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
#include <cutil.h>
#include <cutil_inline.h>
//DSCUDA Includes
//#include "dscuda.h"
//CUBLAS include
//#include <hipblas.h>
//#include <hip/hip_runtime.h>

// Kernel Includes
#include "matrixMulkernel.cu"

#define WA (4 * block_size) // Matrix A width
#define HA (6 * block_size) // Matrix A height
#define WB (4 * block_size) // Matrix B width
#define HB WA  // Matrix B height
#define WC WB  // Matrix C width 
#define HC HA  // Matrix C height
#define MEGA 1024*1024




///To measure the time...
static void
get_cputime(double *splittime, double *laptime)
{
    struct timeval x;

    gettimeofday(&x, NULL);

    *splittime = x.tv_sec + x.tv_usec/1000000.0 - *laptime;
    *laptime = x.tv_sec + x.tv_usec/1000000.0;
}

////Random numbers generator for matrix
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

////Matrix Multiplication routine in CPU
void KernelCPU(float* C, const float* A, const float* B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    for (unsigned int i = 0; i < hA; ++i)
        for (unsigned int j = 0; j < wB; ++j) {
            double sum = 0;
            for (unsigned int k = 0; k < wA; ++k) {
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }
            C[i * wB + j] = (float)sum;
        }
}

////////////////////////////////Main//////////////////////////////////
//////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
	printf("\n\n[matrixMul starting...]\n");

	char *as=NULL;
	char num ='1';

	int devID=0;
	int iSizeMultiple=5;
	hipDeviceProp_t props;
	double lt=0.0, st=0.0;

	cutilSafeCall(hipGetDeviceCount(&devID));
	printf("\n# %d device%s found.\n", devID, devID > 1 ? "s" : "");
	devID=0;
	cutilSafeCall(hipSetDevice(devID));
	cutilSafeCall(hipGetDeviceProperties(&props, devID));
	int block_size = 32;
	
	printf("\n|||||||||||||||||||||||||||||||||||||||||DS_CUDA mulMatrix....|||||");
	printf("\n\nDevice %d: \"%s\" with Compute %d.%d capability\n", 0, props.name, props.major, props.minor);
	
	// Optional Command-line multiplier for matrix sizes
	unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;

	//Checking for Arguments....
	////////////////////////////////////
	if ( argc < 2 ) /* argc should be 2 for correct execution */
		{
			iSizeMultiple = 5;
		}
		else
		{
			//ts = (int) argv[1];
			as = argv[1];

			for(int i=1;i<11;i++)
				{
					if(num == as[0])
					{
						iSizeMultiple = i;
						break;
					}
					num++;
				}
		}
	/////////////////////////////////////
	
	// For GPUs with fewer # of SM's, we limit the maximum size of the matrix
	if (props.multiProcessorCount <= 4) {
		uiWA = 2 * block_size * iSizeMultiple;
		uiHA = 4 * block_size * iSizeMultiple;
		uiWB = 2 * block_size * iSizeMultiple;
		uiHB = 4 * block_size * iSizeMultiple;
		uiWC = 2 * block_size * iSizeMultiple;
		uiHC = 4 * block_size * iSizeMultiple;
	} else {
		uiWA = WA * iSizeMultiple;
		uiHA = HA * iSizeMultiple;
		uiWB = WB * iSizeMultiple;
		uiHB = HB * iSizeMultiple;
		uiWC = WC * iSizeMultiple;
		uiHC = HC * iSizeMultiple;
	}


	//Variables for GPU kernell
	// setup execution parameters
	printf("\nMatrix Sizes:A(%u x %u), B(%u x %u), C(%u x %u)\n\n",uiWA, uiHA, uiWB, uiHB, uiWC, uiHC);
	dim3 threads(block_size, block_size);
	dim3 grid(uiWC / threads.x, uiHC / threads.y);
	
	printf("\nRunning kernels......\n");
	int nIter = 60;
	printf("\nNumber of iterations for each kernel %i",nIter);
	
	
	// allocate host memory for matrices A,B and C
	unsigned int size_A = uiWA * uiHA;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float* h_A = (float*)malloc(mem_size_A);
	
	unsigned int size_B = uiWB * uiHB;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float* h_B = (float*)malloc(mem_size_B);
	
	unsigned int size_C = uiWC * uiHC;	
	unsigned int mem_size_C = sizeof(float) * size_C;
	float* h_C      = (float*) malloc(mem_size_C);
	
	//Generate Aleatory Data
	srand(2013);
	randomInit(h_A, size_A);
	randomInit(h_B, size_B);

	
	// Allocate device (GPU) memory
	float* d_A, *d_B, *d_C;
	
	cutilSafeCall(hipMalloc((void**) &d_A, mem_size_A));
	cutilSafeCall(hipMalloc((void**) &d_B, mem_size_B));
	cutilSafeCall(hipMalloc((void**) &d_C, mem_size_C));
	
	// Copy data from CPU memory to GPU memory
	cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) );
	cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) );
	printf("\nTotal amount of memory to be sent from CPU to GPU: %d Bytes",mem_size_A+mem_size_B);

	
	
	// Make warmup operation and Synchronize GPU
	matrixMulDS<<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
	//matrixMul<32><<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
	hipDeviceSynchronize();
	
	
	// execute the kernel in GPU
	get_cputime(&lt,&st);
	for (int j = 0;j< nIter;j++){
		matrixMulDS<<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
		//matrixMul<32><<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
	}
	hipDeviceSynchronize();
	get_cputime(&lt,&st);

	
	// Copying memory back from GPU to CPU
	cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost) );
	

	//Calculating of Performance .... Gflops
	printf("\nTotal amount of memory to be sent from GPU to CPU: %d Bytes",mem_size_C);
	double dSeconds = lt/((double)nIter);
	double dNumOps = 2.0 * (double)uiWA * (double)uiHA * (double)uiWB;
	double gflops = 1.0e-9 * dNumOps/dSeconds;
	printf("\n>DSCUDA\t%.4f GFlop/s,\tTime:%.8f s,\tSize:%.0f Ops ",gflops,dSeconds,dNumOps);

#if 1	
	// Variables for measuring the time
	lt = 0.0;
	st = 0.0;
	nIter = 2;
	dSeconds = 0.0;
	dNumOps = 0.0;
	gflops = 0.0;
	bool correct = true;
	float* reference = (float*)malloc(mem_size_C);
	
	// execute kernel in CPU
	get_cputime(&lt,&st);
	
	for(int i=0;i<nIter;i++)
	{
		KernelCPU(reference, h_A, h_B, uiHA, uiWA, uiWB);
		
	}
	get_cputime(&lt,&st);
	
	
	
	dSeconds = lt/((double)nIter);
	dNumOps = 2.0 * (double)uiWA * (double)uiHA * (double)uiWB;
	gflops = 1.0e-9 * dNumOps/dSeconds;
	printf("\n> CPU\t\t%.4f GFlop/s\t, Time:%.5f s, Size:%.0f Ops ",gflops,dSeconds,dNumOps);

	printf("\n\nComparing GPU results with CPU calculation...");
	// Comparing Results between CPU and GPU calculations
	for (int i = 0; i < size_C; i++)
	    {
	        if (fabs(h_C[i] - reference[i]) > 1e-3)
	        {
	            printf("Error! GPUmem[%05d]=%.8f, CPUmem=%.8f error term is %.8f > 1e-3\n", i, h_C[i],reference[i],fabs(h_C[i] - reference[i]));
	            correct = false;
	        }
	    }

	printf("%s\n", correct ? "OK" : "FAIL");
	if (correct){
		printf("\nMatrix");
		for (int i=0;i<10;i++){
			printf("\nGPUmem[%05d]=%.8f, CPUmem=%.8f ---- Difference... %.8f < 1e-3", i, h_C[i],reference[i],fabs(h_C[i] - reference[i]));
		}
	}
	
	//Free memory
	free(reference);
	
#endif

	//Cleaning Memory....
	free(h_A);
	free(h_B);
	free(h_C);
	
	cutilSafeCall(hipFree(d_A));
	cutilSafeCall(hipFree(d_B));
	cutilSafeCall(hipFree(d_C));
	printf("\n\nExit mulMatrix Program...\n\n");

	
    exit(0);
    return 0;
}
