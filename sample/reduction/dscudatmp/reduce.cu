static char *Ptxdata = 
    "	.version 1.4\n"
    "	.target sm_10, map_f64_to_f32\n"
    "	// compiled with /usr/local/cuda-6.0/open64/lib//be\n"
    "	// nvopencc 4.1 built on 2014-03-13\n"
    "\n"
    "	//-----------------------------------------------------------\n"
    "	// Compiling /tmp/tmpxft_00001a01_00000000-9_reduce.cpp3.i (/tmp/ccBI#.LKNZh2)\n"
    "	//-----------------------------------------------------------\n"
    "\n"
    "	//-----------------------------------------------------------\n"
    "	// Options:\n"
    "	//-----------------------------------------------------------\n"
    "	//  Target:ptx, ISA:sm_10, Endian:little, Pointer Size:64\n"
    "	//  -O3	(Optimization level)\n"
    "	//  -g0	(Debug level)\n"
    "	//  -m2	(Report advisories)\n"
    "	//-----------------------------------------------------------\n"
    "\n"
    "	.file	1	\"<command-line>\"\n"
    "	.file	2	\"/tmp/tmpxft_00001a01_00000000-8_reduce.cudafe2.gpu\"\n"
    "	.file	3	\"/usr/lib/gcc/x86_64-linux-gnu/4.6/include/stddef.h\"\n"
    "	.file	4	\"/usr/local/cuda/include/crt/device_runtime.h\"\n"
    "	.file	5	\"/usr/local/cuda/include/host_defines.h\"\n"
    "	.file	6	\"/usr/local/cuda/include/builtin_types.h\"\n"
    "	.file	7	\"/usr/local/cuda/include/device_types.h\"\n"
    "	.file	8	\"/usr/local/cuda/include/hip/driver_types.h\"\n"
    "	.file	9	\"/usr/local/cuda/include/surface_types.h\"\n"
    "	.file	10	\"/usr/local/cuda/include/texture_types.h\"\n"
    "	.file	11	\"/usr/local/cuda/include/hip/hip_vector_types.h\"\n"
    "	.file	12	\"/usr/local/cuda/include/\"\n"
    "	.file	13	\"/usr/local/cuda/include/crt/storage_class.h\"\n"
    "	.file	14	\"reduce.cuh\"\n"
    "	.file	15	\"/usr/local/cuda/include/common_functions.h\"\n"
    "	.file	16	\"/usr/local/cuda/include/math_functions.h\"\n"
    "	.file	17	\"/usr/local/cuda/include/hip/hip_math_constants.h\"\n"
    "	.file	18	\"/usr/local/cuda/include/hip/device_functions.h\"\n"
    "	.file	19	\"/usr/local/cuda/include/sm_11_atomic_functions.h\"\n"
    "	.file	20	\"/usr/local/cuda/include/sm_12_atomic_functions.h\"\n"
    "	.file	21	\"/usr/local/cuda/include/sm_13_double_functions.h\"\n"
    "	.file	22	\"/usr/local/cuda/include/sm_20_atomic_functions.h\"\n"
    "	.file	23	\"/usr/local/cuda/include/sm_32_atomic_functions.h\"\n"
    "	.file	24	\"/usr/local/cuda/include/sm_35_atomic_functions.h\"\n"
    "	.file	25	\"/usr/local/cuda/include/sm_20_intrinsics.h\"\n"
    "	.file	26	\"/usr/local/cuda/include/sm_30_intrinsics.h\"\n"
    "	.file	27	\"/usr/local/cuda/include/sm_32_intrinsics.h\"\n"
    "	.file	28	\"/usr/local/cuda/include/sm_35_intrinsics.h\"\n"
    "	.file	29	\"/usr/local/cuda/include/surface_functions.h\"\n"
    "	.file	30	\"/usr/local/cuda/include/\"\n"
    "	.file	31	\"/usr/local/cuda/include/texture_indirect_functions.h\"\n"
    "	.file	32	\"/usr/local/cuda/include/surface_indirect_functions.h\"\n"
    "	.file	33	\"/usr/local/cuda/include/math_functions_dbl_ptx1.h\"\n"
    "\n"
    "	.extern	.shared .align 4 .b8 __smem[];\n"
    "\n"
    "	.entry _Z6reduceiPiS_ (\n"
    "		.param .s32 __cudaparm__Z6reduceiPiS__n,\n"
    "		.param .u64 __cudaparm__Z6reduceiPiS__g_idata,\n"
    "		.param .u64 __cudaparm__Z6reduceiPiS__g_odata)\n"
    "	{\n"
    "	.reg .u32 %r<20>;\n"
    "	.reg .u64 %rd<17>;\n"
    "	.reg .pred %p<7>;\n"
    "	.loc	14	2	0\n"
    "$LDWbegin__Z6reduceiPiS_:\n"
    "	cvt.u32.u16 	%r1, %ntid.x;\n"
    "	cvt.u32.u16 	%r2, %ctaid.x;\n"
    "	mul.lo.u32 	%r3, %r1, %r2;\n"
    "	cvt.u32.u16 	%r4, %tid.x;\n"
    "	add.u32 	%r5, %r3, %r4;\n"
    "	ld.param.s32 	%r6, [__cudaparm__Z6reduceiPiS__n];\n"
    "	setp.le.u32 	%p1, %r6, %r5;\n"
    "	@%p1 bra 	$Lt_0_3842;\n"
    "	.loc	14	11	0\n"
    "	ld.param.u64 	%rd1, [__cudaparm__Z6reduceiPiS__g_idata];\n"
    "	cvt.u64.u32 	%rd2, %r5;\n"
    "	mul.wide.u32 	%rd3, %r5, 4;\n"
    "	add.u64 	%rd4, %rd1, %rd3;\n"
    "	ld.global.s32 	%r7, [%rd4+0];\n"
    "	bra.uni 	$Lt_0_3586;\n"
    "$Lt_0_3842:\n"
    "	mov.s32 	%r7, 0;\n"
    "$Lt_0_3586:\n"
    "	mov.u64 	%rd5, __smem;\n"
    "	cvt.u64.u32 	%rd6, %r4;\n"
    "	mul.wide.u32 	%rd7, %r4, 4;\n"
    "	add.u64 	%rd8, %rd5, %rd7;\n"
    "	st.shared.s32 	[%rd8+0], %r7;\n"
    "	.loc	14	13	0\n"
    "	bar.sync 	0;\n"
    "	mov.u32 	%r8, 1;\n"
    "	setp.le.u32 	%p2, %r1, %r8;\n"
    "	@%p2 bra 	$Lt_0_4098;\n"
    "	mov.u32 	%r9, 1;\n"
    "$Lt_0_4610:\n"
    "	mul.lo.u32 	%r10, %r9, 2;\n"
    "	rem.u32 	%r11, %r4, %r10;\n"
    "	mov.u32 	%r12, 0;\n"
    "	setp.ne.u32 	%p3, %r11, %r12;\n"
    "	@%p3 bra 	$Lt_0_4866;\n"
    "	.loc	14	20	0\n"
    "	ld.shared.s32 	%r13, [%rd8+0];\n"
    "	add.u32 	%r14, %r9, %r4;\n"
    "	cvt.u64.u32 	%rd9, %r14;\n"
    "	mul.wide.u32 	%rd10, %r14, 4;\n"
    "	add.u64 	%rd11, %rd5, %rd10;\n"
    "	ld.shared.s32 	%r15, [%rd11+0];\n"
    "	add.s32 	%r16, %r13, %r15;\n"
    "	st.shared.s32 	[%rd8+0], %r16;\n"
    "$Lt_0_4866:\n"
    "	.loc	14	22	0\n"
    "	bar.sync 	0;\n"
    "	.loc	14	17	0\n"
    "	mov.s32 	%r9, %r10;\n"
    "	setp.lt.u32 	%p4, %r10, %r1;\n"
    "	@%p4 bra 	$Lt_0_4610;\n"
    "$Lt_0_4098:\n"
    "	mov.u32 	%r17, 0;\n"
    "	setp.ne.u32 	%p5, %r4, %r17;\n"
    "	@%p5 bra 	$Lt_0_5634;\n"
    "	.loc	14	27	0\n"
    "	ld.shared.s32 	%r18, [__smem+0];\n"
    "	ld.param.u64 	%rd12, [__cudaparm__Z6reduceiPiS__g_odata];\n"
    "	cvt.u64.u32 	%rd13, %r2;\n"
    "	mul.wide.u32 	%rd14, %r2, 4;\n"
    "	add.u64 	%rd15, %rd12, %rd14;\n"
    "	st.global.s32 	[%rd15+0], %r18;\n"
    "$Lt_0_5634:\n"
    "	.loc	14	29	0\n"
    "	exit;\n"
    "$LDWend__Z6reduceiPiS_:\n"
    "	} // _Z6reduceiPiS_\n"
    "\n";
#pragma dscuda endofptx
#include "dscuda.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cutil.h>
#include <cutil_inline.h>

#include "reduce.cuh"

#define NTHREAD    (64)
#define NBLOCKMAX  (65536)

int
main(int argc, char **argv)
{
    int i, nelem, nblock;
    int sum, h_sum;
    int *h_idata, *h_odata;
    int *d_idata, *d_odata;

    if (argc < 2) {
        fprintf(stderr, "usage: %s <# of elements>\n", argv[0]);
        exit(1);
    }
    nelem = atoi(argv[1]);
    nblock = (nelem - 1) / NTHREAD + 1;
    if (NBLOCKMAX < nblock) {
        fprintf(stderr, "# of elements exceeds the limit (=%d).\n", NTHREAD * NBLOCKMAX);
        exit(1);
    }
    fprintf(stderr, "nelem:%d  nthread:%d  nblock:%d\n", nelem, NTHREAD, nblock);

    h_idata = (int *)malloc(sizeof(int) * nelem);
    h_odata = (int *)malloc(sizeof(int) * nblock);
    cutilSafeCall(hipMalloc((void**) &d_idata, sizeof(int) * nelem));
    cutilSafeCall(hipMalloc((void**) &d_odata, sizeof(int) * nblock));

    h_sum = 0;
    for (i = 0; i < nelem; i++) {
        h_idata[i] = lrand48() % (1 << 8);
        h_sum += h_idata[i];
    }
    cutilSafeCall(hipMemcpy(d_idata, h_idata, sizeof(int) * nelem, hipMemcpyHostToDevice));

    for (i = 0; i < nblock; i++) {
        h_odata[i] = 0;
    }
    cutilSafeCall(hipMemcpy(d_odata, h_odata, sizeof(int) * nblock, hipMemcpyHostToDevice));

    dim3 threads(NTHREAD, 1, 1);
    dim3 grids(nblock, 1, 1);
    int smemsize = sizeof(int) * NTHREAD;

    reduce<<<grids, threads, smemsize>>>(nelem, d_idata, d_odata);

    cutilSafeCall(hipMemcpy(h_odata, d_odata, sizeof(int) * nblock, hipMemcpyDeviceToHost));    

    sum = 0;
    for (i = 0; i < nblock; i++) {
        fprintf(stderr, "block[%d]:%d\n", i, h_odata[i]);
        sum += h_odata[i];
    }
    printf("  sum: %d\n", sum);
    printf("h_sum: %d\n", h_sum);
}
