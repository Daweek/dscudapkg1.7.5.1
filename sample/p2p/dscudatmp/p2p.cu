static char *Ptxdata = 
    "	.version 1.4\n"
    "	.target sm_10, map_f64_to_f32\n"
    "	// compiled with /usr/local/cuda-4.1/open64/lib//be\n"
    "	// nvopencc 4.1 built on 2012-01-12\n"
    "\n"
    "	//-----------------------------------------------------------\n"
    "	// Compiling /tmp/tmpxft_00001d0b_00000000-9_p2p.cpp3.i (/tmp/ccBI#.F1D2le)\n"
    "	//-----------------------------------------------------------\n"
    "\n"
    "	//-----------------------------------------------------------\n"
    "	// Options:\n"
    "	//-----------------------------------------------------------\n"
    "	//  Target:ptx, ISA:sm_10, Endian:little, Pointer Size:64\n"
    "	//  -O3	(Optimization level)\n"
    "	//  -g0	(Debug level)\n"
    "	//  -m2	(Report advisories)\n"
    "	//-----------------------------------------------------------\n"
    "\n"
    "	.file	1	\"<command-line>\"\n"
    "	.file	2	\"/tmp/tmpxft_00001d0b_00000000-8_p2p.cudafe2.gpu\"\n"
    "	.file	3	\"/usr/lib/gcc/x86_64-linux-gnu/4.4.7/include/stddef.h\"\n"
    "	.file	4	\"/usr/local/cuda/include/crt/device_runtime.h\"\n"
    "	.file	5	\"/usr/local/cuda/include/host_defines.h\"\n"
    "	.file	6	\"/usr/local/cuda/include/builtin_types.h\"\n"
    "	.file	7	\"/usr/local/cuda/include/device_types.h\"\n"
    "	.file	8	\"/usr/local/cuda/include/hip/driver_types.h\"\n"
    "	.file	9	\"/usr/local/cuda/include/surface_types.h\"\n"
    "	.file	10	\"/usr/local/cuda/include/texture_types.h\"\n"
    "	.file	11	\"/usr/local/cuda/include/hip/hip_vector_types.h\"\n"
    "	.file	12	\"/usr/local/cuda/include/\"\n"
    "	.file	13	\"/usr/local/cuda/include/crt/storage_class.h\"\n"
    "	.file	14	\"/usr/local/cuda/include/common_functions.h\"\n"
    "	.file	15	\"/usr/local/cuda/include/math_functions.h\"\n"
    "	.file	16	\"/usr/local/cuda/include/hip/hip_math_constants.h\"\n"
    "	.file	17	\"/usr/local/cuda/include/hip/device_functions.h\"\n"
    "	.file	18	\"/usr/local/cuda/include/sm_11_atomic_functions.h\"\n"
    "	.file	19	\"/usr/local/cuda/include/sm_12_atomic_functions.h\"\n"
    "	.file	20	\"/usr/local/cuda/include/sm_13_double_functions.h\"\n"
    "	.file	21	\"/usr/local/cuda/include/sm_20_atomic_functions.h\"\n"
    "	.file	22	\"/usr/local/cuda/include/sm_20_intrinsics.h\"\n"
    "	.file	23	\"/usr/local/cuda/include/surface_functions.h\"\n"
    "	.file	24	\"/usr/local/cuda/include/\"\n"
    "	.file	25	\"/usr/local/cuda/include/math_functions_dbl_ptx1.h\"\n"
    "\n"
    "\n";
#pragma dscuda endofptx
#include "dscuda.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#include <cutil.h>
#include <cutil_inline.h>

static const double MEGA  = 1e6;

enum {
    COPY_BY_MEMCPY,
    COPY_BY_MEMCPYPEER,
};

// warn CUDA API errors, but do not exit.
#define unsafeCall(err)           __unsafeCall   (err, __FILE__, __LINE__)

static inline void
__unsafeCall(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err) {
        fprintf(stderr, "%s(%i) : cudaSafeCall() Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
    }
}

static void
get_cputime(double *nowp, double *deltap)
{
    struct timeval t;
    double now0;

    gettimeofday(&t, NULL);
    now0 = t.tv_sec + t.tv_usec/1000000.0;
    *deltap = now0 - *nowp;
    *nowp   = now0;
}

int
main(int argc, char **argv)
{
    double ratio = 2.5;
    double nloop = 2e8;
    double sized;
    double now, dt;
    int maxsize = 1024 * 1024 * 10.0;
    int i, j, size;
    int *bufA, *bufB;
    int *dbufA, *dbufB;
    int srcdev, dstdev;
    int copyapi;
    char copyapistr[128];

    if (argc < 4) {
        fprintf(stderr,
                "copies an array of int from one device to another.\n"
                "usage: %s <c|p> <src_dev> <dst_dev>\n"
                "           'c' for hipMemcpy()\n"
                "           'p' for hipMemcpyPeer()\n",
                argv[0]);
        exit(1);
    }
    switch (argv[1][0]) {
      case 'c':
        copyapi = COPY_BY_MEMCPY;
        sprintf(copyapistr, "hipMemcpy() ");
        break;
      case 'p':
        copyapi = COPY_BY_MEMCPYPEER;
        sprintf(copyapistr, "hipMemcpyPeer() ");
        break;
      default:
        fprintf(stderr, "arg1 should be 'c' or 'p'.\n");
        exit(1);
    }

    srcdev = atoi(argv[2]);
    dstdev = atoi(argv[3]);
    fprintf(stderr, "%s from device %d to device %d.\n",
            copyapistr, srcdev, dstdev);

    bufA = (int *)malloc(maxsize);
    bufB = (int *)malloc(maxsize);

    cutilSafeCall(hipSetDevice(srcdev));
    hipMalloc((void**) &dbufA, maxsize);
    cutilSafeCall(hipSetDevice(dstdev));
    hipMalloc((void**) &dbufB, maxsize);

    // set randomly generated data to the source device.
    for (i = 0; i < maxsize / sizeof(int); i++) {
        bufA[i] = rand() % 64;
        bufB[i] = rand() % 64;
    }
    cutilSafeCall(hipSetDevice(srcdev));
    cutilSafeCall(hipMemcpy(dbufA, bufA, maxsize, hipMemcpyHostToDevice));
    cutilSafeCall(hipSetDevice(dstdev));

    // copy data from the source device to the destination device.
    if (copyapi == COPY_BY_MEMCPY) {
        cutilSafeCall(hipMemcpy(dbufB, dbufA, maxsize, hipMemcpyDefault));
    }
    else {
        unsafeCall(hipMemcpyPeer(dbufB, dstdev, dbufA, srcdev, maxsize));
    }

    // send the data back from the destination device to the host.
    cutilSafeCall(hipMemcpy(bufB, dbufB, maxsize, hipMemcpyDeviceToHost));

    // comparing the result with the original to check the correctness of
    // hipMemcpy() / hipMemcpyPeer()
    for (i = 0; i < maxsize / sizeof(int); i++) {
        if (bufA[i] != bufB[i]) {
            fprintf(stderr, "NG\n");
            fprintf(stderr, "bufA[%d]:0x%08x\n", i, bufA[i]);
            fprintf(stderr, "bufB[%d]:0x%08x\n", i, bufB[i]);
            exit(1);
        }
    }
    fprintf(stderr, "OK\n");

    // measure the bandwidth.
    for (sized = 4096; sized < maxsize; sized *= ratio) {
        //    for ( nloop = 2e8, sized = 4096 * 1; ; ) { // !!!
        size = (size_t)sized;

	get_cputime(&now, &dt);
	for (j = 0; j < nloop/size; j++) {

            if (copyapi == COPY_BY_MEMCPY) {
                hipMemcpy(dbufB, dbufA, size, hipMemcpyDefault);
            }
            else {
                hipMemcpyPeer(dbufB, dstdev, dbufA, srcdev, size);
            }
        }
        hipDeviceSynchronize();
	get_cputime(&now, &dt);
	printf("%d byte    %f sec    %f MB/s\n",
               size, dt, nloop/MEGA/dt);
	fflush(stdout);
    }

    exit(0);
}
